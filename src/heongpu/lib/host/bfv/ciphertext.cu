// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bfv/ciphertext.cuh"

namespace heongpu
{
    __host__
    Ciphertext<Scheme::BFV>::Ciphertext(HEContext<Scheme::BFV>& context,
                                        const ExecutionOptions& options)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        coeff_modulus_count_ = context.Q_size;
        cipher_size_ = 2;
        ring_size_ = context.n;

        int cipher_memory_size =
            cipher_size_ * coeff_modulus_count_ * ring_size_;

        in_ntt_domain_ =
            (static_cast<int>(scheme_) == static_cast<int>(scheme_type::ckks))
                ? true
                : false;

        relinearization_required_ = false;

        storage_type_ = options.storage_;

        if (storage_type_ == storage_type::DEVICE)
        {
            device_locations_ =
                DeviceVector<Data64>(cipher_memory_size, options.stream_);
        }
        else
        {
            host_locations_ = HostVector<Data64>(cipher_memory_size);
        }
    }

    void Ciphertext<Scheme::BFV>::store_in_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            if (memory_size() == 0)
            {
                // pass
            }
            else
            {
                device_locations_ =
                    DeviceVector<Data64>(host_locations_, stream);
                host_locations_.resize(0);
                host_locations_.shrink_to_fit();
            }

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Ciphertext<Scheme::BFV>::store_in_host(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            if (memory_size() == 0)
            {
                // pass
            }
            else
            {
                int cipher_memory_size =
                    cipher_size_ * coeff_modulus_count_ * ring_size_;
                host_locations_ = HostVector<Data64>(cipher_memory_size);
                hipMemcpyAsync(host_locations_.data(),
                                device_locations_.data(),
                                cipher_memory_size * sizeof(Data64),
                                hipMemcpyDeviceToHost, stream);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                device_locations_.resize(0, stream);
                device_locations_.shrink_to_fit(stream);
            }

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    Data64* Ciphertext<Scheme::BFV>::data()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_locations_.data();
        }
        else
        {
            return host_locations_.data();
        }
    }

    void Ciphertext<Scheme::BFV>::get_data(std::vector<Data64>& cipher,
                                           hipStream_t stream)
    {
        int cipher_memory_size =
            cipher_size_ * coeff_modulus_count_ * ring_size_;

        if (cipher.size() < cipher_memory_size)
        {
            cipher.resize(cipher_memory_size);
        }

        if (storage_type_ == storage_type::DEVICE)
        {
            hipMemcpyAsync(cipher.data(), device_locations_.data(),
                            cipher_memory_size * sizeof(Data64),
                            hipMemcpyDeviceToHost, stream);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            std::memcpy(cipher.data(), host_locations_.data(),
                        host_locations_.size() * sizeof(Data64));
        }
    }

    void Ciphertext<Scheme::BFV>::get_data(HostVector<Data64>& cipher,
                                           hipStream_t stream)
    {
        int cipher_memory_size =
            cipher_size_ * coeff_modulus_count_ * ring_size_;

        if (cipher.size() < cipher_memory_size)
        {
            cipher.resize(cipher_memory_size);
        }

        if (storage_type_ == storage_type::DEVICE)
        {
            hipMemcpyAsync(cipher.data(), device_locations_.data(),
                            cipher_memory_size * sizeof(Data64),
                            hipMemcpyDeviceToHost, stream);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            std::memcpy(cipher.data(), host_locations_.data(),
                        host_locations_.size() * sizeof(Data64));
        }
    }

    void Ciphertext<Scheme::BFV>::save(std::ostream& os) const
    {
        if (ciphertext_generated_)
        {
            os.write((char*) &scheme_, sizeof(scheme_));

            if (scheme_ != scheme_type::bfv)
            {
                throw std::runtime_error("Invalid scheme binary!");
            }

            os.write((char*) &ring_size_, sizeof(ring_size_));

            os.write((char*) &coeff_modulus_count_,
                     sizeof(coeff_modulus_count_));

            os.write((char*) &cipher_size_, sizeof(cipher_size_));

            os.write((char*) &in_ntt_domain_, sizeof(in_ntt_domain_));

            os.write((char*) &storage_type_, sizeof(storage_type_));

            os.write((char*) &relinearization_required_,
                     sizeof(relinearization_required_));

            os.write((char*) &ciphertext_generated_,
                     sizeof(ciphertext_generated_));

            if (storage_type_ == storage_type::DEVICE)
            {
                uint32_t ciphertext_memory_size =
                    cipher_size_ * coeff_modulus_count_ * ring_size_;
                HostVector<Data64> host_locations_temp(ciphertext_memory_size);
                hipMemcpy(host_locations_temp.data(), device_locations_.data(),
                           ciphertext_memory_size * sizeof(Data64),
                           hipMemcpyDeviceToHost);
                HEONGPU_CUDA_CHECK(hipGetLastError());
                hipDeviceSynchronize();

                os.write((char*) &ciphertext_memory_size,
                         sizeof(ciphertext_memory_size));
                os.write((char*) host_locations_temp.data(),
                         sizeof(Data64) * ciphertext_memory_size);
            }
            else
            {
                uint32_t ciphertext_memory_size = host_locations_.size();
                os.write((char*) &ciphertext_memory_size,
                         sizeof(ciphertext_memory_size));
                os.write((char*) host_locations_.data(),
                         sizeof(Data64) * ciphertext_memory_size);
            }
        }
        else
        {
            throw std::runtime_error(
                "Ciphertext is not generated so can not be serialized!");
        }
    }

    void Ciphertext<Scheme::BFV>::load(std::istream& is)
    {
        if ((!ciphertext_generated_))
        {
            is.read((char*) &scheme_, sizeof(scheme_));

            is.read((char*) &ring_size_, sizeof(ring_size_));

            is.read((char*) &coeff_modulus_count_,
                    sizeof(coeff_modulus_count_));

            is.read((char*) &cipher_size_, sizeof(cipher_size_));

            is.read((char*) &in_ntt_domain_, sizeof(in_ntt_domain_));

            is.read((char*) &storage_type_, sizeof(storage_type_));

            is.read((char*) &relinearization_required_,
                    sizeof(relinearization_required_));

            is.read((char*) &ciphertext_generated_,
                    sizeof(ciphertext_generated_));

            storage_type_ = storage_type::DEVICE;
            ciphertext_generated_ = true;

            uint32_t ciphertext_memory_size;
            is.read((char*) &ciphertext_memory_size,
                    sizeof(ciphertext_memory_size));

            if (ciphertext_memory_size !=
                (cipher_size_ * ring_size_ * coeff_modulus_count_))
            {
                throw std::runtime_error("Invalid ciphertext size!");
            }

            HostVector<Data64> host_locations_temp(ciphertext_memory_size);
            is.read((char*) host_locations_temp.data(),
                    sizeof(Data64) * ciphertext_memory_size);

            device_locations_.resize(ciphertext_memory_size);
            hipMemcpy(device_locations_.data(), host_locations_temp.data(),
                       ciphertext_memory_size * sizeof(Data64),
                       hipMemcpyHostToDevice);
            HEONGPU_CUDA_CHECK(hipGetLastError());
            hipDeviceSynchronize();
        }
        else
        {
            throw std::runtime_error("Ciphertext has been already exist!");
        }
    }

    int Ciphertext<Scheme::BFV>::memory_size()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_locations_.size();
        }
        else
        {
            return host_locations_.size();
        }
    }

    void Ciphertext<Scheme::BFV>::memory_clear(hipStream_t stream)
    {
        if (device_locations_.size() > 0)
        {
            device_locations_.resize(0, stream);
            device_locations_.shrink_to_fit(stream);
        }

        if (host_locations_.size() > 0)
        {
            host_locations_.resize(0);
            host_locations_.shrink_to_fit();
        }
    }

    void Ciphertext<Scheme::BFV>::memory_set(
        DeviceVector<Data64>&& new_device_vector)
    {
        storage_type_ = storage_type::DEVICE;
        device_locations_ = std::move(new_device_vector);

        if (host_locations_.size() > 0)
        {
            host_locations_.resize(0);
            host_locations_.shrink_to_fit();
        }
    }

    void Ciphertext<Scheme::BFV>::copy_to_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            if (memory_size() == 0)
            {
                // pass
            }
            else
            {
                device_locations_ =
                    DeviceVector<Data64>(host_locations_, stream);
            }

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Ciphertext<Scheme::BFV>::remove_from_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            device_locations_.resize(0, stream);
            device_locations_.shrink_to_fit(stream);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    void Ciphertext<Scheme::BFV>::remove_from_host()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            host_locations_.resize(0);
            host_locations_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

} // namespace heongpu