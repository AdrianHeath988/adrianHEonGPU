#include "hip/hip_runtime.h"
// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bfv/evaluationkey.cuh"

namespace heongpu
{
    __host__ Relinkey<Scheme::BFV>::Relinkey(HEContext<Scheme::BFV>& context,
                                             bool store_in_gpu)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        key_type = context.keyswitching_type_;

        ring_size = context.n;
        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;

        storage_type_ =
            store_in_gpu ? storage_type::DEVICE : storage_type::HOST;

        switch (static_cast<int>(context.keyswitching_type_))
        {
            case 1: // KEYSWITCHING_METHOD_I
            {
                relinkey_size_ = 2 * Q_size_ * Q_prime_size_ * ring_size;
            }
            break;
            case 2: // KEYSWITCHING_METHOD_II
            {
                d_ = context.d;
                relinkey_size_ = 2 * d_ * Q_prime_size_ * ring_size;
            }
            break;
            case 3: // KEYSWITCHING_METHOD_III
            {
                d_ = context.d;
                d_tilda_ = context.d_tilda;
                r_prime_ = context.r_prime;
                relinkey_size_ = 2 * d_ * d_tilda_ * r_prime_ * ring_size;
            }
            break;
            default:
                break;
        }
    }

    void Relinkey<Scheme::BFV>::store_in_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            device_location_ = DeviceVector<Data64>(host_location_, stream);
            host_location_.resize(0);
            host_location_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Relinkey<Scheme::BFV>::store_in_host(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            host_location_ = HostVector<Data64>(relinkey_size_);
            hipMemcpyAsync(host_location_.data(), device_location_.data(),
                            relinkey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, stream);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            device_location_.resize(0, stream);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    Data64* Relinkey<Scheme::BFV>::data()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_location_.data();
        }
        else
        {
            return host_location_.data();
        }
    }

    void Relinkey<Scheme::BFV>::save(std::ostream& os) const
    {
        if (relin_key_generated_)
        {
            os.write((char*) &scheme_, sizeof(scheme_));

            os.write((char*) &key_type, sizeof(key_type));

            os.write((char*) &ring_size, sizeof(ring_size));

            os.write((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            os.write((char*) &Q_size_, sizeof(Q_size_));

            os.write((char*) &d_, sizeof(d_));

            os.write((char*) &d_tilda_, sizeof(d_tilda_));

            os.write((char*) &r_prime_, sizeof(r_prime_));

            os.write((char*) &storage_type_, sizeof(storage_type_));

            os.write((char*) &relin_key_generated_,
                     sizeof(relin_key_generated_));

            os.write((char*) &relinkey_size_, sizeof(relinkey_size_));

            if (storage_type_ == storage_type::DEVICE)
            {
                HostVector<Data64> host_locations_temp(relinkey_size_);
                hipMemcpy(host_locations_temp.data(), device_location_.data(),
                           relinkey_size_ * sizeof(Data64),
                           hipMemcpyDeviceToHost);
                HEONGPU_CUDA_CHECK(hipGetLastError());
                hipDeviceSynchronize();

                os.write((char*) host_locations_temp.data(),
                         sizeof(Data64) * relinkey_size_);
            }
            else
            {
                os.write((char*) host_location_.data(),
                         sizeof(Data64) * relinkey_size_);
            }
        }
        else
        {
            throw std::runtime_error(
                "Relinkey is not generated so can not be serialized!");
        }
    }

    void Relinkey<Scheme::BFV>::load(std::istream& is)
    {
        if ((!relin_key_generated_))
        {
            is.read((char*) &scheme_, sizeof(scheme_));

            if (scheme_ != scheme_type::bfv)
            {
                throw std::runtime_error("Invalid scheme binary!");
            }

            is.read((char*) &key_type, sizeof(key_type));

            is.read((char*) &ring_size, sizeof(ring_size));

            is.read((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            is.read((char*) &Q_size_, sizeof(Q_size_));

            is.read((char*) &d_, sizeof(d_));

            is.read((char*) &d_tilda_, sizeof(d_tilda_));

            is.read((char*) &r_prime_, sizeof(r_prime_));

            is.read((char*) &storage_type_, sizeof(storage_type_));

            is.read((char*) &relin_key_generated_,
                    sizeof(relin_key_generated_));

            is.read((char*) &relinkey_size_, sizeof(relinkey_size_));

            storage_type_ = storage_type::DEVICE;
            relin_key_generated_ = true;

            HostVector<Data64> host_locations_temp(relinkey_size_);
            is.read((char*) host_locations_temp.data(),
                    sizeof(Data64) * relinkey_size_);

            device_location_.resize(relinkey_size_);
            hipMemcpy(device_location_.data(), host_locations_temp.data(),
                       relinkey_size_ * sizeof(Data64), hipMemcpyHostToDevice);
            HEONGPU_CUDA_CHECK(hipGetLastError());
            hipDeviceSynchronize();
        }
        else
        {
            throw std::runtime_error("Relinkey has been already exist!");
        }
    }

    int Relinkey<Scheme::BFV>::memory_size()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_location_.size();
        }
        else
        {
            return host_location_.size();
        }
    }

    void Relinkey<Scheme::BFV>::memory_clear(hipStream_t stream)
    {
        if (device_location_.size() > 0)
        {
            device_location_.resize(0, stream);
            device_location_.shrink_to_fit(stream);
        }

        if (host_location_.size() > 0)
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();
        }
    }

    void
    Relinkey<Scheme::BFV>::memory_set(DeviceVector<Data64>&& new_device_vector)
    {
        storage_type_ = storage_type::DEVICE;
        device_location_ = std::move(new_device_vector);

        if (host_location_.size() > 0)
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();
        }
    }

    void Relinkey<Scheme::BFV>::copy_to_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            if (memory_size() == 0)
            {
                // pass
            }
            else
            {
                device_location_ = DeviceVector<Data64>(host_location_, stream);
            }

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Relinkey<Scheme::BFV>::remove_from_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            device_location_.resize(0, stream);
            device_location_.shrink_to_fit(stream);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    void Relinkey<Scheme::BFV>::remove_from_host()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

    __host__ MultipartyRelinkey<Scheme::BFV>::MultipartyRelinkey(
        HEContext<Scheme::BFV>& context, const RNGSeed seed, bool store_in_gpu)
        : Relinkey(context, store_in_gpu), seed_(seed)
    {
    }

    __host__ Galoiskey<Scheme::BFV>::Galoiskey(HEContext<Scheme::BFV>& context,
                                               bool store_in_gpu)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        key_type = context.keyswitching_type_;

        ring_size = context.n;
        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;

        storage_type_ =
            store_in_gpu ? storage_type::DEVICE : storage_type::HOST;

        customized = false;

        group_order_ = 3;

        switch (static_cast<int>(context.keyswitching_type_))
        {
            case 1: // KEYSWITCHING_METHOD_I
            {
                galoiskey_size_ = 2 * Q_size_ * Q_prime_size_ * ring_size;

                for (int i = 0; i < MAX_SHIFT; i++)
                {
                    int power = pow(2, i);
                    galois_elt[power] =
                        steps_to_galois_elt(power, ring_size, group_order_);
                    galois_elt[(-power)] =
                        steps_to_galois_elt((-power), ring_size, group_order_);
                }

                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);
            }
            break;
            case 2: // KEYSWITCHING_METHOD_II
            {
                for (int i = 0; i < MAX_SHIFT; i++)
                {
                    int power = pow(2, i);
                    galois_elt[power] =
                        steps_to_galois_elt(power, ring_size, group_order_);
                    galois_elt[(-power)] =
                        steps_to_galois_elt((-power), ring_size, group_order_);
                }

                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);

                d_ = context.d;
                galoiskey_size_ = 2 * d_ * Q_prime_size_ * ring_size;
            }
            break;
            case 3: // KEYSWITCHING_METHOD_III
                throw std::invalid_argument(
                    "Galoiskey does not support KEYSWITCHING_METHOD_III");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }
    }

    __host__ Galoiskey<Scheme::BFV>::Galoiskey(HEContext<Scheme::BFV>& context,
                                               std::vector<int>& shift_vec,
                                               bool store_in_gpu)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        key_type = context.keyswitching_type_;

        ring_size = context.n;
        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;

        storage_type_ =
            store_in_gpu ? storage_type::DEVICE : storage_type::HOST;

        customized = false;

        group_order_ = 3;

        switch (static_cast<int>(context.keyswitching_type_))
        {
            case 1: // KEYSWITCHING_METHOD_I
            {
                galoiskey_size_ = 2 * Q_size_ * Q_prime_size_ * ring_size;

                for (int shift : shift_vec)
                {
                    galois_elt[shift] =
                        steps_to_galois_elt(shift, ring_size, group_order_);
                }

                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);
            }
            break;
            case 2: // KEYSWITCHING_METHOD_II
            {
                for (int shift : shift_vec)
                {
                    galois_elt[shift] =
                        steps_to_galois_elt(shift, ring_size, group_order_);
                }

                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);

                d_ = context.d;
                galoiskey_size_ = 2 * d_ * Q_prime_size_ * ring_size;
            }
            break;
            case 3: // KEYSWITCHING_METHOD_III Galoiskey
                throw std::invalid_argument(
                    "Galoiskey does not support KEYSWITCHING_METHOD_III");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }
    }

    __host__
    Galoiskey<Scheme::BFV>::Galoiskey(HEContext<Scheme::BFV>& context,
                                      std::vector<uint32_t>& galois_elts,
                                      bool store_in_gpu)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        key_type = context.keyswitching_type_;

        ring_size = context.n;
        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;

        storage_type_ =
            store_in_gpu ? storage_type::DEVICE : storage_type::HOST;

        customized = true;

        group_order_ = 3;

        switch (static_cast<int>(context.keyswitching_type_))
        {
            case 1: // KEYSWITCHING_METHOD_I
            {
                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);
                galoiskey_size_ = 2 * Q_size_ * Q_prime_size_ * ring_size;
                custom_galois_elt = galois_elts;
            }
            break;
            case 2: // KEYSWITCHING_METHOD_II
            {
                d_ = context.d;
                galois_elt_zero =
                    steps_to_galois_elt(0, ring_size, group_order_);
                galoiskey_size_ = 2 * d_ * Q_prime_size_ * ring_size;
                custom_galois_elt = galois_elts;
            }
            break;
            case 3: // KEYSWITCHING_METHOD_III Galoiskey
                throw std::invalid_argument(
                    "Galoiskey does not support KEYSWITCHING_METHOD_III");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }
    }

    void Galoiskey<Scheme::BFV>::store_in_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            for (const auto& galois_ : host_location_)
            {
                device_location_[galois_.first] =
                    DeviceVector<Data64>(galois_.second, stream);
            }

            zero_device_location_ =
                DeviceVector<Data64>(zero_host_location_, stream);

            host_location_.clear();
            zero_host_location_.resize(0);
            zero_host_location_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Galoiskey<Scheme::BFV>::store_in_host(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            for (auto& galois_ : device_location_)
            {
                host_location_[galois_.first] =
                    HostVector<Data64>(galoiskey_size_);
                hipMemcpyAsync(host_location_[galois_.first].data(),
                                galois_.second.data(),
                                galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, stream);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                galois_.second.resize(0, stream);
            }

            zero_host_location_ = HostVector<Data64>(galoiskey_size_);
            hipMemcpyAsync(zero_host_location_.data(),
                            zero_device_location_.data(),
                            galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, stream);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            device_location_.clear();
            zero_device_location_.resize(0);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    Data64* Galoiskey<Scheme::BFV>::data(size_t i)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_location_[i].data();
        }
        else
        {
            return host_location_[i].data();
        }
    }

    Data64* Galoiskey<Scheme::BFV>::c_data()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return zero_device_location_.data();
        }
        else
        {
            return zero_host_location_.data();
        }
    }

    void Galoiskey<Scheme::BFV>::save(std::ostream& os) const
    {
        if (galois_key_generated_)
        {
            os.write((char*) &scheme_, sizeof(scheme_));

            os.write((char*) &key_type, sizeof(key_type));

            os.write((char*) &ring_size, sizeof(ring_size));

            os.write((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            os.write((char*) &Q_size_, sizeof(Q_size_));

            os.write((char*) &d_, sizeof(d_));

            os.write((char*) &customized, sizeof(customized));

            os.write((char*) &group_order_, sizeof(group_order_));

            os.write((char*) &storage_type_, sizeof(storage_type_));

            os.write((char*) &galois_key_generated_,
                     sizeof(galois_key_generated_));

            if (customized)
            {
                uint32_t custom_galois_elt_size = custom_galois_elt.size();
                os.write((char*) &custom_galois_elt_size,
                         sizeof(custom_galois_elt_size));
                os.write((char*) custom_galois_elt.data(),
                         sizeof(u_int32_t) * custom_galois_elt_size);
            }
            else
            {
                uint32_t galois_elt_size = galois_elt.size();
                os.write((char*) &galois_elt_size, sizeof(galois_elt_size));
                for (auto& galois : galois_elt)
                {
                    os.write((char*) &galois.first, sizeof(galois.first));
                    os.write((char*) &galois.second, sizeof(galois.second));
                }
            }

            os.write((char*) &galois_elt_zero, sizeof(galois_elt_zero));

            os.write((char*) &galoiskey_size_, sizeof(galoiskey_size_));

            if (storage_type_ == storage_type::DEVICE)
            {
                uint32_t key_count = device_location_.size();
                os.write((char*) &key_count, sizeof(key_count));

                for (auto& galois_key_mem : device_location_)
                {
                    HostVector<Data64> host_locations_temp(galoiskey_size_);
                    hipMemcpy(host_locations_temp.data(),
                               galois_key_mem.second.data(),
                               galoiskey_size_ * sizeof(Data64),
                               hipMemcpyDeviceToHost);
                    HEONGPU_CUDA_CHECK(hipGetLastError());
                    hipDeviceSynchronize();

                    os.write((char*) &galois_key_mem.first,
                             sizeof(galois_key_mem.first));
                    os.write((char*) host_locations_temp.data(),
                             sizeof(Data64) * galoiskey_size_);
                }

                HostVector<Data64> host_locations_temp(galoiskey_size_);
                hipMemcpy(
                    host_locations_temp.data(), zero_device_location_.data(),
                    galoiskey_size_ * sizeof(Data64), hipMemcpyDeviceToHost);
                HEONGPU_CUDA_CHECK(hipGetLastError());
                hipDeviceSynchronize();

                os.write((char*) host_locations_temp.data(),
                         sizeof(Data64) * galoiskey_size_);
            }
            else
            {
                uint32_t key_count = host_location_.size();
                os.write((char*) &key_count, sizeof(key_count));

                for (auto& galois_key_mem : host_location_)
                {
                    os.write((char*) &galois_key_mem.first,
                             sizeof(galois_key_mem.first));
                    os.write((char*) galois_key_mem.second.data(),
                             sizeof(Data64) * galoiskey_size_);
                }

                os.write((char*) zero_host_location_.data(),
                         sizeof(Data64) * galoiskey_size_);
            }
        }
        else
        {
            throw std::runtime_error(
                "Galoiskey is not generated so can not be serialized!");
        }
    }

    void Galoiskey<Scheme::BFV>::load(std::istream& is)
    {
        if ((!galois_key_generated_))
        {
            is.read((char*) &scheme_, sizeof(scheme_));

            if (scheme_ != scheme_type::bfv)
            {
                throw std::runtime_error("Invalid scheme binary!");
            }

            is.read((char*) &key_type, sizeof(key_type));

            is.read((char*) &ring_size, sizeof(ring_size));

            is.read((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            is.read((char*) &Q_size_, sizeof(Q_size_));

            is.read((char*) &d_, sizeof(d_));

            is.read((char*) &customized, sizeof(customized));

            is.read((char*) &group_order_, sizeof(group_order_));

            is.read((char*) &storage_type_, sizeof(storage_type_));

            is.read((char*) &galois_key_generated_,
                    sizeof(galois_key_generated_));

            storage_type_ = storage_type::DEVICE;
            galois_key_generated_ = true;

            if (customized)
            {
                uint32_t custom_galois_elt_size;
                is.read((char*) &custom_galois_elt_size,
                        sizeof(custom_galois_elt_size));
                custom_galois_elt.resize(custom_galois_elt_size);
                is.read((char*) custom_galois_elt.data(),
                        sizeof(u_int32_t) * custom_galois_elt_size);
            }
            else
            {
                uint32_t galois_elt_size;
                is.read((char*) &galois_elt_size, sizeof(galois_elt_size));
                for (int i = 0; i < galois_elt_size; i++)
                {
                    int first;
                    int second;
                    is.read((char*) &first, sizeof(first));
                    is.read((char*) &second, sizeof(second));
                    galois_elt[first] = second;
                }
            }

            is.read((char*) &galois_elt_zero, sizeof(galois_elt_zero));

            is.read((char*) &galoiskey_size_, sizeof(galoiskey_size_));

            uint32_t key_count;
            is.read((char*) &key_count, sizeof(key_count));

            for (int i = 0; i < key_count; i++)
            {
                int first;
                is.read((char*) &first, sizeof(first));
                HostVector<Data64> host_locations_temp(galoiskey_size_);
                is.read((char*) host_locations_temp.data(),
                        sizeof(Data64) * galoiskey_size_);
                device_location_[first] =
                    DeviceVector<Data64>(host_locations_temp);
                hipDeviceSynchronize();
            }

            HostVector<Data64> host_locations_temp(galoiskey_size_);
            is.read((char*) host_locations_temp.data(),
                    sizeof(Data64) * galoiskey_size_);

            zero_device_location_.resize(galoiskey_size_);
            hipMemcpy(zero_device_location_.data(), host_locations_temp.data(),
                       galoiskey_size_ * sizeof(Data64),
                       hipMemcpyHostToDevice);
            HEONGPU_CUDA_CHECK(hipGetLastError());
            hipDeviceSynchronize();
        }
        else
        {
            throw std::runtime_error("Galoiskey has been already exist!");
        }
    }

    __host__ MultipartyGaloiskey<Scheme::BFV>::MultipartyGaloiskey(
        HEContext<Scheme::BFV>& context, const RNGSeed seed, bool store_in_gpu)
        : Galoiskey(context, store_in_gpu), seed_(seed)
    {
    }

    __host__ MultipartyGaloiskey<Scheme::BFV>::MultipartyGaloiskey(
        HEContext<Scheme::BFV>& context, std::vector<int>& shift_vec,
        const RNGSeed seed, bool store_in_gpu)
        : Galoiskey(context, shift_vec, store_in_gpu), seed_(seed)
    {
    }

    __host__ MultipartyGaloiskey<Scheme::BFV>::MultipartyGaloiskey(
        HEContext<Scheme::BFV>& context, std::vector<uint32_t>& galois_elts,
        const RNGSeed seed, bool store_in_gpu)
        : Galoiskey(context, galois_elts, store_in_gpu), seed_(seed)
    {
    }

    __host__ Switchkey<Scheme::BFV>::Switchkey(HEContext<Scheme::BFV>& context,
                                               bool store_in_gpu)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;
        key_type = context.keyswitching_type_;

        ring_size = context.n;
        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;

        storage_type_ =
            store_in_gpu ? storage_type::DEVICE : storage_type::HOST;

        switch (static_cast<int>(context.keyswitching_type_))
        {
            case 1: // KEYSWITCHING_METHOD_I
                switchkey_size_ = 2 * Q_size_ * Q_prime_size_ * ring_size;
                break;
            case 2: // KEYSWITCHING_METHOD_II
            {
                d_ = context.d;
                switchkey_size_ = 2 * d_ * Q_prime_size_ * ring_size;
            }
            break;
            case 3: // KEYSWITCHING_METHOD_III
                throw std::invalid_argument(
                    "Switchkey does not support KEYSWITCHING_METHOD_III");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }
    }

    void Switchkey<Scheme::BFV>::store_in_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            device_location_ = DeviceVector<Data64>(host_location_, stream);
            host_location_.resize(0);
            host_location_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Switchkey<Scheme::BFV>::store_in_host(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            host_location_ = HostVector<Data64>(switchkey_size_);
            hipMemcpyAsync(host_location_.data(), device_location_.data(),
                            switchkey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, stream);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            device_location_.resize(0, stream);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    Data64* Switchkey<Scheme::BFV>::data()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_location_.data();
        }
        else
        {
            return host_location_.data();
        }
    }

    void Switchkey<Scheme::BFV>::save(std::ostream& os) const
    {
        if (switch_key_generated_)
        {
            os.write((char*) &scheme_, sizeof(scheme_));

            os.write((char*) &key_type, sizeof(key_type));

            os.write((char*) &ring_size, sizeof(ring_size));

            os.write((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            os.write((char*) &Q_size_, sizeof(Q_size_));

            os.write((char*) &d_, sizeof(d_));

            os.write((char*) &storage_type_, sizeof(storage_type_));

            os.write((char*) &switch_key_generated_,
                     sizeof(switch_key_generated_));

            os.write((char*) &switchkey_size_, sizeof(switchkey_size_));

            if (storage_type_ == storage_type::DEVICE)
            {
                HostVector<Data64> host_locations_temp(switchkey_size_);
                hipMemcpy(host_locations_temp.data(), device_location_.data(),
                           switchkey_size_ * sizeof(Data64),
                           hipMemcpyDeviceToHost);
                HEONGPU_CUDA_CHECK(hipGetLastError());
                hipDeviceSynchronize();

                os.write((char*) host_locations_temp.data(),
                         sizeof(Data64) * switchkey_size_);
            }
            else
            {
                os.write((char*) host_location_.data(),
                         sizeof(Data64) * switchkey_size_);
            }
        }
        else
        {
            throw std::runtime_error(
                "Switchkey is not generated so can not be serialized!");
        }
    }

    void Switchkey<Scheme::BFV>::load(std::istream& is)
    {
        if ((!switch_key_generated_))
        {
            is.read((char*) &scheme_, sizeof(scheme_));

            is.read((char*) &key_type, sizeof(key_type));

            is.read((char*) &ring_size, sizeof(ring_size));

            is.read((char*) &Q_prime_size_, sizeof(Q_prime_size_));

            is.read((char*) &Q_size_, sizeof(Q_size_));

            is.read((char*) &d_, sizeof(d_));

            is.read((char*) &storage_type_, sizeof(storage_type_));

            is.read((char*) &switch_key_generated_,
                    sizeof(switch_key_generated_));

            is.read((char*) &switchkey_size_, sizeof(switchkey_size_));

            storage_type_ = storage_type::DEVICE;
            switch_key_generated_ = true;

            HostVector<Data64> host_locations_temp(switchkey_size_);
            is.read((char*) host_locations_temp.data(),
                    sizeof(Data64) * switchkey_size_);

            hipMemcpy(device_location_.data(), host_locations_temp.data(),
                       switchkey_size_ * sizeof(Data64),
                       hipMemcpyHostToDevice);
            HEONGPU_CUDA_CHECK(hipGetLastError());
            hipDeviceSynchronize();
        }
        else
        {
            throw std::runtime_error("Switchkey has been already exist!");
        }
    }

    int Switchkey<Scheme::BFV>::memory_size()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            return device_location_.size();
        }
        else
        {
            return host_location_.size();
        }
    }

    void Switchkey<Scheme::BFV>::memory_clear(hipStream_t stream)
    {
        if (device_location_.size() > 0)
        {
            device_location_.resize(0, stream);
            device_location_.shrink_to_fit(stream);
        }

        if (host_location_.size() > 0)
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();
        }
    }

    void
    Switchkey<Scheme::BFV>::memory_set(DeviceVector<Data64>&& new_device_vector)
    {
        storage_type_ = storage_type::DEVICE;
        device_location_ = std::move(new_device_vector);

        if (host_location_.size() > 0)
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();
        }
    }

    void Switchkey<Scheme::BFV>::copy_to_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            if (memory_size() == 0)
            {
                // pass
            }
            else
            {
                device_location_ = DeviceVector<Data64>(host_location_, stream);
            }

            storage_type_ = storage_type::DEVICE;
        }
    }

    void Switchkey<Scheme::BFV>::remove_from_device(hipStream_t stream)
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            device_location_.resize(0, stream);
            device_location_.shrink_to_fit(stream);

            storage_type_ = storage_type::HOST;
        }
        else
        {
            // pass
        }
    }

    void Switchkey<Scheme::BFV>::remove_from_host()
    {
        if (storage_type_ == storage_type::DEVICE)
        {
            // pass
        }
        else
        {
            host_location_.resize(0);
            host_location_.shrink_to_fit();

            storage_type_ = storage_type::DEVICE;
        }
    }

} // namespace heongpu