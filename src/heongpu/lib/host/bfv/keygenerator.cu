﻿// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bfv/keygenerator.cuh"

namespace heongpu
{
    __host__
    HEKeyGenerator<Scheme::BFV>::HEKeyGenerator(HEContext<Scheme::BFV>& context)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme = context.scheme_;

        std::random_device rd;
        std::mt19937 gen(rd());
        seed_ = gen();
        offset_ = gen();

        new_seed_ = RNGSeed();

        n = context.n;
        n_power = context.n_power;

        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;
        P_size_ = context.P_size;

        modulus_ = context.modulus_;
        ntt_table_ = context.ntt_table_;
        intt_table_ = context.intt_table_;
        n_inverse_ = context.n_inverse_;
        factor_ = context.factor_;

        d_ = context.d;
        d_tilda_ = context.d_tilda;
        r_prime_ = context.r_prime;

        B_prime_ = context.B_prime_;
        B_prime_ntt_tables_ = context.B_prime_ntt_tables_;
        B_prime_intt_tables_ = context.B_prime_intt_tables_;
        B_prime_n_inverse_ = context.B_prime_n_inverse_;

        base_change_matrix_D_to_B_ = context.base_change_matrix_D_to_B_;
        base_change_matrix_B_to_D_ = context.base_change_matrix_B_to_D_;
        Mi_inv_D_to_B_ = context.Mi_inv_D_to_B_;
        Mi_inv_B_to_D_ = context.Mi_inv_B_to_D_;
        prod_D_to_B_ = context.prod_D_to_B_;
        prod_B_to_D_ = context.prod_B_to_D_;

        I_j_ = context.I_j_;
        I_location_ = context.I_location_;
        Sk_pair_ = context.Sk_pair_;
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_secret_key(
        Secretkey<Scheme::BFV>& sk, const ExecutionOptions& options)
    {
        if (sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<int> secret_key_without_rns((n), options.stream_);

                secretkey_gen_kernel<<<dim3((n >> 8), 1, 1), 256, 0,
                                       options.stream_>>>(
                    secret_key_without_rns.data(), sk_.hamming_weight_, n_power,
                    seed_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                DeviceVector<Data64> secret_key_rns(
                    (sk_.coeff_modulus_count() * n), options.stream_);

                secretkey_rns_kernel<<<dim3((n >> 8), 1, 1), 256, 0,
                                       options.stream_>>>(
                    secret_key_without_rns.data(), secret_key_rns.data(),
                    modulus_->data(), n_power, Q_prime_size_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                    .n_power = n_power,
                    .ntt_type = gpuntt::FORWARD,
                    .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                    .zero_padding = false,
                    .stream = options.stream_};

                gpuntt::GPU_NTT_Inplace(secret_key_rns.data(),
                                        ntt_table_->data(), modulus_->data(),
                                        cfg_ntt, Q_prime_size_, Q_prime_size_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                sk_.in_ntt_domain_ = true;
                sk_.secret_key_generated_ = true;

                sk_.memory_set(std::move(secret_key_rns));
            },
            options, true);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_public_key(
        Publickey<Scheme::BFV>& pk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (pk.public_key_generated_)
        {
            throw std::logic_error("Publickey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    pk,
                    [&](Publickey<Scheme::BFV>& pk_)
                    {
                        DeviceVector<Data64> output_memory(
                            (2 * Q_prime_size_ * n), options.stream_);

                        DeviceVector<Data64> errors_a(2 * Q_prime_size_ * n,
                                                      options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, 1, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, 1, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(errors_a.data(),
                                                ntt_table_->data(),
                                                modulus_->data(), cfg_ntt,
                                                Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        publickey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk_.data(), error_poly,
                            a_poly, modulus_->data(), n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        pk_.memory_set(std::move(output_memory));

                        pk_.in_ntt_domain_ = true;
                        pk_.public_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_multi_party_public_key_piece(
        MultipartyPublickey<Scheme::BFV>& pk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (pk.public_key_generated_)
        {
            throw std::logic_error("Publickey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    pk,
                    [&](MultipartyPublickey<Scheme::BFV>& pk_)
                    {
                        DeviceVector<Data64> output_memory(
                            (2 * Q_prime_size_ * n), options.stream_);

                        RNGSeed common_seed = pk_.seed();

                        DeviceVector<Data64> errors_a(2 * Q_prime_size_ * n,
                                                      options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * n);

                        RandomNumberGenerator::instance().set(
                            common_seed.key_, common_seed.nonce_,
                            common_seed.personalization_string_,
                            options.stream_);
                        RandomNumberGenerator::instance()
                            .modular_ternary_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, 1, options.stream_);

                        RNGSeed gen_seed;
                        RandomNumberGenerator::instance().set(
                            gen_seed.key_, gen_seed.nonce_,
                            gen_seed.personalization_string_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, 1, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(errors_a.data(),
                                                ntt_table_->data(),
                                                modulus_->data(), cfg_ntt,
                                                Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        publickey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk_.data(), error_poly,
                            a_poly, modulus_->data(), n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        pk_.memory_set(std::move(output_memory));

                        pk_.in_ntt_domain_ = true;
                        pk_.public_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_multi_party_public_key(
        std::vector<MultipartyPublickey<Scheme::BFV>>& all_pk,
        Publickey<Scheme::BFV>& pk, const ExecutionOptions& options)
    {
        int participant_count = all_pk.size();

        if (participant_count == 0)
        {
            throw std::invalid_argument(
                "No participant to generate common publickey!");
        }

        for (int i = 0; i < participant_count; i++)
        {
            if (!all_pk[i].public_key_generated_)
            {
                throw std::invalid_argument(
                    "MultipartyPublickey is not generated!");
            }
        }

        input_vector_storage_manager(
            all_pk,
            [&](std::vector<MultipartyPublickey<Scheme::BFV>>& all_pk_)
            {
                output_storage_manager(
                    pk,
                    [&](Publickey<Scheme::BFV>& pk_)
                    {
                        DeviceVector<Data64> output_memory(
                            (2 * Q_prime_size_ * n), options.stream_);

                        global_memory_replace_kernel<<<
                            dim3((n >> 8), Q_prime_size_, 2), 256, 0,
                            options.stream_>>>(all_pk[0].data(),
                                               output_memory.data(), n_power);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        for (int i = 1; i < participant_count; i++)
                        {
                            threshold_pk_addition<<<dim3((n >> 8),
                                                         Q_prime_size_, 1),
                                                    256, 0, options.stream_>>>(
                                all_pk[i].data(), output_memory.data(),
                                output_memory.data(), modulus_->data(), n_power,
                                false);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }

                        pk_.memory_set(std::move(output_memory));

                        pk_.in_ntt_domain_ = true;
                        pk_.public_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_relin_key_method_I(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * Q_size_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly =
                            error_poly + (Q_prime_size_ * Q_size_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                            output_memory.data(), sk_.data(), error_poly,
                            a_poly, modulus_->data(), factor_->data(), n_power,
                            Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::
        generate_multi_party_relin_key_piece_method_I_stage_I(
            MultipartyRelinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
            const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_)
                    {
                        RNGSeed common_seed = rk.seed();

                        DeviceVector<Data64> random_values(
                            Q_prime_size_ * ((3 * Q_size_) + 1) * n,
                            options.stream_);
                        Data64* e0 = random_values.data();
                        Data64* e1 = e0 + (Q_prime_size_ * Q_size_ * n);
                        Data64* u = e1 + (Q_prime_size_ * Q_size_ * n);
                        Data64* common_a = u + (Q_prime_size_ * n);

                        RandomNumberGenerator::instance().set(
                            common_seed.key_, common_seed.nonce_,
                            common_seed.personalization_string_,
                            options.stream_);
                        RandomNumberGenerator::instance()
                            .modular_ternary_random_number_generation(
                                common_a, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RNGSeed gen_seed1;
                        RandomNumberGenerator::instance().set(
                            gen_seed1.key_, gen_seed1.nonce_,
                            gen_seed1.personalization_string_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, e0, modulus_->data(), n_power,
                                Q_prime_size_, 2 * Q_size_, options.stream_);

                        RandomNumberGenerator::instance().set(
                            new_seed_.key_, new_seed_.nonce_,
                            new_seed_.personalization_string_, options.stream_);
                        RandomNumberGenerator::instance()
                            .modular_ternary_random_number_generation(
                                u, modulus_->data(), n_power, Q_prime_size_, 1,
                                options.stream_);

                        RNGSeed gen_seed2;
                        RandomNumberGenerator::instance().set(
                            gen_seed2.key_, gen_seed2.nonce_,
                            gen_seed2.personalization_string_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            random_values.data(), ntt_table_->data(),
                            modulus_->data(), cfg_ntt,
                            Q_prime_size_ * ((2 * Q_size_) + 1), Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        multi_party_relinkey_piece_method_I_stage_I_kernel<<<
                            dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                            options.stream_>>>(
                            output_memory.data(), sk.data(), common_a, u, e0,
                            e1, modulus_->data(), factor_->data(), n_power,
                            Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::
        generate_multi_party_relin_key_piece_method_I_stage_II(
            MultipartyRelinkey<Scheme::BFV>& rk_stage_1,
            MultipartyRelinkey<Scheme::BFV>& rk_stage_2,
            Secretkey<Scheme::BFV>& sk, const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!rk_stage_1.relin_key_generated_)
        {
            throw std::logic_error("Relinkey1 is not generated!");
        }

        if (rk_stage_2.relin_key_generated_)
        {
            throw std::logic_error("Relinkey2 is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                input_storage_manager(
                    rk_stage_1,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_stage_1_)
                    {
                        output_storage_manager(
                            rk_stage_2,
                            [&](MultipartyRelinkey<Scheme::BFV>& rk_stage_2_)
                            {
                                DeviceVector<Data64> random_values(
                                    Q_prime_size_ * ((2 * Q_size_) + 1) * n,
                                    options.stream_);
                                Data64* e0 = random_values.data();
                                Data64* e1 = e0 + (Q_prime_size_ * Q_size_ * n);
                                Data64* u = e1 + (Q_prime_size_ * Q_size_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, e0, modulus_->data(),
                                        n_power, Q_prime_size_, 2,
                                        options.stream_);

                                RandomNumberGenerator::instance().set(
                                    new_seed_.key_, new_seed_.nonce_,
                                    new_seed_.personalization_string_,
                                    options.stream_);
                                RandomNumberGenerator::instance()
                                    .modular_ternary_random_number_generation(
                                        u, modulus_->data(), n_power,
                                        Q_prime_size_, 1, options.stream_);

                                RNGSeed gen_seed;
                                RandomNumberGenerator::instance().set(
                                    gen_seed.key_, gen_seed.nonce_,
                                    gen_seed.personalization_string_,
                                    options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    random_values.data(), ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    Q_prime_size_ * ((2 * Q_size_) + 1),
                                    Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    rk_stage_2_.relinkey_size_,
                                    options.stream_);

                                multi_party_relinkey_piece_method_I_II_stage_II_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    rk_stage_1_.data(), output_memory.data(),
                                    sk.data(), u, e0, e1, modulus_->data(),
                                    n_power, Q_prime_size_, Q_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                rk_stage_2_.memory_set(
                                    std::move(output_memory));

                                rk_stage_2_.relin_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::
        generate_bfv_multi_party_relin_key_piece_method_II_stage_I(
            MultipartyRelinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
            const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_)
                    {
                        RNGSeed common_seed = rk.seed();

                        DeviceVector<Data64> random_values(
                            Q_prime_size_ * ((3 * d_) + 1) * n,
                            options.stream_);
                        Data64* e0 = random_values.data();
                        Data64* e1 = e0 + (Q_prime_size_ * d_ * n);
                        Data64* u = e1 + (Q_prime_size_ * d_ * n);
                        Data64* common_a = u + (Q_prime_size_ * n);

                        RandomNumberGenerator::instance().set(
                            common_seed.key_, common_seed.nonce_,
                            common_seed.personalization_string_,
                            options.stream_);
                        RandomNumberGenerator::instance()
                            .modular_ternary_random_number_generation(
                                common_a, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RNGSeed gen_seed1;
                        RandomNumberGenerator::instance().set(
                            gen_seed1.key_, gen_seed1.nonce_,
                            gen_seed1.personalization_string_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, e0, modulus_->data(), n_power,
                                Q_prime_size_, 2 * d_, options.stream_);

                        RandomNumberGenerator::instance().set(
                            new_seed_.key_, new_seed_.nonce_,
                            new_seed_.personalization_string_, options.stream_);
                        RandomNumberGenerator::instance()
                            .modular_ternary_random_number_generation(
                                u, modulus_->data(), n_power, Q_prime_size_, 1,
                                options.stream_);

                        RNGSeed gen_seed2;
                        RandomNumberGenerator::instance().set(
                            gen_seed2.key_, gen_seed2.nonce_,
                            gen_seed2.personalization_string_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            random_values.data(), ntt_table_->data(),
                            modulus_->data(), cfg_ntt,
                            Q_prime_size_ * ((2 * d_) + 1), Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        multi_party_relinkey_piece_method_II_stage_I_kernel<<<
                            dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                            options.stream_>>>(
                            output_memory.data(), sk.data(), common_a, u, e0,
                            e1, modulus_->data(), factor_->data(),
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::
        generate_bfv_multi_party_relin_key_piece_method_II_stage_II(
            MultipartyRelinkey<Scheme::BFV>& rk_stage_1,
            MultipartyRelinkey<Scheme::BFV>& rk_stage_2,
            Secretkey<Scheme::BFV>& sk, const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!rk_stage_1.relin_key_generated_)
        {
            throw std::logic_error("Relinkey1 is not generated!");
        }

        if (rk_stage_2.relin_key_generated_)
        {
            throw std::logic_error("Relinkey2 is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                input_storage_manager(
                    rk_stage_1,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_stage_1_)
                    {
                        output_storage_manager(
                            rk_stage_2,
                            [&](MultipartyRelinkey<Scheme::BFV>& rk_stage_2_)
                            {
                                DeviceVector<Data64> random_values(
                                    Q_prime_size_ * ((2 * d_) + 1) * n,
                                    options.stream_);
                                Data64* e0 = random_values.data();
                                Data64* e1 = e0 + (Q_prime_size_ * d_ * n);
                                Data64* u = e1 + (Q_prime_size_ * d_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, e0, modulus_->data(),
                                        n_power, Q_prime_size_, 2,
                                        options.stream_);

                                RandomNumberGenerator::instance().set(
                                    new_seed_.key_, new_seed_.nonce_,
                                    new_seed_.personalization_string_,
                                    options.stream_);
                                RandomNumberGenerator::instance()
                                    .modular_ternary_random_number_generation(
                                        u, modulus_->data(), n_power,
                                        Q_prime_size_, 1, options.stream_);

                                RNGSeed gen_seed;
                                RandomNumberGenerator::instance().set(
                                    gen_seed.key_, gen_seed.nonce_,
                                    gen_seed.personalization_string_,
                                    options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    random_values.data(), ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    Q_prime_size_ * ((2 * d_) + 1),
                                    Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    rk_stage_2.relinkey_size_, options.stream_);

                                multi_party_relinkey_piece_method_I_II_stage_II_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    rk_stage_1.data(), output_memory.data(),
                                    sk.data(), u, e0, e1, modulus_->data(),
                                    n_power, Q_prime_size_, d_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                rk_stage_2.memory_set(std::move(output_memory));

                                rk_stage_2.relin_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_multi_party_relin_key(
        std::vector<MultipartyRelinkey<Scheme::BFV>>& all_rk,
        MultipartyRelinkey<Scheme::BFV>& rk, const ExecutionOptions& options)
    {
        int participant_count = all_rk.size();

        if (participant_count == 0)
        {
            throw std::invalid_argument(
                "No participant to generate common publickey!");
        }

        for (int i = 0; i < participant_count; i++)
        {
            if (!all_rk[i].relin_key_generated_)
            {
                throw std::invalid_argument(
                    "MultipartyRelinkey is not generated!");
            }
        }

        int dimension;
        switch (static_cast<int>(rk.key_type))
        {
            case 1: // KEYSWITCHING_METHOD_I
                dimension = rk.Q_size_;
                break;
            case 2: // KEYSWITCHING_METHOD_II
                dimension = rk.d_;
                break;
            case 3: // KEYSWITCHING_METHOD_III
                throw std::invalid_argument(
                    "Key Switching Type III is not supported for multi "
                    "party key generation.");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }

        input_vector_storage_manager(
            all_rk,
            [&](std::vector<MultipartyRelinkey<Scheme::BFV>>& all_rk_)
            {
                output_storage_manager(
                    rk,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        multi_party_relinkey_method_I_stage_I_kernel<<<
                            dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                            options.stream_>>>(all_rk[0].data(),
                                               output_memory.data(),
                                               modulus_->data(), n_power,
                                               Q_prime_size_, dimension, true);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        for (int i = 1; i < participant_count; i++)
                        {
                            multi_party_relinkey_method_I_stage_I_kernel<<<
                                dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                options.stream_>>>(
                                all_rk[i].data(), output_memory.data(),
                                modulus_->data(), n_power, Q_prime_size_,
                                dimension, false);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_multi_party_relin_key(
        std::vector<MultipartyRelinkey<Scheme::BFV>>& all_rk,
        MultipartyRelinkey<Scheme::BFV>& rk_common_stage1,
        Relinkey<Scheme::BFV>& rk, const ExecutionOptions& options)
    {
        int participant_count = all_rk.size();

        if (participant_count == 0)
        {
            throw std::invalid_argument(
                "No participant to generate common publickey!");
        }

        for (int i = 0; i < participant_count; i++)
        {
            if (!all_rk[i].relin_key_generated_)
            {
                throw std::invalid_argument(
                    "MultipartyRelinkey is not generated!");
            }
        }

        if (!rk_common_stage1.relin_key_generated_)
        {
            throw std::logic_error("Common Relinkey is not generated!");
        }

        int dimension;
        switch (static_cast<int>(rk.key_type))
        {
            case 1: // KEYSWITCHING_METHOD_I
                dimension = rk.Q_size_;
                break;
            case 2: // KEYSWITCHING_METHOD_II
                dimension = rk.d_;
                break;
            case 3: // KEYSWITCHING_METHOD_III
                throw std::invalid_argument(
                    "Key Switching Type III is not supported for multi "
                    "party key generation.");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }

        input_vector_storage_manager(
            all_rk,
            [&](std::vector<MultipartyRelinkey<Scheme::BFV>>& all_rk_)
            {
                input_storage_manager(
                    rk_common_stage1,
                    [&](MultipartyRelinkey<Scheme::BFV>& rk_common_stage1_)
                    {
                        output_storage_manager(
                            rk,
                            [&](Relinkey<Scheme::BFV>& rk_)
                            {
                                DeviceVector<Data64> output_memory(
                                    rk_.relinkey_size_, options.stream_);

                                multi_party_relinkey_method_I_stage_II_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    all_rk[0].data(), rk_common_stage1.data(),
                                    output_memory.data(), modulus_->data(),
                                    n_power, Q_prime_size_, dimension);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                for (int i = 1; i < participant_count; i++)
                                {
                                    multi_party_relinkey_method_I_stage_II_kernel<<<
                                        dim3((n >> 8), Q_prime_size_, 1), 256,
                                        0, options.stream_>>>(
                                        all_rk[i].data(), output_memory.data(),
                                        modulus_->data(), n_power,
                                        Q_prime_size_, dimension);
                                    HEONGPU_CUDA_CHECK(hipGetLastError());
                                }

                                rk_.memory_set(std::move(output_memory));

                                rk_.relin_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_bfv_relin_key_method_II(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * d_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                      1),
                                                 256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), Sk_pair_->data(),
                            n_power, Q_prime_size_, d_, Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_relin_key_method_III(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * d_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> temp_calculation(
                            2 * Q_prime_size_ * d_ * n, options.stream_);

                        relinkey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                      1),
                                                 256, 0, options.stream_>>>(
                            temp_calculation.data(), sk.data(), error_poly,
                            a_poly, modulus_->data(), factor_->data(),
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::INVERSE,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .mod_inverse = n_inverse_->data(),
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            temp_calculation.data(), intt_table_->data(),
                            modulus_->data(), cfg_intt, 2 * Q_prime_size_ * d_,
                            Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_DtoB_kernel<<<dim3((n >> 8), d_tilda_,
                                                    (d_ << 1)),
                                               256, 0, options.stream_>>>(
                            temp_calculation.data(), output_memory.data(),
                            modulus_->data(), B_prime_->data(),
                            base_change_matrix_D_to_B_->data(),
                            Mi_inv_D_to_B_->data(), prod_D_to_B_->data(),
                            I_j_->data(), I_location_->data(), n_power,
                            Q_prime_size_, d_tilda_, d_, r_prime_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        gpuntt::GPU_NTT_Inplace(
                            output_memory.data(), B_prime_ntt_tables_->data(),
                            B_prime_->data(), cfg_ntt,
                            2 * d_tilda_ * d_ * r_prime_, r_prime_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_galois_key_method_I(
        Galoiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<Data64> errors_a(2 * Q_prime_size_ * Q_size_ * n,
                                              options.stream_);
                Data64* error_poly = errors_a.data();
                Data64* a_poly = error_poly + (Q_prime_size_ * Q_size_ * n);

                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            Q_size_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            Q_size_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_galois_key_method_II(
        Galoiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<Data64> errors_a(2 * Q_prime_size_ * d_ * n,
                                              options.stream_);
                Data64* error_poly = errors_a.data();
                Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            d_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            d_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_multi_party_galois_key_piece_method_I(
        MultipartyGaloiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        RNGSeed common_seed = gk.seed();

        DeviceVector<Data64> errors_a(2 * Q_prime_size_ * Q_size_ * n,
                                      options.stream_);
        Data64* error_poly = errors_a.data();
        Data64* a_poly = error_poly + (Q_prime_size_ * Q_size_ * n);

        RandomNumberGenerator::instance().set(
            common_seed.key_, common_seed.nonce_,
            common_seed.personalization_string_, options.stream_);
        RandomNumberGenerator::instance()
            .modular_ternary_random_number_generation(a_poly, modulus_->data(),
                                                      n_power, Q_prime_size_,
                                                      Q_size_, options.stream_);

        RNGSeed gen_seed1;
        RandomNumberGenerator::instance().set(gen_seed1.key_, gen_seed1.nonce_,
                                              gen_seed1.personalization_string_,
                                              options.stream_);

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::
        generate_bfv_multi_party_galois_key_piece_method_II(
            MultipartyGaloiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
            const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        RNGSeed common_seed = gk.seed();

        DeviceVector<Data64> errors_a(2 * Q_prime_size_ * d_ * n,
                                      options.stream_);
        Data64* error_poly = errors_a.data();
        Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

        RandomNumberGenerator::instance().set(
            common_seed.key_, common_seed.nonce_,
            common_seed.personalization_string_, options.stream_);
        RandomNumberGenerator::instance()
            .modular_ternary_random_number_generation(a_poly, modulus_->data(),
                                                      n_power, Q_prime_size_,
                                                      d_, options.stream_);

        RNGSeed gen_seed1;
        RandomNumberGenerator::instance().set(gen_seed1.key_, gen_seed1.nonce_,
                                              gen_seed1.personalization_string_,
                                              options.stream_);

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_multi_party_galois_key(
        std::vector<MultipartyGaloiskey<Scheme::BFV>>& all_gk,
        Galoiskey<Scheme::BFV>& gk, const ExecutionOptions& options)
    {
        int participant_count = all_gk.size();

        if (participant_count == 0)
        {
            throw std::invalid_argument(
                "No participant to generate common galois!");
        }

        for (int i = 0; i < participant_count; i++)
        {
            if ((gk.customized != all_gk[i].customized) ||
                (gk.group_order_ != all_gk[i].group_order_))
            {
                throw std::invalid_argument(
                    "MultipartyGaloiskey context is not valid || "
                    "MultipartyGaloiskey is not generated!");
            }
        }

        for (int i = 0; i < participant_count; i++)
        {
            if (!all_gk[i].galois_key_generated_)
            {
                throw std::invalid_argument(
                    "MultipartyGaloiskey is not generated!");
            }
        }

        std::vector<storage_type> input_storage_types;
        for (int i = 0; i < participant_count; i++)
        {
            input_storage_types.push_back(all_gk[i].storage_type_);

            if (all_gk[i].storage_type_ == storage_type::DEVICE)
            {
                for (const auto& pair : all_gk[i].device_location_)
                {
                    if (pair.second.size() < all_gk[i].galoiskey_size_)
                    {
                        throw std::invalid_argument(
                            "MultipartyGaloiskeys size is not valid || "
                            "MultipartyGaloiskeys is not generated!");
                    }
                }
            }
            else
            {
                for (const auto& pair : all_gk[i].host_location_)
                {
                    if (pair.second.size() < all_gk[i].galoiskey_size_)
                    {
                        throw std::invalid_argument(
                            "MultipartyGaloiskeys size is not valid || "
                            "MultipartyGaloiskeys is not generated!");
                    }
                }

                all_gk[i].store_in_device();
            }
        }

        int dimension;
        switch (static_cast<int>(gk.key_type))
        {
            case 1: // KEYSWITCHING_METHOD_I
                dimension = gk.Q_size_;
                break;
            case 2: // KEYSWITCHING_METHOD_II
                dimension = gk.d_;
                break;
            case 3: // KEYSWITCHING_METHOD_III
                throw std::invalid_argument(
                    "Key Switching Type III is not supported for multi "
                    "party key generation.");
                break;
            default:
                throw std::invalid_argument("Invalid Key Switching Type");
                break;
        }

        for (auto& galois : gk.galois_elt)
        {
            DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                               options.stream_);

            multi_party_galoiskey_gen_method_I_II_kernel<<<
                dim3((n >> 8), Q_prime_size_, 1), 256, 0, options.stream_>>>(
                output_memory.data(),
                all_gk[0].device_location_[galois.second].data(),
                modulus_->data(), n_power, Q_prime_size_, dimension, true);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            for (int i = 1; i < participant_count; i++)
            {
                multi_party_galoiskey_gen_method_I_II_kernel<<<
                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                    options.stream_>>>(
                    output_memory.data(),
                    all_gk[i].device_location_[galois.second].data(),
                    modulus_->data(), n_power, Q_prime_size_, dimension, false);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }

            if (options.storage_ == storage_type::DEVICE)
            {
                gk.device_location_[galois.second] = std::move(output_memory);
            }
            else
            {
                gk.host_location_[galois.second] =
                    HostVector<Data64>(gk.galoiskey_size_);
                hipMemcpyAsync(gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }
        }

        DeviceVector<Data64> output_memory(gk.galoiskey_size_, options.stream_);

        multi_party_galoiskey_gen_method_I_II_kernel<<<
            dim3((n >> 8), Q_prime_size_, 1), 256, 0, options.stream_>>>(
            output_memory.data(), all_gk[0].zero_device_location_.data(),
            modulus_->data(), n_power, Q_prime_size_, dimension, true);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        for (int i = 1; i < participant_count; i++)
        {
            multi_party_galoiskey_gen_method_I_II_kernel<<<
                dim3((n >> 8), Q_prime_size_, 1), 256, 0, options.stream_>>>(
                output_memory.data(), all_gk[i].zero_device_location_.data(),
                modulus_->data(), n_power, Q_prime_size_, dimension, false);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        if (options.storage_ == storage_type::DEVICE)
        {
            gk.zero_device_location_ = std::move(output_memory);
        }
        else
        {
            gk.zero_host_location_ = HostVector<Data64>(gk.galoiskey_size_);
            hipMemcpyAsync(gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        if (options.keep_initial_condition_)
        {
            for (int i = 0; i < participant_count; i++)
            {
                if (input_storage_types[i] == storage_type::DEVICE)
                {
                    // pass
                }
                else
                {
                    all_gk[i].store_in_host();
                }
            }
        }
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_switch_key_method_I(
        Switchkey<Scheme::BFV>& swk, Secretkey<Scheme::BFV>& new_sk,
        Secretkey<Scheme::BFV>& old_sk, const ExecutionOptions& options)
    {
        if (!old_sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!new_sk.secret_key_generated_)
        {
            throw std::logic_error("Ner Secretkey is not generated!");
        }

        if (swk.switch_key_generated_)
        {
            throw std::logic_error("Switchkey is already generated!");
        }

        input_storage_manager(
            old_sk,
            [&](Secretkey<Scheme::BFV>& old_sk_)
            {
                input_storage_manager(
                    new_sk,
                    [&](Secretkey<Scheme::BFV>& new_sk_)
                    {
                        output_storage_manager(
                            swk,
                            [&](Switchkey<Scheme::BFV>& swk_)
                            {
                                DeviceVector<Data64> errors_a(
                                    2 * Q_prime_size_ * Q_size_ * n,
                                    options.stream_);
                                Data64* error_poly = errors_a.data();
                                Data64* a_poly =
                                    error_poly + (Q_prime_size_ * Q_size_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_uniform_random_number_generation(
                                        a_poly, modulus_->data(), n_power,
                                        Q_prime_size_, Q_size_,
                                        options.stream_);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, error_poly,
                                        modulus_->data(), n_power,
                                        Q_prime_size_, Q_size_,
                                        options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    error_poly, ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    Q_size_ * Q_prime_size_, Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    swk_.switchkey_size_, options.stream_);

                                switchkey_gen_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    output_memory.data(), new_sk.data(),
                                    old_sk.data(), error_poly, a_poly,
                                    modulus_->data(), factor_->data(), n_power,
                                    Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                swk.memory_set(std::move(output_memory));

                                swk.switch_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_switch_key_method_II(
        Switchkey<Scheme::BFV>& swk, Secretkey<Scheme::BFV>& new_sk,
        Secretkey<Scheme::BFV>& old_sk, const ExecutionOptions& options)
    {
        if (!old_sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!new_sk.secret_key_generated_)
        {
            throw std::logic_error("Ner Secretkey is not generated!");
        }

        if (swk.switch_key_generated_)
        {
            throw std::logic_error("Switchkey is already generated!");
        }

        input_storage_manager(
            old_sk,
            [&](Secretkey<Scheme::BFV>& old_sk_)
            {
                input_storage_manager(
                    new_sk,
                    [&](Secretkey<Scheme::BFV>& new_sk_)
                    {
                        output_storage_manager(
                            swk,
                            [&](Switchkey<Scheme::BFV>& swk_)
                            {
                                DeviceVector<Data64> errors_a(
                                    2 * Q_prime_size_ * d_ * n,
                                    options.stream_);
                                Data64* error_poly = errors_a.data();
                                Data64* a_poly =
                                    error_poly + (Q_prime_size_ * d_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_uniform_random_number_generation(
                                        a_poly, modulus_->data(), n_power,
                                        Q_prime_size_, d_, options.stream_);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, error_poly,
                                        modulus_->data(), n_power,
                                        Q_prime_size_, d_, options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    error_poly, ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    d_ * Q_prime_size_, Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    swk_.switchkey_size_, options.stream_);

                                switchkey_gen_II_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    output_memory.data(), new_sk.data(),
                                    old_sk.data(), error_poly, a_poly,
                                    modulus_->data(), factor_->data(),
                                    Sk_pair_->data(), n_power, Q_prime_size_,
                                    d_, Q_size_, P_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                swk.memory_set(std::move(output_memory));

                                swk.switch_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

} // namespace heongpu