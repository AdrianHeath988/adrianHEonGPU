﻿// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bfv/encryptor.cuh"

namespace heongpu
{
    __host__
    HEEncryptor<Scheme::BFV>::HEEncryptor(HEContext<Scheme::BFV>& context,
                                          Publickey<Scheme::BFV>& public_key)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;

        std::random_device rd;
        std::mt19937 gen(rd());
        seed_ = gen();
        offset_ = gen();

        if (public_key.storage_type_ == storage_type::DEVICE)
        {
            public_key_ = public_key.device_locations_;
        }
        else
        {
            public_key.store_in_device();
            public_key_ = public_key.device_locations_;
        }

        n = context.n;
        n_power = context.n_power;

        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;
        P_size_ = context.P_size;

        modulus_ = context.modulus_;

        last_q_modinv_ = context.last_q_modinv_;

        ntt_table_ = context.ntt_table_;
        intt_table_ = context.intt_table_;

        n_inverse_ = context.n_inverse_;

        half_ = context.half_p_;

        half_mod_ = context.half_mod_;

        n = context.n;
        n_power = context.n_power;

        if (scheme_ == scheme_type::bfv)
        {
            plain_modulus_ = context.plain_modulus_;

            Q_mod_t_ = context.Q_mod_t_;

            upper_threshold_ = context.upper_threshold_;

            coeeff_div_plainmod_ = context.coeeff_div_plainmod_;
        }
        else
        {
        }
    }

    __host__ void
    HEEncryptor<Scheme::BFV>::encrypt_bfv(Ciphertext<Scheme::BFV>& ciphertext,
                                          Plaintext<Scheme::BFV>& plaintext,
                                          const hipStream_t stream)
    {
        DeviceVector<Data64> output_memory((2 * n * Q_size_), stream);

        DeviceVector<Data64> gpu_space(5 * Q_prime_size_ * n, stream);
        Data64* u_poly = gpu_space.data();
        Data64* error_poly = u_poly + (Q_prime_size_ * n);
        Data64* pk_u_poly = error_poly + (2 * Q_prime_size_ * n);

        RandomNumberGenerator::instance()
            .modular_ternary_random_number_generation(
                u_poly, modulus_->data(), n_power, Q_prime_size_, 1, stream);

        RandomNumberGenerator::instance()
            .modular_gaussian_random_number_generation(
                error_std_dev, error_poly, modulus_->data(), n_power,
                Q_prime_size_, 2, stream);

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        gpuntt::GPU_NTT_Inplace(u_poly, ntt_table_->data(), modulus_->data(),
                                cfg_ntt, Q_prime_size_, Q_prime_size_);

        pk_u_kernel<<<dim3((n >> 8), Q_prime_size_, 2), 256, 0, stream>>>(
            public_key_.data(), u_poly, pk_u_poly, modulus_->data(), n_power,
            Q_prime_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT_Inplace(pk_u_poly, intt_table_->data(),
                                modulus_->data(), cfg_intt, 2 * Q_prime_size_,
                                Q_prime_size_);

        enc_div_lastq_bfv_kernel<<<dim3((n >> 8), Q_size_, 2), 256, 0,
                                   stream>>>(
            pk_u_poly, error_poly, plaintext.data(), output_memory.data(),
            modulus_->data(), half_->data(), half_mod_->data(),
            last_q_modinv_->data(), plain_modulus_, Q_mod_t_, upper_threshold_,
            coeeff_div_plainmod_->data(), n_power, Q_prime_size_, Q_size_,
            P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        ciphertext.memory_set(std::move(output_memory));
    }

} // namespace heongpu