#include "hip/hip_runtime.h"
#include "secretkey_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/secretkey.cuh" // The C++ class we are wrapping
#include "util/hostvector.cuh"
#include "util/schemes.h"
#include "util/storagemanager.cuh"

#include <vector>
#include <sstream>
#include <iostream>
#include <algorithm> // For std::min
#include <cstring>   // For std::memcpy
#include <new>       // For std::nothrow

// Define the opaque struct
typedef struct HE_CKKS_SecretKey_s HE_CKKS_SecretKey;
// Helper to safely access underlying C++ HEContext pointer
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) { // Assuming cpp_context from context_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

// Helper to map C++ scheme_type to C_scheme_type (should be in a common C API util or context_c_api.cu)
static C_scheme_type map_cpp_to_c_scheme_type_sk(heongpu::scheme_type cpp_type) {
    switch (cpp_type) {
        case heongpu::scheme_type::none: return C_SCHEME_TYPE_NONE;
        case heongpu::scheme_type::bfv:  return C_SCHEME_TYPE_BFV;
        case heongpu::scheme_type::ckks: return C_SCHEME_TYPE_CKKS;
        case heongpu::scheme_type::bgv:  return C_SCHEME_TYPE_BGV;
        default: return static_cast<C_scheme_type>(-1); // Error/Unknown
    }
}

// Helper to map C++ storage_type to C_storage_type (should be in a common C API util or context_c_api.cu)
static C_storage_type map_cpp_to_c_storage_type_sk(heongpu::storage_type cpp_type) {
    switch (cpp_type) {
        case heongpu::storage_type::HOST:   return C_STORAGE_TYPE_HOST;
        case heongpu::storage_type::DEVICE: return C_STORAGE_TYPE_DEVICE;
        default: return C_STORAGE_TYPE_INVALID;
    }
}

extern "C" {

// --- CKKS SecretKey Lifecycle & Serialization ---

HE_CKKS_SecretKey* HEonGPU_CKKS_SecretKey_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        // Uses the constructor SecretKey(HEContext<Scheme::CKKS>& context);
        heongpu::SecretKey<heongpu::Scheme::CKKS>* cpp_sk_obj =
            new (std::nothrow) heongpu::SecretKey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_sk_obj) {
            std::cerr << "HEonGPU_CKKS_SecretKey_Create failed: C++ SecretKey allocation failed." << std::endl;
            return nullptr;
        }

        HE_CKKS_SecretKey* c_api_sk = new (std::nothrow) HE_CKKS_SecretKey_s;
        if (!c_api_sk) {
            std::cerr << "HEonGPU_CKKS_SecretKey_Create failed: C API SecretKey wrapper allocation failed." << std::endl;
            delete cpp_sk_obj;
            return nullptr;
        }
        c_api_sk->cpp_secretkey = cpp_sk_obj;
        return c_api_sk;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_SecretKey_Delete(HE_CKKS_SecretKey* sk) {
    if (sk) {
        delete sk->cpp_secretkey;
        delete sk;
    }
}

HE_CKKS_SecretKey* HEonGPU_CKKS_SecretKey_Clone(const HE_CKKS_SecretKey* other_sk) {
    if (!other_sk || !other_sk->cpp_secretkey) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Clone failed: Invalid source secret key pointer." << std::endl;
        return nullptr;
    }
    try {
        heongpu::SecretKey<heongpu::Scheme::CKKS>* cpp_cloned_sk =
            new (std::nothrow) heongpu::SecretKey<heongpu::Scheme::CKKS>(*(other_sk->cpp_secretkey));
         if (!cpp_cloned_sk) {
            std::cerr << "HEonGPU_CKKS_SecretKey_Clone failed: C++ SecretKey (clone) allocation failed." << std::endl;
            return nullptr;
        }
        HE_CKKS_SecretKey* c_api_cloned_sk = new (std::nothrow) HE_CKKS_SecretKey_s;
        if (!c_api_cloned_sk) {
            std::cerr << "HEonGPU_CKKS_SecretKey_Clone failed: C API SecretKey wrapper (clone) allocation failed." << std::endl;
            delete cpp_cloned_sk;
            return nullptr;
        }
        c_api_cloned_sk->cpp_secretkey = cpp_cloned_sk;
        return c_api_cloned_sk;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Clone failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Clone failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

int HEonGPU_CKKS_SecretKey_Assign_Copy(HE_CKKS_SecretKey* dest_sk,
                                       const HE_CKKS_SecretKey* src_sk) {
    if (!dest_sk || !dest_sk->cpp_secretkey || !src_sk || !src_sk->cpp_secretkey) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Assign_Copy failed: Invalid destination or source secret key pointer." << std::endl;
        return -1; 
    }
    try {
        *(dest_sk->cpp_secretkey) = *(src_sk->cpp_secretkey);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Assign_Copy failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Assign_Copy failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_SecretKey_Save(HE_CKKS_SecretKey* sk,
                                unsigned char** out_bytes,
                                size_t* out_len) {
    if (!sk || !sk->cpp_secretkey || !out_bytes || !out_len) {
        if(out_bytes) *out_bytes = nullptr;
        if(out_len) *out_len = 0;
        return -1;
    }
    *out_bytes = nullptr;
    *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        sk->cpp_secretkey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len == 0) {
            *out_bytes = nullptr;
            return 0; 
        }
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) {
            *out_len = 0;
            std::cerr << "HEonGPU_CKKS_SecretKey_Save failed: Memory allocation error." << std::endl;
            return -2;
        }
        std::memcpy(*out_bytes, str_data.data(), *out_len);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Save failed with C++ exception: " << e.what() << std::endl;
        if(*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Save failed due to an unknown C++ exception." << std::endl;
        if(*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    }
}

HE_CKKS_SecretKey* HEonGPU_CKKS_SecretKey_Load(HE_CKKS_Context* context,
                                               const unsigned char* bytes,
                                               size_t len) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Load failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    if (!bytes && len > 0) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Load failed: Invalid bytes pointer for non-zero length." << std::endl;
        return nullptr;
    }

    HE_CKKS_SecretKey* c_api_sk = nullptr;
    heongpu::SecretKey<heongpu::Scheme::CKKS>* cpp_sk = nullptr;
    try {
        // Create a SecretKey object using the context, then load into it.
        cpp_sk = new (std::nothrow) heongpu::SecretKey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_sk) {
             std::cerr << "HEonGPU_CKKS_SecretKey_Load failed: C++ SecretKey allocation failed." << std::endl;
            return nullptr;
        }

        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_sk->load(iss);
        }
        // If len is 0, cpp_sk is a newly constructed (default) secret key for that context.

        c_api_sk = new (std::nothrow) HE_CKKS_SecretKey_s;
        if (!c_api_sk) {
            std::cerr << "HEonGPU_CKKS_SecretKey_Load failed: C API SecretKey wrapper allocation failed." << std::endl;
            delete cpp_sk;
            return nullptr;
        }
        c_api_sk->cpp_secretkey = cpp_sk;
        return c_api_sk;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Load failed with C++ exception: " << e.what() << std::endl;
        delete cpp_sk;
        delete c_api_sk;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_Load failed due to an unknown C++ exception." << std::endl;
        delete cpp_sk;
        delete c_api_sk;
        return nullptr;
    }
}

// --- CKKS SecretKey Getters ---
C_scheme_type HEonGPU_CKKS_SecretKey_GetScheme(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_sk(sk->cpp_secretkey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}

int HEonGPU_CKKS_SecretKey_GetRingSize(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return 0;
    try { return sk->cpp_secretkey->ring_size(); } catch (...) { return 0; }
}

int HEonGPU_CKKS_SecretKey_GetCoeffModulusCount(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return 0;
    try { return sk->cpp_secretkey->coeff_modulus_count(); } catch (...) { return 0; }
}

int HEonGPU_CKKS_SecretKey_GetNPower(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return 0;
    try { return sk->cpp_secretkey->n_power(); } catch (...) { return 0; }
}

int HEonGPU_CKKS_SecretKey_GetHammingWeight(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return 0; // Or -1 as HW can be 0.
    try { return sk->cpp_secretkey->hamming_weight(); } catch (...) { return -1; }
}

bool HEonGPU_CKKS_SecretKey_IsInNttDomain(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return false;
    try { return sk->cpp_secretkey->is_in_ntt_domain(); } catch (...) { return false; }
}

bool HEonGPU_CKKS_SecretKey_IsGenerated(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return false;
    try { return sk->cpp_secretkey->is_generated(); } catch (...) { return false; }
}

C_storage_type HEonGPU_CKKS_SecretKey_GetStorageType(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_sk(sk->cpp_secretkey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}

size_t HEonGPU_CKKS_SecretKey_GetData(HE_CKKS_SecretKey* sk,
                                      uint64_t* data_buffer,
                                      size_t buffer_elements,
                                      C_cudaStream_t stream) {
    if (!sk || !sk->cpp_secretkey || (!data_buffer && buffer_elements > 0)) {
        std::cerr << "Error: Invalid arguments in SecretKey GetData." << std::endl;
        return 0;
    }
    try {
        heongpu::HostVector<heongpu::Data64> temp_host_vector;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        
        sk->cpp_secretkey->get_data(temp_host_vector, cpp_stream);

        size_t elements_in_sk = temp_host_vector.size();
        size_t elements_to_copy = std::min(buffer_elements, elements_in_sk);

        if (elements_to_copy > 0 && data_buffer) {
            std::memcpy(data_buffer, temp_host_vector.data(), elements_to_copy * sizeof(heongpu::Data64));
        }
        return elements_to_copy;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_GetData failed with C++ exception: " << e.what() << std::endl;
        return 0;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_GetData failed due to an unknown C++ exception." << std::endl;
        return 0;
    }
}

// --- CKKS SecretKey Setters ---
int HEonGPU_CKKS_SecretKey_SetData(HE_CKKS_SecretKey* sk,
                                   const uint64_t* data_buffer, // heongpu::Data64 is uint64_t
                                   size_t num_elements,
                                   C_cudaStream_t stream) {
    if (!sk || !sk->cpp_secretkey || (!data_buffer && num_elements > 0)) {
        std::cerr << "Error: Invalid arguments in SecretKey SetData." << std::endl;
        return -1; // Error
    }
    try {
        heongpu::HostVector<heongpu::Data64> input_host_vector(num_elements);
        if (num_elements > 0 && data_buffer) {
             std::memcpy(input_host_vector.data(), data_buffer, num_elements * sizeof(heongpu::Data64));
        }
        
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        sk->cpp_secretkey->set_data(input_host_vector, cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_SecretKey_SetData failed with C++ exception: " << e.what() << std::endl;
        return -2; // Error
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_SecretKey_SetData failed due to an unknown C++ exception." << std::endl;
        return -2; // Error
    }
}

} // extern "C"