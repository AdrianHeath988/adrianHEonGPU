#include "keygenerator_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/secretkey.cuh"
#include "ckks/publickey.cuh"
#include "ckks/evaluationkey.cuh"
#include "ckks/keygenerator.cuh" // The C++ class we are wrapping

#include "random.cuh"         // For heongpu::RNGSeed
#include "storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct
typedef struct HE_CKKS_KeyGenerator_s HE_CKKS_KeyGenerator;
// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context_kg(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr;
    return context->cpp_context;
}
static heongpu::SecretKey<heongpu::Scheme::CKKS>* get_cpp_secretkey(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr;
    return sk->cpp_secretkey;
}
static const heongpu::SecretKey<heongpu::Scheme::CKKS>* get_const_cpp_secretkey(const HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr;
    return sk->cpp_secretkey;
}
static heongpu::Publickey<heongpu::Scheme::CKKS>* get_cpp_publickey(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return nullptr;
    return pk->cpp_publickey;
}
static heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* get_cpp_mp_publickey(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return nullptr;
    return mp_pk->cpp_mp_publickey;
}
static const heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* get_const_cpp_mp_publickey(const HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return nullptr;
    return mp_pk->cpp_mp_publickey;
}
static heongpu::Relinkey<heongpu::Scheme::CKKS>* get_cpp_relinkey(HE_CKKS_RelinKey* rlk) {
    if (!rlk || !rlk->cpp_relinkey) return nullptr;
    return rlk->cpp_relinkey;
}
static heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* get_cpp_mp_relinkey(HE_CKKS_MultipartyRelinKey* mp_rlk) {
    if (!mp_rlk || !mp_rlk->cpp_mp_relinkey) return nullptr;
    return mp_rlk->cpp_mp_relinkey;
}
static const heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* get_const_cpp_mp_relinkey(const HE_CKKS_MultipartyRelinKey* mp_rlk) {
    if (!mp_rlk || !mp_rlk->cpp_mp_relinkey) return nullptr;
    return mp_rlk->cpp_mp_relinkey;
}
static heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_cpp_galoiskey(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}
static const heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_const_cpp_galoiskey(const HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}

// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_kg(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEKeyGenerator Lifecycle ---
HE_CKKS_KeyGenerator* HEonGPU_CKKS_KeyGenerator_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_kg(context);
    if (!cpp_h_context) {
        std::cerr << "KeyGenerator_Create failed: Invalid context." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEKeyGenerator<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) { std::cerr << "KeyGenerator_Create: C++ allocation failed.\n"; return nullptr; }
        auto c_api_obj = new (std::nothrow) HE_CKKS_KeyGenerator_s;
        if (!c_api_obj) { delete cpp_obj; std::cerr << "KeyGenerator_Create: C API wrapper allocation failed.\n"; return nullptr; }
        c_api_obj->cpp_keygen = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "KeyGenerator_Create Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "KeyGenerator_Create Unknown Error" << std::endl; return nullptr; }
}

void HEonGPU_CKKS_KeyGenerator_Delete(HE_CKKS_KeyGenerator* kg) {
    if (kg) { delete kg->cpp_keygen; delete kg; }
}

// --- Seed Configuration ---
void HEonGPU_CKKS_KeyGenerator_SetSeed(HE_CKKS_KeyGenerator* kg, const C_RNGSeed_Const_Data* seed_c) {
    if (!kg || !kg->cpp_keygen || !seed_c) {
        std::cerr << "KeyGenerator_SetSeed: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::RNGSeed cpp_seed;
        if (seed_c->key_data && seed_c->key_len > 0) {
            cpp_seed.key_.assign(seed_c->key_data, seed_c->key_data + seed_c->key_len);
        }
        if (seed_c->nonce_data && seed_c->nonce_len > 0) {
            cpp_seed.nonce_.assign(seed_c->nonce_data, seed_c->nonce_data + seed_c->nonce_len);
        }
        if (seed_c->pstring_data && seed_c->pstring_len > 0) {
            cpp_seed.personalization_string_.assign(seed_c->pstring_data, seed_c->pstring_data + seed_c->pstring_len);
        }
        kg->cpp_keygen->set_seed(cpp_seed);
    } catch (const std::exception& e) { std::cerr << "KeyGenerator_SetSeed Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "KeyGenerator_SetSeed Unknown Error" << std::endl; }
}

// --- Standard Key Generation ---
#define WRAP_STD_KEYGEN_FUNC(FuncName, CppKeyType, CKeyType, GetCppKeyFunc) \
int FuncName(HE_CKKS_KeyGenerator* kg, CKeyType* key_out_c, const HE_CKKS_SecretKey* sk_c, const C_ExecutionOptions* options_c) { \
    if (!kg || !kg->cpp_keygen || !key_out_c || !GetCppKeyFunc(key_out_c) || !get_const_cpp_secretkey(sk_c)) { \
        std::cerr << #FuncName " Error: Invalid argument(s).\n"; return -1; \
    } \
    try { \
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c); \
        kg->cpp_keygen->CppKeyType(*(GetCppKeyFunc(key_out_c)), *(get_const_cpp_secretkey(sk_c)), cpp_options); \
        return 0; \
    } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return -2; } \
      catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return -2; } \
}

int HEonGPU_CKKS_KeyGenerator_GenerateSecretKey(HE_CKKS_KeyGenerator* kg, HE_CKKS_SecretKey* sk_c, int hamming_weight, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !sk_c || !get_cpp_secretkey(sk_c)) {
        std::cerr << "GenerateSecretKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_secret_key(*(get_cpp_secretkey(sk_c)), hamming_weight, cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "GenerateSecretKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "GenerateSecretKey Unknown Error" << std::endl; return -2; }
}

WRAP_STD_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GeneratePublicKey, generate_public_key, HE_CKKS_PublicKey, get_cpp_publickey)
WRAP_STD_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GenerateRelinKey, generate_relinkey, HE_CKKS_RelinKey, get_cpp_relinkey)
WRAP_STD_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GenerateGaloisKey, generate_galoiskey, HE_CKKS_GaloisKey, get_cpp_galoiskey)


// --- Multiparty Key Generation ---
#define WRAP_MP_KEYGEN_FUNC(FuncName, CppFuncName, CppKeyType, CKeyType, GetCppKeyFunc) \
int FuncName(HE_CKKS_KeyGenerator* kg, CKeyType* key_out_c, const HE_CKKS_SecretKey* sk_c, const C_ExecutionOptions* options_c) { \
    if (!kg || !kg->cpp_keygen || !key_out_c || !GetCppKeyFunc(key_out_c) || !get_const_cpp_secretkey(sk_c)) { \
        std::cerr << #FuncName " Error: Invalid argument(s).\n"; return -1; \
    } \
    try { \
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c); \
        kg->cpp_keygen->CppFuncName(*(GetCppKeyFunc(key_out_c)), *(get_const_cpp_secretkey(sk_c)), cpp_options); \
        return 0; \
    } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return -2; } \
      catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return -2; } \
}

WRAP_MP_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GenerateMultipartyPublicKey, generate_multiparty_public_key, heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>, HE_CKKS_MultipartyPublicKey, get_cpp_mp_publickey)
WRAP_MP_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GenerateMultipartyRelinKey, generate_multiparty_relinkey, heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>, HE_CKKS_MultipartyRelinKey, get_cpp_mp_relinkey)
WRAP_MP_KEYGEN_FUNC(HEonGPU_CKKS_KeyGenerator_GenerateMultipartyGaloisKey, generate_multiparty_galoiskey, heongpu::Galoiskey<heongpu::Scheme::CKKS>, HE_CKKS_GaloisKey, get_cpp_galoiskey) /* C++ uses Galoiskey here */


// --- Multiparty Key Aggregation ---
int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyPublicKey(HE_CKKS_KeyGenerator* kg, const HE_CKKS_MultipartyPublicKey* const* public_keys_array_c, size_t num_public_keys, HE_CKKS_PublicKey* aggregated_pk_c, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || (num_public_keys > 0 && !public_keys_array_c) || !aggregated_pk_c || !get_cpp_publickey(aggregated_pk_c)) {
        std::cerr << "AggregateMultipartyPublicKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        std::vector<heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>> cpp_pk_vec;
        cpp_pk_vec.reserve(num_public_keys);
        for (size_t i = 0; i < num_public_keys; ++i) {
            const heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* cpp_mp_pk = get_const_cpp_mp_publickey(public_keys_array_c[i]);
            if (!cpp_mp_pk) { std::cerr << "AggregateMultipartyPublicKey Error: Null key in array at index " << i << std::endl; return -1; }
            cpp_pk_vec.push_back(*cpp_mp_pk); // Makes a copy, C++ method takes vector of objects
        }
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->aggregate_multiparty_public_key(cpp_pk_vec, *(get_cpp_publickey(aggregated_pk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "AggregateMultipartyPublicKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "AggregateMultipartyPublicKey Unknown Error" << std::endl; return -2; }
}


int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyRelinKey(HE_CKKS_KeyGenerator* kg, const HE_CKKS_MultipartyRelinKey* const* relin_keys_array_c, size_t num_relin_keys, HE_CKKS_RelinKey* aggregated_rlk_c, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || (num_relin_keys > 0 && !relin_keys_array_c) || !aggregated_rlk_c || !get_cpp_relinkey(aggregated_rlk_c)) {
         std::cerr << "AggregateMultipartyRelinKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        std::vector<heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>> cpp_rlk_vec;
        cpp_rlk_vec.reserve(num_relin_keys);
        for (size_t i = 0; i < num_relin_keys; ++i) {
             const heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* cpp_mp_rlk = get_const_cpp_mp_relinkey(relin_keys_array_c[i]);
             if (!cpp_mp_rlk) { std::cerr << "AggregateMultipartyRelinKey Error: Null key in array at index " << i << std::endl; return -1; }
            cpp_rlk_vec.push_back(*cpp_mp_rlk);
        }
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->aggregate_multiparty_relinkey(cpp_rlk_vec, *(get_cpp_relinkey(aggregated_rlk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "AggregateMultipartyRelinKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "AggregateMultipartyRelinKey Unknown Error" << std::endl; return -2; }
}

int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyGaloisKey(HE_CKKS_KeyGenerator* kg, const HE_CKKS_GaloisKey* const* galois_keys_array_c, size_t num_galois_keys, HE_CKKS_GaloisKey* aggregated_gk_c, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || (num_galois_keys > 0 && !galois_keys_array_c) || !aggregated_gk_c || !get_cpp_galoiskey(aggregated_gk_c) ) {
        std::cerr << "AggregateMultipartyGaloisKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        std::vector<heongpu::Galoiskey<heongpu::Scheme::CKKS>> cpp_gk_vec; // C++ takes vector of Galoiskey
        cpp_gk_vec.reserve(num_galois_keys);
        for (size_t i = 0; i < num_galois_keys; ++i) {
            const heongpu::Galoiskey<heongpu::Scheme::CKKS>* cpp_gk = get_const_cpp_galoiskey(galois_keys_array_c[i]);
            if (!cpp_gk) { std::cerr << "AggregateMultipartyGaloisKey Error: Null key in array at index " << i << std::endl; return -1; }
            cpp_gk_vec.push_back(*cpp_gk);
        }
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->aggregate_multiparty_galoiskey(cpp_gk_vec, *(get_cpp_galoiskey(aggregated_gk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "AggregateMultipartyGaloisKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "AggregateMultipartyGaloisKey Unknown Error" << std::endl; return -2; }
}

} // extern "C"