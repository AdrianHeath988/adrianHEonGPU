#include "keygenerator_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/secretkey.cuh"
#include "ckks/publickey.cuh"
#include "ckks/evaluationkey.cuh"
#include "ckks/keygenerator.cuh" // The C++ class we are wrapping

#include "random.cuh"         // For heongpu::RNGSeed
#include "storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct
typedef struct HE_CKKS_KeyGenerator_s HE_CKKS_KeyGenerator;

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context_kg(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr;
    return context->cpp_context;
}
static heongpu::Secretkey<heongpu::Scheme::CKKS>* get_cpp_secretkey(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr;
    return sk->cpp_secretkey;
}
static const heongpu::Secretkey<heongpu::Scheme::CKKS>* get_const_cpp_secretkey(const HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr;
    return sk->cpp_secretkey;
}
static heongpu::Publickey<heongpu::Scheme::CKKS>* get_cpp_publickey(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return nullptr;
    return pk->cpp_publickey;
}
static heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* get_cpp_mp_publickey(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return nullptr;
    return mp_pk->cpp_mp_publickey;
}
static const heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* get_const_cpp_mp_publickey(const HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return nullptr;
    return mp_pk->cpp_mp_publickey;
}
static heongpu::Relinkey<heongpu::Scheme::CKKS>* get_cpp_relinkey(HE_CKKS_RelinKey* rlk) {
    if (!rlk || !rlk->cpp_relinkey) return nullptr;
    return rlk->cpp_relinkey;
}
static heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* get_cpp_mp_relinkey(HE_CKKS_MultipartyRelinKey* mp_rlk) {
    if (!mp_rlk || !mp_rlk->cpp_mp_relinkey) return nullptr;
    return mp_rlk->cpp_mp_relinkey;
}
static const heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* get_const_cpp_mp_relinkey(const HE_CKKS_MultipartyRelinKey* mp_rlk) {
    if (!mp_rlk || !mp_rlk->cpp_mp_relinkey) return nullptr;
    return mp_rlk->cpp_mp_relinkey;
}
static heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_cpp_galoiskey(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}
static const heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_const_cpp_galoiskey(const HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}

// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_kg(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEKeyGenerator Lifecycle ---
HE_CKKS_KeyGenerator* HEonGPU_CKKS_KeyGenerator_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_kg(context);
    if (!cpp_h_context) {
        std::cerr << "KeyGenerator_Create failed: Invalid context." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEKeyGenerator<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) { std::cerr << "KeyGenerator_Create: C++ allocation failed.\n"; return nullptr; }
        auto c_api_obj = new (std::nothrow) HE_CKKS_KeyGenerator_s;
        if (!c_api_obj) { delete cpp_obj; std::cerr << "KeyGenerator_Create: C API wrapper allocation failed.\n"; return nullptr; }
        c_api_obj->cpp_keygen = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "KeyGenerator_Create Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "KeyGenerator_Create Unknown Error" << std::endl; return nullptr; }
}

void HEonGPU_CKKS_KeyGenerator_Delete(HE_CKKS_KeyGenerator* kg) {
    if (kg) { delete kg->cpp_keygen; delete kg; }
}

// --- Seed Configuration ---
void HEonGPU_CKKS_KeyGenerator_SetSeed(HE_CKKS_KeyGenerator* kg, int seed_c) {
    if (!kg || !kg->cpp_keygen || !seed_c) {
        std::cerr << "KeyGenerator_SetSeed: Invalid argument(s).\n"; return;
    }
    try {
        
        kg->cpp_keygen->set_seed(seed_c);
    } catch (const std::exception& e) { std::cerr << "KeyGenerator_SetSeed Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "KeyGenerator_SetSeed Unknown Error" << std::endl; }
}

// --- Standard Key Generation ---

int HEonGPU_CKKS_KeyGenerator_GenerateSecretKey(HE_CKKS_KeyGenerator* kg, HE_CKKS_SecretKey* sk_c, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !sk_c || !get_cpp_secretkey(sk_c)) {
        std::cerr << "GenerateSecretKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_secret_key(*(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "GenerateSecretKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "GenerateSecretKey Unknown Error" << std::endl; return -2; }
}

int HEonGPU_CKKS_KeyGenerator_GeneratePublicKey(HE_CKKS_KeyGenerator* kg,
                                                HE_CKKS_PublicKey* key_out_c,
                                                HE_CKKS_SecretKey* sk_c,
                                                const C_ExecutionOptions* options_c) {
    // Ensure you have non-const helpers like get_cpp_publickey and get_cpp_secretkey defined
    if (!kg || !kg->cpp_keygen || !key_out_c || !get_cpp_publickey(key_out_c) || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GeneratePublicKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        // The call uses a non-const pointer for sk_c, which becomes a non-const reference
        kg->cpp_keygen->generate_public_key(*(get_cpp_publickey(key_out_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GeneratePublicKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GeneratePublicKey" " Unknown Error" << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_KeyGenerator_GenerateRelinKey(HE_CKKS_KeyGenerator* kg,
                                               HE_CKKS_RelinKey* key_out_c,
                                               HE_CKKS_SecretKey* sk_c,
                                               const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !key_out_c || !get_cpp_relinkey(key_out_c) || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateRelinKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_relin_key(*(get_cpp_relinkey(key_out_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateRelinKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateRelinKey" " Unknown Error" << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_KeyGenerator_GenerateGaloisKey(HE_CKKS_KeyGenerator* kg,
                                                HE_CKKS_GaloisKey* key_out_c,
                                                HE_CKKS_SecretKey* sk_c,
                                                const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !key_out_c || !get_cpp_galoiskey(key_out_c) || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateGaloisKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_galois_key(*(get_cpp_galoiskey(key_out_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateGaloisKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateGaloisKey" " Unknown Error" << std::endl;
        return -2;
    }
}


// --- Multiparty Key Generation ---
int HEonGPU_CKKS_KeyGenerator_GenerateMultipartyPublicKey(HE_CKKS_KeyGenerator* kg,
                                                          HE_CKKS_MultipartyPublicKey* key_out_c,
                                                          HE_CKKS_SecretKey* sk_c,
                                                          const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !key_out_c || !get_cpp_mp_publickey(key_out_c) || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyPublicKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_multi_party_public_key_piece(*(get_cpp_mp_publickey(key_out_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyPublicKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyPublicKey" " Unknown Error" << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_KeyGenerator_GenerateMultipartyRelinKey(HE_CKKS_KeyGenerator* kg,
                                                         HE_CKKS_MultipartyRelinKey* key_out_c,
                                                         HE_CKKS_SecretKey* sk_c,
                                                         const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !key_out_c || !get_cpp_mp_relinkey(key_out_c) || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyRelinKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_multi_party_relin_key_piece(*(get_cpp_mp_relinkey(key_out_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyRelinKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyRelinKey" " Unknown Error" << std::endl;
        return -2;
    }
}


int HEonGPU_CKKS_KeyGenerator_GenerateMultipartyGaloisKey(HE_CKKS_KeyGenerator* kg,
                                                          HE_CKKS_MultipartyGaloisKey* key_out_c,
                                                          HE_CKKS_SecretKey* sk_c,
                                                          const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || !key_out_c || !key_out_c->cpp_mp_galoiskey || !get_cpp_secretkey(sk_c)) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyGaloisKey" " Error: Invalid argument(s).\n";
        return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_multi_party_galios_key_piece(*(key_out_c->cpp_mp_galoiskey), *(get_cpp_secretkey(sk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyGaloisKey" " Error: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_KeyGenerator_GenerateMultipartyGaloisKey" " Unknown Error" << std::endl;
        return -2;
    }
}
 /* C++ uses Galoiskey here */


// --- Multiparty Key Aggregation ---
int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyPublicKey(HE_CKKS_KeyGenerator* kg, const HE_CKKS_MultipartyPublicKey* const* public_keys_array_c, size_t num_public_keys, HE_CKKS_PublicKey* aggregated_pk_c, const C_ExecutionOptions* options_c) {
    if (!kg || !kg->cpp_keygen || (num_public_keys > 0 && !public_keys_array_c) || !aggregated_pk_c || !get_cpp_publickey(aggregated_pk_c)) {
        std::cerr << "AggregateMultipartyPublicKey Error: Invalid argument(s).\n"; return -1;
    }
    try {
        std::vector<heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>> cpp_pk_vec;
        cpp_pk_vec.reserve(num_public_keys);
        for (size_t i = 0; i < num_public_keys; ++i) {
            const heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* cpp_mp_pk = get_const_cpp_mp_publickey(public_keys_array_c[i]);
            if (!cpp_mp_pk) { std::cerr << "AggregateMultipartyPublicKey Error: Null key in array at index " << i << std::endl; return -1; }
            cpp_pk_vec.push_back(*cpp_mp_pk); // Makes a copy, C++ method takes vector of objects
        }
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
        kg->cpp_keygen->generate_multi_party_public_key(cpp_pk_vec, *(get_cpp_publickey(aggregated_pk_c)), cpp_options);
        return 0;
    } catch (const std::exception& e) { std::cerr << "AggregateMultipartyPublicKey Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "AggregateMultipartyPublicKey Unknown Error" << std::endl; return -2; }
}

//If necessary, will fix later
// int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyRelinKey(HE_CKKS_KeyGenerator* kg, HE_CKKS_SecretKey* sk_c, const HE_CKKS_MultipartyRelinKey* const* relin_keys_array_c, size_t num_relin_keys, HE_CKKS_RelinKey* aggregated_rlk_c, const C_ExecutionOptions* options_c) {
//     if (!kg || !kg->cpp_keygen || (num_relin_keys > 0 && !relin_keys_array_c) || !aggregated_rlk_c || !get_cpp_relinkey(aggregated_rlk_c)) {
//          std::cerr << "AggregateMultipartyRelinKey Error: Invalid argument(s).\n"; return -1;
//     }
//     try {
//         std::vector<heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>> cpp_rlk_vec;
//         cpp_rlk_vec.reserve(num_relin_keys);
//         for (size_t i = 0; i < num_relin_keys; ++i) {
//              const heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* cpp_mp_rlk = get_const_cpp_mp_relinkey(relin_keys_array_c[i]);
//              if (!cpp_mp_rlk) { std::cerr << "AggregateMultipartyRelinKey Error: Null key in array at index " << i << std::endl; return -1; }
//             cpp_rlk_vec.push_back(*cpp_mp_rlk);
//         }
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
//         kg->cpp_keygen->generate_multi_party_relin_key(cpp_rlk_vec, *(get_cpp_relinkey(aggregated_rlk_c)), *(get_cpp_secretkey(sk_c)), cpp_options);
//         return 0;
//     } catch (const std::exception& e) { std::cerr << "AggregateMultipartyRelinKey Error: " << e.what() << std::endl; return -2; }
//       catch (...) { std::cerr << "AggregateMultipartyRelinKey Unknown Error" << std::endl; return -2; }
// }

// int HEonGPU_CKKS_KeyGenerator_AggregateMultipartyGaloisKey(HE_CKKS_KeyGenerator* kg, const HE_CKKS_GaloisKey* const* galois_keys_array_c, size_t num_galois_keys, HE_CKKS_GaloisKey* aggregated_gk_c, const C_ExecutionOptions* options_c) {
//     if (!kg || !kg->cpp_keygen || (num_galois_keys > 0 && !galois_keys_array_c) || !aggregated_gk_c || !get_cpp_galoiskey(aggregated_gk_c) ) {
//         std::cerr << "AggregateMultipartyGaloisKey Error: Invalid argument(s).\n"; return -1;
//     }
//     try {
//         std::vector<heongpu::Galoiskey<heongpu::Scheme::CKKS>> cpp_gk_vec; // C++ takes vector of Galoiskey
//         cpp_gk_vec.reserve(num_galois_keys);
//         for (size_t i = 0; i < num_galois_keys; ++i) {
//             const heongpu::Galoiskey<heongpu::Scheme::CKKS>* cpp_gk = get_const_cpp_galoiskey(galois_keys_array_c[i]);
//             if (!cpp_gk) { std::cerr << "AggregateMultipartyGaloisKey Error: Null key in array at index " << i << std::endl; return -1; }
//             cpp_gk_vec.push_back(*cpp_gk);
//         }
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_kg(options_c);
//         kg->cpp_keygen->aggregate_multiparty_galoiskey(cpp_gk_vec, *(get_cpp_galoiskey(aggregated_gk_c)), cpp_options);
//         return 0;
//     } catch (const std::exception& e) { std::cerr << "AggregateMultipartyGaloisKey Error: " << e.what() << std::endl; return -2; }
//       catch (...) { std::cerr << "AggregateMultipartyGaloisKey Unknown Error" << std::endl; return -2; }
// }

} // extern "C"