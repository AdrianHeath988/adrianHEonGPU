#include "hip/hip_runtime.h"
#include "encoder_c_api.h"
#include "heongpu.cuh"
#include "heongpu_c_api_internal.h"
#include "ckks/context.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/encoder.cuh" // The C++ class we are wrapping
#include "hostvector.cuh"
#include "complex.cuh"   // For heongpu::Complex64
#include "schemes.h"     // For heongpu::Data64

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct to hold the actual C++ HEEncoder object

typedef struct HE_CKKS_Encoder_s HE_CKKS_Encoder;

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) { // Assuming cpp_context from context_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) { // Assuming cpp_plaintext from plaintext_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Plaintext pointer." << std::endl;
        return nullptr;
    }
    return pt->cpp_plaintext;
}
// Helper to map C_ExecutionOptions to heongpu::ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_enc(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Initializes with C++ defaults
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        // If C_STORAGE_TYPE_INVALID or other, it keeps the C++ default (DEVICE)
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS Encoder Lifecycle ---

HE_CKKS_Encoder* HEonGPU_CKKS_Encoder_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_encoder_obj =
            new (std::nothrow) heongpu::HEEncoder<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_encoder_obj) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C++ HEEncoder allocation failed." << std::endl;
            return nullptr;
        }

        HE_CKKS_Encoder* c_api_encoder = new (std::nothrow) HE_CKKS_Encoder_s;
        if (!c_api_encoder) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C API Encoder wrapper allocation failed." << std::endl;
            delete cpp_encoder_obj;
            return nullptr;
        }
        c_api_encoder->cpp_encoder = cpp_encoder_obj;
        return c_api_encoder;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Encoder_Delete(HE_CKKS_Encoder* encoder) {
    if (encoder) {
        delete encoder->cpp_encoder;
        delete encoder;
    }
}

// --- CKKS Encoding Functions ---

int HEonGPU_CKKS_Encoder_Encode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       const double* message_data,
                                       size_t message_len,
                                       double scale,
                                       const C_ExecutionOptions* c_options) { // Parameter name matches .h
    if (!encoder || !encoder->cpp_encoder || !pt || (message_len > 0 && !message_data)) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Double." << std::endl;
        return -1; // Error for invalid pointers or message data for non-zero length
    }

    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) {
        return -1; // Error due to invalid plaintext wrapper
    }

    try {
        std::vector<double> cpp_message;
        if (message_len > 0) {
            cpp_message.assign(message_data, message_data + message_len);
        }
        // Else, cpp_message remains empty, which is valid for some encode overloads
        // though the C++ function you provided takes const std::vector<double>& message,
        // so an empty vector will be passed if message_len is 0.

        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);

        // Call the C++ encode method that takes std::vector<double> and ExecutionOptions
        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_exec_options);
        
        return 0; // Success
    } catch (const std::invalid_argument& e) { // Catch specific known exceptions if possible
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed (invalid argument): " << e.what() << std::endl;
        return -3;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Encode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        const C_ComplexDouble* message_data,
                                        size_t message_len,
                                        double scale,
                                        const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_data) {
         std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        std::vector<Complex64> cpp_message(message_len);
        for (size_t i = 0; i < message_len; ++i) {
            cpp_message[i] = Complex64(message_data[i].real, message_data[i].imag);
        }
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_exec_options);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

// --- CKKS Decoding Functions ---

int HEonGPU_CKKS_Encoder_Decode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       double* message_buffer,
                                       size_t buffer_len,
                                       const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Double." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        heongpu::HostVector<double> cpp_message_vec; // HEEncoder::decode_ckks populates this
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->decode(cpp_message_vec, *cpp_pt, cpp_exec_options);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);

        if (elements_to_copy > 0) {
            std::memcpy(message_buffer, cpp_message_vec.data(), elements_to_copy * sizeof(double));
        }
        
        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Double buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Decode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        C_ComplexDouble* message_buffer,
                                        size_t buffer_len,
                                        const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        heongpu::HostVector<Complex64> cpp_message_vec;

        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->decode(cpp_message_vec, *cpp_pt, cpp_exec_options);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);

        for (size_t i = 0; i < elements_to_copy; ++i) {
            message_buffer[i].real = cpp_message_vec[i].real();
            message_buffer[i].imag = cpp_message_vec[i].imag();
        }
        
        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Complex buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

} // extern "C"