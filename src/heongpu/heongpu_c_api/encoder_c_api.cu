#include "hip/hip_runtime.h"
#include "encoder_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/encoder.cuh" // The C++ class we are wrapping
#include "hostvector.cuh"
#include "complex.cuh"   // For heongpu::Complex64
#include "schemes.h"     // For heongpu::Data64

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct to hold the actual C++ HEEncoder object

typedef struct HE_CKKS_Encoder_s HE_CKKS_Encoder;

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) { // Assuming cpp_context from context_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) { // Assuming cpp_plaintext from plaintext_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Plaintext pointer." << std::endl;
        return nullptr;
    }
    return pt->cpp_plaintext;
}


extern "C" {

// --- CKKS Encoder Lifecycle ---

HE_CKKS_Encoder* HEonGPU_CKKS_Encoder_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_encoder_obj =
            new (std::nothrow) heongpu::HEEncoder<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_encoder_obj) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C++ HEEncoder allocation failed." << std::endl;
            return nullptr;
        }

        HE_CKKS_Encoder* c_api_encoder = new (std::nothrow) HE_CKKS_Encoder_s;
        if (!c_api_encoder) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C API Encoder wrapper allocation failed." << std::endl;
            delete cpp_encoder_obj;
            return nullptr;
        }
        c_api_encoder->cpp_encoder = cpp_encoder_obj;
        return c_api_encoder;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Encoder_Delete(HE_CKKS_Encoder* encoder) {
    if (encoder) {
        delete encoder->cpp_encoder;
        delete encoder;
    }
}

// --- CKKS Encoding Functions ---

int HEonGPU_CKKS_Encoder_Encode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       const double* message_data,
                                       size_t message_len,
                                       double scale,
                                       C_cudaStream_t stream_in) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_data) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Double." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        // Convert C array to std::vector<double> or heongpu::HostVector<double>
        // std::vector is fine as HEEncoder has overloads for it.
        std::vector<double> cpp_message(message_data, message_data + message_len);
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_in);

        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Encode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        const C_ComplexDouble* message_data,
                                        size_t message_len,
                                        double scale,
                                        C_cudaStream_t stream_in) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_data) {
         std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        std::vector<heongpu::Complex64> cpp_message(message_len);
        for (size_t i = 0; i < message_len; ++i) {
            cpp_message[i] = heongpu::Complex64(message_data[i].real, message_data[i].imag);
        }
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_in);

        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

// --- CKKS Decoding Functions ---

int HEonGPU_CKKS_Encoder_Decode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       double* message_buffer,
                                       size_t buffer_len,
                                       C_cudaStream_t stream_in) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Double." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        heongpu::HostVector<double> cpp_message_vec; // HEEncoder::decode_ckks populates this
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_in);

        encoder->cpp_encoder->decode_ckks(cpp_message_vec, *cpp_pt, cpp_stream);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);

        if (elements_to_copy > 0) {
            std::memcpy(message_buffer, cpp_message_vec.data(), elements_to_copy * sizeof(double));
        }
        
        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Double buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Decode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        C_ComplexDouble* message_buffer,
                                        size_t buffer_len,
                                        C_cudaStream_t stream_in) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        heongpu::HostVector<heongpu::Complex64> cpp_message_vec;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_in);

        encoder->cpp_encoder->decode_ckks(cpp_message_vec, *cpp_pt, cpp_stream);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);

        for (size_t i = 0; i < elements_to_copy; ++i) {
            message_buffer[i].real = cpp_message_vec[i].real();
            message_buffer[i].imag = cpp_message_vec[i].imag();
        }
        
        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Complex buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

} // extern "C"