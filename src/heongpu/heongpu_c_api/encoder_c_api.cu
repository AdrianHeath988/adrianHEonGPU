#include "hip/hip_runtime.h"
#include "encoder_c_api.h"
#include "heongpu.cuh"
#include "heongpu_c_api_internal.h"
#include "ckks/context.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/encoder.cuh" // The C++ class we are wrapping
#include "hostvector.cuh"
#include "complex.cuh"   // For heongpu::Complex64
#include "schemes.h"     // For heongpu::Data64

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow
#include <hip/hip_runtime.h>
// Define the opaque struct to hold the actual C++ HEEncoder object

typedef struct HE_CKKS_Encoder_s HE_CKKS_Encoder;

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) { // Assuming cpp_context from context_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) { // Assuming cpp_plaintext from plaintext_c_api.cu
        std::cerr << "Error: Invalid HE_CKKS_Plaintext pointer." << std::endl;
        return nullptr;
    }
    return pt->cpp_plaintext;
}
// Helper to map C_ExecutionOptions to heongpu::ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_enc(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Initializes with C++ defaults
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        // If C_STORAGE_TYPE_INVALID or other, it keeps the C++ default (DEVICE)
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS Encoder Lifecycle ---

HE_CKKS_Encoder* HEonGPU_CKKS_Encoder_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_encoder_obj =
            new (std::nothrow) heongpu::HEEncoder<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_encoder_obj) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C++ HEEncoder allocation failed." << std::endl;
            return nullptr;
        }

        HE_CKKS_Encoder* c_api_encoder = new (std::nothrow) HE_CKKS_Encoder_s;
        if (!c_api_encoder) {
            std::cerr << "HEonGPU_CKKS_Encoder_Create failed: C API Encoder wrapper allocation failed." << std::endl;
            delete cpp_encoder_obj;
            return nullptr;
        }
        c_api_encoder->cpp_encoder = cpp_encoder_obj;
        return c_api_encoder;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Encoder_Delete(HE_CKKS_Encoder* encoder) {
    if (encoder) {
        delete encoder->cpp_encoder;
        delete encoder;
    }
}

// --- CKKS Encoding Functions ---

int HEonGPU_CKKS_Encoder_Encode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       const double* message_data,
                                       size_t message_len,
                                       double scale,
                                       const C_ExecutionOptions* c_options) { // Parameter name matches .h



    // std::cout << "--- Entering HEonGPU_CKKS_Encoder_Encode_Double ---" << std::endl;
    // std::cout << "  encoder: " << encoder << std::endl;
    // std::cout << "  pt: " << pt << std::endl;
    // std::cout << "  message_data address: " << message_data << std::endl;
    // std::cout << "  message_len: " << message_len << std::endl;
    // if (message_data && message_len > 0) {
    //     std::cout << "  message_data contents: [";
    //     for (size_t i = 0; i < 10; ++i) {
    //         std::cout << message_data[i] << (i == message_len - 1 ? "" : ", ");
    //     }
    //     std::cout << "]" << std::endl;
    // } else {
    //     std::cout << "  message_data contents: null or empty" << std::endl;
    // }
    // std::cout << "  scale: " << scale << std::endl;
    // std::cout << "  c_options: " << c_options << std::endl;
    // std::cout << "--------------------------------------------------" << std::endl;


    if (!encoder || !encoder->cpp_encoder || !pt || (message_len > 0 && !message_data)) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Double." << std::endl;
        return -1; // Error for invalid pointers or message data for non-zero length
    }

    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) {
        return -1; // Error due to invalid plaintext wrapper
    }

    try {
        std::vector<double> cpp_message;
        if (message_len > 0) {
            cpp_message.assign(message_data, message_data + message_len);
        }
        // Else, cpp_message remains empty, which is valid for some encode overloads
        // though the C++ function you provided takes const std::vector<double>& message,
        // so an empty vector will be passed if message_len is 0.

        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        
        // Call the C++ encode method that takes std::vector<double> and ExecutionOptions
        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_exec_options);
        
        

        return 0; // Success
    } catch (const std::invalid_argument& e) { // Catch specific known exceptions if possible
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed (invalid argument): " << e.what() << std::endl;
        return -3;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Encode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        const C_ComplexDouble* message_data,
                                        size_t message_len,
                                        double scale,
                                        const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_data) {
         std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Encode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        std::vector<Complex64> cpp_message(message_len);
        for (size_t i = 0; i < message_len; ++i) {
            cpp_message[i] = Complex64(message_data[i].real, message_data[i].imag);
        }
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->encode(*cpp_pt, cpp_message, scale, cpp_exec_options);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Encode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

// --- CKKS Decoding Functions ---

int HEonGPU_CKKS_Encoder_Decode_Double(HE_CKKS_Encoder* encoder,
                                       HE_CKKS_Plaintext* pt,
                                       double* message_buffer,
                                       size_t buffer_len,
                                       const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Double." << std::endl;
        return -1; // Error
    }
    HE_CKKS_Plaintext* pt_clone = HEonGPU_CKKS_Plaintext_Clone(pt);
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt_clone);
    if (!cpp_pt) return -1;

    try {
        // std::cout << "--- Inspecting Raw Plaintext Data ---" << std::endl;
        // uint64_t* device_ptr = HEonGPU_CKKS_Plaintext_GetData(pt_clone);

        // if (!device_ptr) {
        //     std::cerr << "Failed to get plaintext data pointer." << std::endl;
        // }


        // const int values_to_print = 10;
        // std::vector<uint64_t> host_buffer(values_to_print);
        // hipError_t cuda_status = hipMemcpy(
        //     host_buffer.data(),                        // Destination (CPU buffer)
        //     device_ptr,                                // Source (GPU pointer)
        //     values_to_print * sizeof(uint64_t),        // Total bytes to copy
        //     hipMemcpyDeviceToHost                     // Direction of copy
        // );
        // if (cuda_status == hipSuccess) {
        //     std::cout << "First 10 raw encoded values (from GPU):" << std::endl;
        //     for (int i = 0; i < values_to_print; ++i) {
        //         std::cout << "Value " << i << ": " << host_buffer[i] << std::endl;
        //     }
        // } else {
        //     std::cerr << "hipMemcpy failed: " << hipGetErrorString(cuda_status) << std::endl;
        // }

        // std::cout << "------------------------------------" << std::endl;
        // std::cout << "Plaintext Size: " << HEonGPU_CKKS_Plaintext_GetPlainSize(pt_clone) << std::endl;
        // std::cout << "Plaintext Depth: " << HEonGPU_CKKS_Plaintext_GetDepth(pt_clone) << std::endl;
        // std::cout << "Plaintext Scale: " << HEonGPU_CKKS_Plaintext_GetScale(pt_clone) << std::endl;
        // std::cout << "Plaintext is in NTT Domain: " << std::boolalpha << HEonGPU_CKKS_Plaintext_IsInNttDomain(pt_clone) << std::endl;
        // std::cout << "Plaintext is on Device: " << std::boolalpha << HEonGPU_CKKS_Plaintext_IsOnDevice(pt_clone) << std::endl;











        std::vector<double> cpp_message_vec; // HEEncoder::decode_ckks populates this
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->decode(cpp_message_vec, *cpp_pt, cpp_exec_options);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);
        // std::cout << "--- Debugging Post-Decode ---" << std::endl;
        
        // std::cout << "  Decoded vector size (decoded_len): " << decoded_len << std::endl;
        // std::cout << "  encoder: " << encoder << std::endl;
        // std::cout << "  pt_clone: " << pt_clone << std::endl;
        // std::cout << "  pt: " << pt << std::endl;
        // std::cout << "  Underlying cpp_pt address: " << cpp_pt << std::endl;
        // if (decoded_len > 0) {
        //     std::cout << "  Contents of cpp_message_vec: [";
            
        //     for (int i = 0; i < 10; ++i) {
        //         std::cout << cpp_message_vec[i] << (i == 10 - 1 ? "" : ", ");
        //     }
        //     if (decoded_len > 10) {
        //         std::cout << "...";
        //     }
        //     std::cout << "]" << std::endl;
        // } else {
        //     std::cout << "  cpp_message_vec is empty." << std::endl;
        // }
        // std::cout << "  Destination buffer capacity (buffer_len): " << buffer_len << std::endl;
        // std::cout << "  Elements we will copy: " << elements_to_copy << std::endl;
        // std::cout << "  Destination buffer address (message_buffer): " << (void*)message_buffer << std::endl;
        // std::cout << "  Source vector data address: " << (void*)cpp_message_vec.data() << std::endl;
        


        if (elements_to_copy > 0) {
            std::memcpy(message_buffer, cpp_message_vec.data(), elements_to_copy * sizeof(double));
        }

        // std::cout << "  Check result DECODE:" << std::endl;
        // std::cout <<"  total elements is: " << elements_to_copy << std::endl;
        // std::cout << "  And the buffer is:" <<std::endl;
        // for (int i=0;i<10;i++) {
        //     std::cout << message_buffer[i] << " ";
        // }
        // std::cout << std::endl;
        // std::cout << "-----------------------------" << std::endl;
        

        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Double buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Double failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Encoder_Decode_Complex(HE_CKKS_Encoder* encoder,
                                        HE_CKKS_Plaintext* pt,
                                        C_ComplexDouble* message_buffer,
                                        size_t buffer_len,
                                        const C_ExecutionOptions* c_options) {
    if (!encoder || !encoder->cpp_encoder || !pt || !message_buffer) {
        std::cerr << "Error: Invalid argument(s) to HEonGPU_CKKS_Encoder_Decode_Complex." << std::endl;
        return -1; // Error
    }
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = get_cpp_plaintext(pt);
    if (!cpp_pt) return -1;

    try {
        heongpu::HostVector<Complex64> cpp_message_vec;

        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_enc(c_options);
        encoder->cpp_encoder->decode(cpp_message_vec, *cpp_pt, cpp_exec_options);

        size_t decoded_len = cpp_message_vec.size();
        size_t elements_to_copy = std::min(buffer_len, decoded_len);

        for (size_t i = 0; i < elements_to_copy; ++i) {
            message_buffer[i].real = cpp_message_vec[i].real();
            message_buffer[i].imag = cpp_message_vec[i].imag();
        }
        
        if (buffer_len < decoded_len) {
            std::cerr << "Warning: Decode_Complex buffer was smaller than decoded message. Truncated." << std::endl;
        }
        return static_cast<int>(elements_to_copy);
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Encoder_Decode_Complex failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

} // extern "C"