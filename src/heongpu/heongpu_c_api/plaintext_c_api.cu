#include "hip/hip_runtime.h"
#include "plaintext_c_api.h"
#include "heongpu.cuh"
#include "ckks/context.cuh"
#include "ckks/plaintext.cuh"
#include "util/hostvector.cuh"
#include "util/schemes.h"      // For heongpu::scheme_type, heongpu::Data64
#include "util/storagemanager.cuh" // For heongpu::storage_type, heongpu::ExecutionOptions

#include <vector>
#include <sstream>
#include <iostream>
#include <algorithm> // For std::min
#include <cstring>   // For std::memcpy
#include <new>       // For std::nothrow


typedef struct HE_CKKS_Plaintext_s HE_CKKS_Plaintext;

// Helper to safely access the underlying C++ HEContext pointer
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer provided." << std::endl;
        return nullptr;
    }
    return context->cpp_context; // cpp_context is the member name
}

// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options;
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else { // Default to DEVICE if C_STORAGE_TYPE_DEVICE or invalid
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    // If c_options is null, cpp_options uses its default members (hipStreamDefault, DEVICE, true)
    return cpp_options;
}

// Helper to map C++ scheme_type to C_scheme_type
static C_scheme_type map_cpp_to_c_scheme_type(heongpu::scheme_type cpp_type) {
    switch (cpp_type) {
        case heongpu::scheme_type::none: return C_SCHEME_TYPE_NONE;
        case heongpu::scheme_type::bfv:  return C_SCHEME_TYPE_BFV;
        case heongpu::scheme_type::ckks: return C_SCHEME_TYPE_CKKS;
        case heongpu::scheme_type::bgv:  return C_SCHEME_TYPE_BGV;
        default: return static_cast<C_scheme_type>(-1); // Error/Unknown
    }
}

// Helper to map C++ storage_type to C_storage_type
static C_storage_type map_cpp_to_c_storage_type(heongpu::storage_type cpp_type) {
    switch (cpp_type) {
        case heongpu::storage_type::HOST:   return C_STORAGE_TYPE_HOST;
        case heongpu::storage_type::DEVICE: return C_STORAGE_TYPE_DEVICE;
        default: return C_STORAGE_TYPE_INVALID;
    }
}


extern "C" {

// --- Lifecycle & Serialization ---

HE_CKKS_Plaintext* HEonGPU_CKKS_Plaintext_Create(HE_CKKS_Context* context,
                                                 const C_ExecutionOptions* options) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options(options);
        heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt =
            new (std::nothrow) heongpu::Plaintext<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_exec_options);
        if (!cpp_pt) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Create failed: C++ Plaintext allocation failed." << std::endl;
            return nullptr;
        }
        HE_CKKS_Plaintext* c_api_plaintext = new (std::nothrow) HE_CKKS_Plaintext_s;
        if (!c_api_plaintext) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Create failed: C API Plaintext wrapper allocation failed." << std::endl;
            delete cpp_pt;
            return nullptr;
        }
        c_api_plaintext->cpp_plaintext = cpp_pt;
        return c_api_plaintext;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Plaintext_Delete(HE_CKKS_Plaintext* plaintext) {
    if (plaintext) {
        delete plaintext->cpp_plaintext;
        delete plaintext;
    }
}

HE_CKKS_Plaintext* HEonGPU_CKKS_Plaintext_Clone(const HE_CKKS_Plaintext* other_plaintext) {
    if (!other_plaintext || !other_plaintext->cpp_plaintext) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Clone failed: Invalid source plaintext pointer." << std::endl;
        return nullptr;
    }
    try {
        heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_cloned_pt =
            new (std::nothrow) heongpu::Plaintext<heongpu::Scheme::CKKS>(*(other_plaintext->cpp_plaintext));
        if(!cpp_cloned_pt) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Clone failed: C++ Plaintext (clone) allocation failed." << std::endl;
            return nullptr;
        }
        HE_CKKS_Plaintext* c_api_cloned_plaintext = new (std::nothrow) HE_CKKS_Plaintext_s;
        if (!c_api_cloned_plaintext) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Clone failed: C API Plaintext wrapper (clone) allocation failed." << std::endl;
            delete cpp_cloned_pt;
            return nullptr;
        }
        c_api_cloned_plaintext->cpp_plaintext = cpp_cloned_pt;
        return c_api_cloned_plaintext;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Clone failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Clone failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

int HEonGPU_CKKS_Plaintext_Assign_Copy(HE_CKKS_Plaintext* dest_plaintext,
                                       const HE_CKKS_Plaintext* src_plaintext) {
    if (!dest_plaintext || !dest_plaintext->cpp_plaintext ||
        !src_plaintext || !src_plaintext->cpp_plaintext) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Assign_Copy failed: Invalid destination or source plaintext pointer." << std::endl;
        return -1; 
    }
    try {
        *(dest_plaintext->cpp_plaintext) = *(src_plaintext->cpp_plaintext);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Assign_Copy failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Assign_Copy failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Plaintext_Save(HE_CKKS_Plaintext* plaintext,
                                unsigned char** out_bytes,
                                size_t* out_len) {
    if (!plaintext || !plaintext->cpp_plaintext || !out_bytes || !out_len) {
        if(out_bytes) *out_bytes = nullptr;
        if(out_len) *out_len = 0;
        return -1; 
    }
    *out_bytes = nullptr;
    *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        plaintext->cpp_plaintext->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len == 0) {
            *out_bytes = nullptr; 
            return 0;
        }
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len)); 
        if (!(*out_bytes)) {
            *out_len = 0;
            std::cerr << "HEonGPU_CKKS_Plaintext_Save failed: Memory allocation error." << std::endl;
            return -2;
        }
        std::memcpy(*out_bytes, str_data.data(), *out_len);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Save failed with C++ exception: " << e.what() << std::endl;
        if (*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Save failed due to an unknown C++ exception." << std::endl;
        if (*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    }
}

HE_CKKS_Plaintext* HEonGPU_CKKS_Plaintext_Load(HE_CKKS_Context* context,
                                               const unsigned char* bytes,
                                               size_t len,
                                               const C_ExecutionOptions* options) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Load failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    if (!bytes && len > 0) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Load failed: Invalid bytes pointer for non-zero length." << std::endl;
        return nullptr;
    }

    HE_CKKS_Plaintext* c_api_plaintext = nullptr;
    heongpu::Plaintext<heongpu::Scheme::CKKS>* cpp_pt = nullptr;
    try {
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options(options);
        // First, create a Plaintext object using the constructor that takes ExecutionOptions
        cpp_pt = new (std::nothrow) heongpu::Plaintext<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_exec_options);
        if (!cpp_pt) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Load failed: C++ Plaintext allocation failed." << std::endl;
            return nullptr;
        }

        if (len > 0 && bytes) { // Only load if there's data
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_pt->load(iss);
        }
        // If len is 0, cpp_pt is a newly constructed plaintext with default options.

        c_api_plaintext = new (std::nothrow) HE_CKKS_Plaintext_s;
        if (!c_api_plaintext) {
            std::cerr << "HEonGPU_CKKS_Plaintext_Load failed: C API Plaintext wrapper allocation failed." << std::endl;
            delete cpp_pt;
            return nullptr;
        }
        c_api_plaintext->cpp_plaintext = cpp_pt;
        return c_api_plaintext;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Load failed with C++ exception: " << e.what() << std::endl;
        delete cpp_pt;
        delete c_api_plaintext;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_Load failed due to an unknown C++ exception." << std::endl;
        delete cpp_pt;
        delete c_api_plaintext;
        return nullptr;
    }
}


// --- CKKS Plaintext Getters ---

C_scheme_type HEonGPU_CKKS_Plaintext_GetScheme(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in GetScheme." << std::endl;
        return static_cast<C_scheme_type>(-1); // Error
    }
    try {
        return map_cpp_to_c_scheme_type(plaintext->cpp_plaintext->get_scheme());
    } catch (...) { return static_cast<C_scheme_type>(-1); }
}

int HEonGPU_CKKS_Plaintext_GetPlainSize(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in GetPlainSize." << std::endl;
        return 0;
    }
    try {
        return plaintext->cpp_plaintext->plain_size();
    } catch (...) { return 0; }
}

int HEonGPU_CKKS_Plaintext_GetDepth(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in GetDepth." << std::endl;
        return 0; // Or an error indicator like -1 if 0 can be a valid depth
    }
    try {
        return plaintext->cpp_plaintext->depth();
    } catch (...) { return 0; }
}

double HEonGPU_CKKS_Plaintext_GetScale(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in GetScale." << std::endl;
        return -1.0; // Error indicator
    }
    try {
        return plaintext->cpp_plaintext->get_scale();
    } catch (...) { return -1.0; }
}

bool HEonGPU_CKKS_Plaintext_IsInNttDomain(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in IsInNttDomain." << std::endl;
        return false;
    }
    try {
        return plaintext->cpp_plaintext->is_in_ntt_domain();
    } catch (...) { return false; }
}

C_storage_type HEonGPU_CKKS_Plaintext_GetStorageType(HE_CKKS_Plaintext* plaintext) {
    if (!plaintext || !plaintext->cpp_plaintext) {
        std::cerr << "Error: Invalid plaintext pointer in GetStorageType." << std::endl;
        return C_STORAGE_TYPE_INVALID; 
    }
    try {
        return map_cpp_to_c_storage_type(plaintext->cpp_plaintext->get_storage_type());
    } catch (...) { return C_STORAGE_TYPE_INVALID; }
}

size_t HEonGPU_CKKS_Plaintext_GetData(HE_CKKS_Plaintext* plaintext,
                                      uint64_t* data_buffer, // heongpu::Data64 is uint64_t
                                      size_t buffer_elements,
                                      C_cudaStream_t stream) {
    if (!plaintext || !plaintext->cpp_plaintext || (!data_buffer && buffer_elements > 0)) {
        std::cerr << "Error: Invalid arguments in Plaintext GetData." << std::endl;
        return 0;
    }
    try {
        heongpu::HostVector<heongpu::Data64> temp_host_vector;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        
        plaintext->cpp_plaintext->get_data(temp_host_vector, cpp_stream);

        size_t elements_in_pt = temp_host_vector.size();
        size_t elements_to_copy = std::min(buffer_elements, elements_in_pt);

        if (elements_to_copy > 0 && data_buffer) {
            std::memcpy(data_buffer, temp_host_vector.data(), elements_to_copy * sizeof(heongpu::Data64));
        }
        return elements_to_copy;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_GetData failed with C++ exception: " << e.what() << std::endl;
        return 0;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_GetData failed due to an unknown C++ exception." << std::endl;
        return 0;
    }
}

// --- CKKS Plaintext Setters ---
int HEonGPU_CKKS_Plaintext_SetData(HE_CKKS_Plaintext* plaintext,
                                   const uint64_t* data_buffer, // heongpu::Data64 is uint64_t
                                   size_t num_elements,
                                   C_cudaStream_t stream) {
    if (!plaintext || !plaintext->cpp_plaintext || (!data_buffer && num_elements > 0)) {
        std::cerr << "Error: Invalid arguments in Plaintext SetData." << std::endl;
        return -1; // Error
    }
    try {
        // Create a HostVector from the C buffer.
        // Note: This makes a copy. If Plaintext::set_data takes a const ref
        // and potentially copies internally, this is fine. If it expects to take
        // ownership or avoid a copy, the C++ API would need to reflect that.
        heongpu::HostVector<heongpu::Data64> input_host_vector(num_elements);
        if (num_elements > 0 && data_buffer) {
            std::memcpy(input_host_vector.data(), data_buffer, num_elements * sizeof(heongpu::Data64));
        }
        
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        plaintext->cpp_plaintext->set_data(input_host_vector, cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Plaintext_SetData failed with C++ exception: " << e.what() << std::endl;
        return -2; // Error
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Plaintext_SetData failed due to an unknown C++ exception." << std::endl;
        return -2; // Error
    }
}

} // extern "C"