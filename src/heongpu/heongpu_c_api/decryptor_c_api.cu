#include "decryptor_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/secretkey.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/ciphertext.cuh"
#include "ckks/decryptor.cuh" // The C++ class we are wrapping

#include "util/storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct
struct HE_CKKS_Decryptor_s {
    heongpu::HEDecryptor<heongpu::Scheme::CKKS>* cpp_decryptor;
};

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context_dec(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr; // Assuming cpp_context from context_c_api.cu
    return context->cpp_context;
}
static heongpu::SecretKey<heongpu::Scheme::CKKS>* get_cpp_secretkey_dec(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr; // Assuming cpp_secretkey from secretkey_c_api.cu
    return sk->cpp_secretkey;
}
static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext_dec(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr; // Assuming cpp_plaintext from plaintext_c_api.cu
    return pt->cpp_plaintext;
}
static heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_cpp_ciphertext_dec(HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr; // Assuming cpp_ciphertext from ciphertext_c_api.cu
    return ct->cpp_ciphertext;
}
static const heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_const_cpp_ciphertext_dec(const HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}


// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_dec(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEDecryptor Lifecycle ---

HE_CKKS_Decryptor* HEonGPU_CKKS_Decryptor_Create(HE_CKKS_Context* context, HE_CKKS_SecretKey* sk) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_dec(context);
    heongpu::SecretKey<heongpu::Scheme::CKKS>* cpp_sk = get_cpp_secretkey_dec(sk);
    if (!cpp_h_context || !cpp_sk) {
        std::cerr << "Decryptor_Create: Invalid context or secret key." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEDecryptor<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_sk);
        if (!cpp_obj) { std::cerr << "Decryptor_Create: C++ allocation failed.\n"; return nullptr; }
        auto c_api_obj = new (std::nothrow) HE_CKKS_Decryptor_s;
        if (!c_api_obj) { delete cpp_obj; std::cerr << "Decryptor_Create: C API wrapper allocation failed.\n"; return nullptr; }
        c_api_obj->cpp_decryptor = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "Decryptor_Create Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Decryptor_Create Unknown Error" << std::endl; return nullptr; }
}

void HEonGPU_CKKS_Decryptor_Delete(HE_CKKS_Decryptor* decryptor) {
    if (decryptor) {
        delete decryptor->cpp_decryptor;
        delete decryptor;
    }
}

// --- CKKS Decryption Functions ---

int HEonGPU_CKKS_Decryptor_Decrypt(HE_CKKS_Decryptor* decryptor,
                                   HE_CKKS_Plaintext* pt_out_c,
                                   HE_CKKS_Ciphertext* ct_in_c,
                                   const C_ExecutionOptions* options_c) {
    if (!decryptor || !decryptor->cpp_decryptor || !pt_out_c || !get_cpp_plaintext_dec(pt_out_c) || !ct_in_c || !get_cpp_ciphertext_dec(ct_in_c)) {
        std::cerr << "Decrypt: Invalid argument(s).\n"; return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_dec(options_c);
        decryptor->cpp_decryptor->decrypt(*(get_cpp_plaintext_dec(pt_out_c)), *(get_cpp_ciphertext_dec(ct_in_c)), cpp_options);
        return 0; // Success
    } catch (const std::exception& e) { std::cerr << "Decrypt Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "Decrypt Unknown Error" << std::endl; return -2; }
}

// --- Noise Budget Calculation ---

double HEonGPU_CKKS_Decryptor_CalculateNoiseBudget(HE_CKKS_Decryptor* decryptor,
                                                   HE_CKKS_Ciphertext* ct_c,
                                                   const C_ExecutionOptions* options_c) {
    if (!decryptor || !decryptor->cpp_decryptor || !ct_c || !get_cpp_ciphertext_dec(ct_c)) {
        std::cerr << "CalculateNoiseBudget: Invalid argument(s).\n"; return -1.0; // Error indicator
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_dec(options_c);
        return decryptor->cpp_decryptor->calculate_noise_budget(*(get_cpp_ciphertext_dec(ct_c)), cpp_options);
    } catch (const std::exception& e) { std::cerr << "CalculateNoiseBudget Error: " << e.what() << std::endl; return -1.0; }
      catch (...) { std::cerr << "CalculateNoiseBudget Unknown Error" << std::endl; return -1.0; }
}

// --- Multiparty Decryption Functions ---

int HEonGPU_CKKS_Decryptor_PartialDecrypt(HE_CKKS_Decryptor* decryptor,
                                          HE_CKKS_Plaintext* partial_pt_out_c,
                                          HE_CKKS_Ciphertext* ct_in_c,
                                          C_cudaStream_t stream_c) {
    if (!decryptor || !decryptor->cpp_decryptor || !partial_pt_out_c || !get_cpp_plaintext_dec(partial_pt_out_c) || !ct_in_c || !get_cpp_ciphertext_dec(ct_in_c)) {
        std::cerr << "PartialDecrypt: Invalid argument(s).\n"; return -1;
    }
    try {
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_c);
        decryptor->cpp_decryptor->partial_decrypt(*(get_cpp_plaintext_dec(partial_pt_out_c)), *(get_cpp_ciphertext_dec(ct_in_c)), cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) { std::cerr << "PartialDecrypt Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "PartialDecrypt Unknown Error" << std::endl; return -2; }
}

int HEonGPU_CKKS_Decryptor_DecryptFusion(HE_CKKS_Decryptor* decryptor,
                                         const HE_CKKS_Ciphertext* const* partial_cts_array_c,
                                         size_t num_partial_cts,
                                         HE_CKKS_Plaintext* final_pt_out_c,
                                         C_cudaStream_t stream_c) {
    if (!decryptor || !decryptor->cpp_decryptor || (num_partial_cts > 0 && !partial_cts_array_c) || !final_pt_out_c || !get_cpp_plaintext_dec(final_pt_out_c)) {
        std::cerr << "DecryptFusion: Invalid argument(s).\n"; return -1;
    }
    try {
        std::vector<heongpu::Ciphertext<heongpu::Scheme::CKKS>> cpp_partial_cts_vec;
        cpp_partial_cts_vec.reserve(num_partial_cts);
        for (size_t i = 0; i < num_partial_cts; ++i) {
            const heongpu::Ciphertext<heongpu::Scheme::CKKS>* cpp_ct = get_const_cpp_ciphertext_dec(partial_cts_array_c[i]);
            if (!cpp_ct) {
                std::cerr << "DecryptFusion Error: Null ciphertext in array at index " << i << std::endl; return -1;
            }
            cpp_partial_cts_vec.push_back(*cpp_ct); // Makes a copy for the vector
        }
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream_c);
        decryptor->cpp_decryptor->decrypt_fusion_ckks(cpp_partial_cts_vec, *(get_cpp_plaintext_dec(final_pt_out_c)), cpp_stream);
        return 0; // Success
    } catch (const std::exception& e) { std::cerr << "DecryptFusion Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "DecryptFusion Unknown Error" << std::endl; return -2; }
}

// --- CKKS Decryptor Seed/Offset Management ---
// These are identical to Encryptor's; good candidates for a shared utility if more PRNGs were wrapped

int HEonGPU_CKKS_Decryptor_GetSeed(HE_CKKS_Decryptor* decryptor) {
    if (!decryptor || !decryptor->cpp_decryptor) {
        std::cerr << "GetSeed: Invalid decryptor pointer.\n"; return -1; 
    }
    try { return decryptor->cpp_decryptor->get_seed(); } 
    catch (...) { return -1; }
}

void HEonGPU_CKKS_Decryptor_SetSeed(HE_CKKS_Decryptor* decryptor, int new_seed) {
    if (!decryptor || !decryptor->cpp_decryptor) {
        std::cerr << "SetSeed: Invalid decryptor pointer.\n"; return;
    }
    try { decryptor->cpp_decryptor->set_seed(new_seed); } 
    catch (const std::exception& e) { std::cerr << "SetSeed Error: " << e.what() << std::endl;}
    catch (...) { std::cerr << "SetSeed Unknown Error" << std::endl;}
}

int HEonGPU_CKKS_Decryptor_GetOffset(HE_CKKS_Decryptor* decryptor) {
    if (!decryptor || !decryptor->cpp_decryptor) {
        std::cerr << "GetOffset: Invalid decryptor pointer.\n"; return -1; 
    }
    try { return decryptor->cpp_decryptor->get_offset(); } 
    catch (...) { return -1; }
}

void HEonGPU_CKKS_Decryptor_SetOffset(HE_CKKS_Decryptor* decryptor, int new_offset) {
    if (!decryptor || !decryptor->cpp_decryptor) {
        std::cerr << "SetOffset: Invalid decryptor pointer.\n"; return;
    }
    try { decryptor->cpp_decryptor->set_offset(new_offset); } 
    catch (const std::exception& e) { std::cerr << "SetOffset Error: " << e.what() << std::endl;}
    catch (...) { std::cerr << "SetOffset Unknown Error" << std::endl;}
}

} // extern "C"