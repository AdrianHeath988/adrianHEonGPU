#include "hip/hip_runtime.h"
#include "publickey_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/publickey.cuh" // The C++ class we are wrapping
#include "util/hostvector.cuh"
#include "util/schemes.h"
#include "util/storagemanager.cuh"
#include "util/random.cuh"     // For heongpu::RNGSeed

#include <vector>
#include <sstream>
#include <iostream>
#include <algorithm> // For std::min
#include <cstring>   // For std::memcpy
#include <new>       // For std::nothrow


// Define the opaque structs
struct HE_CKKS_PublicKey_s {
    heongpu::Publickey<heongpu::Scheme::CKKS>* cpp_publickey;
};

struct HE_CKKS_MultipartyPublicKey_s {
    heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* cpp_mp_publickey;
};

// Helper to safely access underlying C++ HEContext pointer
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

// Helper C++ enums to C enums 
static C_scheme_type map_cpp_to_c_scheme_type_pk(heongpu::scheme_type cpp_type) {
    switch (cpp_type) {
        case heongpu::scheme_type::none: return C_SCHEME_TYPE_NONE;
        case heongpu::scheme_type::bfv:  return C_SCHEME_TYPE_BFV;
        case heongpu::scheme_type::ckks: return C_SCHEME_TYPE_CKKS;
        case heongpu::scheme_type::bgv:  return C_SCHEME_TYPE_BGV;
        default: return static_cast<C_scheme_type>(-1); 
    }
}

static C_storage_type map_cpp_to_c_storage_type_pk(heongpu::storage_type cpp_type) {
    switch (cpp_type) {
        case heongpu::storage_type::HOST:   return C_STORAGE_TYPE_HOST;
        case heongpu::storage_type::DEVICE: return C_STORAGE_TYPE_DEVICE;
        default: return C_STORAGE_TYPE_INVALID;
    }
}





extern "C" {

// --- CKKS PublicKey Functions ---

HE_CKKS_PublicKey* HEonGPU_CKKS_PublicKey_Create(HE_CKKS_Context* context) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_PublicKey_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }
    try {
        heongpu::Publickey<heongpu::Scheme::CKKS>* cpp_pk_obj =
            new (std::nothrow) heongpu::Publickey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_pk_obj) {
            std::cerr << "HEonGPU_CKKS_PublicKey_Create failed: C++ Publickey allocation failed." << std::endl;
            return nullptr;
        }
        HE_CKKS_PublicKey* c_api_pk = new (std::nothrow) HE_CKKS_PublicKey_s;
        if (!c_api_pk) {
            std::cerr << "HEonGPU_CKKS_PublicKey_Create failed: C API Publickey wrapper allocation failed." << std::endl;
            delete cpp_pk_obj;
            return nullptr;
        }
        c_api_pk->cpp_publickey = cpp_pk_obj;
        return c_api_pk;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_PublicKey_Create failed: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_PublicKey_Create failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_PublicKey_Delete(HE_CKKS_PublicKey* pk) {
    if (pk) {
        delete pk->cpp_publickey;
        delete pk;
    }
}

HE_CKKS_PublicKey* HEonGPU_CKKS_PublicKey_Clone(const HE_CKKS_PublicKey* other_pk) {
    if (!other_pk || !other_pk->cpp_publickey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::Publickey<heongpu::Scheme::CKKS>(*(other_pk->cpp_publickey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_PublicKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_publickey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_PublicKey_Assign_Copy(HE_CKKS_PublicKey* dest_pk, const HE_CKKS_PublicKey* src_pk) {
    if (!dest_pk || !dest_pk->cpp_publickey || !src_pk || !src_pk->cpp_publickey) return -1;
    try {
        *(dest_pk->cpp_publickey) = *(src_pk->cpp_publickey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_PublicKey_Save(HE_CKKS_PublicKey* pk, unsigned char** out_bytes, size_t* out_len) {
    if (!pk || !pk->cpp_publickey || !out_bytes || !out_len) {
        if(out_bytes) *out_bytes = nullptr;
        if(out_len) *out_len = 0;
        return -1;
    }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        pk->cpp_publickey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len == 0) { *out_bytes = nullptr; return 0; }
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) { *out_len = 0; return -2; }
        std::memcpy(*out_bytes, str_data.data(), *out_len);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "PK Save failed: " << e.what() << std::endl;
        if(*out_bytes) { free(*out_bytes); *out_bytes = nullptr; } *out_len = 0; return -3;
    } catch (...) {
        if(*out_bytes) { free(*out_bytes); *out_bytes = nullptr; } *out_len = 0; return -3;
    }
}

HE_CKKS_PublicKey* HEonGPU_CKKS_PublicKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;

    heongpu::Publickey<heongpu::Scheme::CKKS>* cpp_pk = nullptr;
    HE_CKKS_PublicKey* c_api_pk = nullptr;
    try {
        cpp_pk = new (std::nothrow) heongpu::Publickey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_pk) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_pk->load(iss);
        }
        c_api_pk = new (std::nothrow) HE_CKKS_PublicKey_s;
        if (!c_api_pk) { delete cpp_pk; return nullptr; }
        c_api_pk->cpp_publickey = cpp_pk;
        return c_api_pk;
    } catch (const std::exception& e) {
        std::cerr << "PK Load failed: " << e.what() << std::endl;
        delete cpp_pk; delete c_api_pk; return nullptr;
    } catch (...) {
        delete cpp_pk; delete c_api_pk; return nullptr;
    }
}

// Getters for PublicKey
C_scheme_type HEonGPU_CKKS_PublicKey_GetScheme(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_pk(pk->cpp_publickey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}
int HEonGPU_CKKS_PublicKey_GetRingSize(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return 0;
    try { return pk->cpp_publickey->ring_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_PublicKey_GetCoeffModulusCount(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return 0;
    try { return pk->cpp_publickey->coeff_modulus_count(); } catch (...) { return 0; }
}
bool HEonGPU_CKKS_PublicKey_IsInNttDomain(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return false;
    try { return pk->cpp_publickey->is_in_ntt_domain(); } catch (...) { return false; }
}
bool HEonGPU_CKKS_PublicKey_IsGenerated(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return false;
    try { return pk->cpp_publickey->is_generated(); } catch (...) { return false; }
}
C_storage_type HEonGPU_CKKS_PublicKey_GetStorageType(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_pk(pk->cpp_publickey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}

size_t HEonGPU_CKKS_PublicKey_GetData(HE_CKKS_PublicKey* pk, uint64_t* data_buffer, size_t buffer_elements, C_cudaStream_t stream) {
    if (!pk || !pk->cpp_publickey || (!data_buffer && buffer_elements > 0)) return 0;
    try {
        heongpu::HostVector<heongpu::Data64> temp_host_vector;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        pk->cpp_publickey->get_data(temp_host_vector, cpp_stream);
        size_t elements_to_copy = std::min(buffer_elements, temp_host_vector.size());
        if (elements_to_copy > 0 && data_buffer) {
            std::memcpy(data_buffer, temp_host_vector.data(), elements_to_copy * sizeof(heongpu::Data64));
        }
        return elements_to_copy;
    } catch (...) { return 0; }
}

// Setter for PublicKey
int HEonGPU_CKKS_PublicKey_SetData(HE_CKKS_PublicKey* pk, const uint64_t* data_buffer, size_t num_elements, C_cudaStream_t stream) {
    if (!pk || !pk->cpp_publickey || (!data_buffer && num_elements > 0)) return -1;
    try {
        heongpu::HostVector<heongpu::Data64> input_hv(num_elements);
        if (num_elements > 0 && data_buffer) {
            std::memcpy(input_hv.data(), data_buffer, num_elements * sizeof(heongpu::Data64));
        }
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        pk->cpp_publickey->set_data(input_hv, cpp_stream);
        return 0;
    } catch (...) { return -2; }
}


// --- CKKS MultipartyPublicKey Functions ---

HE_CKKS_MultipartyPublicKey* HEonGPU_CKKS_MultipartyPublicKey_Create(HE_CKKS_Context* context, const C_RNGSeed_Const_Data* seed_c_data) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!seed_c_data) return nullptr;

    try {
        heongpu::RNGSeed cpp_seed;
        if (seed_c_data->key_data && seed_c_data->key_len > 0) {
            cpp_seed.key_.assign(seed_c_data->key_data, seed_c_data->key_data + seed_c_data->key_len);
        }
        if (seed_c_data->nonce_data && seed_c_data->nonce_len > 0) {
            cpp_seed.nonce_.assign(seed_c_data->nonce_data, seed_c_data->nonce_data + seed_c_data->nonce_len);
        }
        if (seed_c_data->pstring_data && seed_c_data->pstring_len > 0) {
            cpp_seed.personalization_string_.assign(seed_c_data->pstring_data, seed_c_data->pstring_data + seed_c_data->pstring_len);
        }

        heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* cpp_mp_pk_obj =
            new (std::nothrow) heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_seed);
        if (!cpp_mp_pk_obj) return nullptr;

        HE_CKKS_MultipartyPublicKey* c_api_mp_pk = new (std::nothrow) HE_CKKS_MultipartyPublicKey_s;
        if (!c_api_mp_pk) { delete cpp_mp_pk_obj; return nullptr; }
        c_api_mp_pk->cpp_mp_publickey = cpp_mp_pk_obj;
        return c_api_mp_pk;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_MultipartyPublicKey_Delete(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (mp_pk) {
        delete mp_pk->cpp_mp_publickey;
        delete mp_pk;
    }
}

HE_CKKS_MultipartyPublicKey* HEonGPU_CKKS_MultipartyPublicKey_Clone(const HE_CKKS_MultipartyPublicKey* other_mp_pk) {
    if (!other_mp_pk || !other_mp_pk->cpp_mp_publickey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>(*(other_mp_pk->cpp_mp_publickey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_MultipartyPublicKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_mp_publickey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_MultipartyPublicKey_Assign_Copy(HE_CKKS_MultipartyPublicKey* dest_mp_pk, const HE_CKKS_MultipartyPublicKey* src_mp_pk) {
    if (!dest_mp_pk || !dest_mp_pk->cpp_mp_publickey || !src_mp_pk || !src_mp_pk->cpp_mp_publickey) return -1;
    try {
        *(dest_mp_pk->cpp_mp_publickey) = *(src_mp_pk->cpp_mp_publickey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_MultipartyPublicKey_Save(HE_CKKS_MultipartyPublicKey* mp_pk, unsigned char** out_bytes, size_t* out_len) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey || !out_bytes || !out_len) {
        if(out_bytes) *out_bytes = nullptr;
        if(out_len) *out_len = 0;
        return -1;
    }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        mp_pk->cpp_mp_publickey->save(oss); // MultipartyPublickey::save should handle base and seed
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len == 0) { *out_bytes = nullptr; return 0; }
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) { *out_len = 0; return -2; }
        std::memcpy(*out_bytes, str_data.data(), *out_len);
        return 0;
    } catch (...) { if(*out_bytes) {free(*out_bytes); *out_bytes = nullptr;} *out_len = 0; return -3; }
}

HE_CKKS_MultipartyPublicKey* HEonGPU_CKKS_MultipartyPublicKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, const C_RNGSeed_Const_Data* seed_for_reconstruction) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;

    // The C++ MultipartyPublickey::load might not need the seed explicitly if it's part of the serialized data.
    // However, its constructor needs a seed. If the seed IS part of the serialized data,
    // the load method handles it. If not, the C API user might need to provide it if load doesn't
    // restore it or if the object must be fully valid post-construction before load.
    // The current C++ MultipartyPublickey constructor requires a seed.
    // Let's assume for load, we construct with a temporary/default seed, then load populates it.
    // Or, the seed_for_reconstruction is used if the C++ load method doesn't restore the seed.
    // The C++ save/load in publickey.cu for MultipartyPublickey handles the seed.

    // TODO: Look more in depth to the load method/orion API to identify if this interpretation is correct.

    heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>* cpp_mp_pk = nullptr;
    HE_CKKS_MultipartyPublicKey* c_api_mp_pk = nullptr;
    try {
        // Default/dummy seed for initial construction before load
        heongpu::RNGSeed temp_seed; 
        if (seed_for_reconstruction && seed_for_reconstruction->key_data) { // Prefer provided seed if available
             if (seed_for_reconstruction->key_data && seed_for_reconstruction->key_len > 0) {
                temp_seed.key_.assign(seed_for_reconstruction->key_data, seed_for_reconstruction->key_data + seed_for_reconstruction->key_len);
            }
            if (seed_for_reconstruction->nonce_data && seed_for_reconstruction->nonce_len > 0) {
                temp_seed.nonce_.assign(seed_for_reconstruction->nonce_data, seed_for_reconstruction->nonce_data + seed_for_reconstruction->nonce_len);
            }
            if (seed_for_reconstruction->pstring_data && seed_for_reconstruction->pstring_len > 0) {
                temp_seed.personalization_string_.assign(seed_for_reconstruction->pstring_data, seed_for_reconstruction->pstring_data + seed_for_reconstruction->pstring_len);
            }
        }


        cpp_mp_pk = new (std::nothrow) heongpu::MultipartyPublickey<heongpu::Scheme::CKKS>(*cpp_h_context, temp_seed);
        if (!cpp_mp_pk) return nullptr;

        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_mp_pk->load(iss); // This should load base class data AND the seed
        }
        
        c_api_mp_pk = new (std::nothrow) HE_CKKS_MultipartyPublicKey_s;
        if (!c_api_mp_pk) { delete cpp_mp_pk; return nullptr; }
        c_api_mp_pk->cpp_mp_publickey = cpp_mp_pk;
        return c_api_mp_pk;
    } catch (...) { delete cpp_mp_pk; delete c_api_mp_pk; return nullptr; }
}


// Getters for MultipartyPublickey
C_scheme_type HEonGPU_CKKS_MultipartyPublicKey_GetScheme(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_pk(mp_pk->cpp_mp_publickey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}
int HEonGPU_CKKS_MultipartyPublicKey_GetRingSize(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return 0;
    try { return mp_pk->cpp_mp_publickey->ring_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_MultipartyPublicKey_GetCoeffModulusCount(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return 0;
    try { return mp_pk->cpp_mp_publickey->coeff_modulus_count(); } catch (...) { return 0; }
}
bool HEonGPU_CKKS_MultipartyPublicKey_IsInNttDomain(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return false;
    try { return mp_pk->cpp_mp_publickey->is_in_ntt_domain(); } catch (...) { return false; }
}
bool HEonGPU_CKKS_MultipartyPublicKey_IsGenerated(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return false;
    try { return mp_pk->cpp_mp_publickey->is_generated(); } catch (...) { return false; }
}
C_storage_type HEonGPU_CKKS_MultipartyPublicKey_GetStorageType(HE_CKKS_MultipartyPublicKey* mp_pk) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_pk(mp_pk->cpp_mp_publickey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}

size_t HEonGPU_CKKS_MultipartyPublicKey_GetData(HE_CKKS_MultipartyPublicKey* mp_pk, uint64_t* data_buffer, size_t buffer_elements, C_cudaStream_t stream) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey || (!data_buffer && buffer_elements > 0)) return 0;
    try {
        heongpu::HostVector<heongpu::Data64> temp_host_vector;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        mp_pk->cpp_mp_publickey->get_data(temp_host_vector, cpp_stream);
        size_t elements_to_copy = std::min(buffer_elements, temp_host_vector.size());
        if (elements_to_copy > 0 && data_buffer) {
            std::memcpy(data_buffer, temp_host_vector.data(), elements_to_copy * sizeof(heongpu::Data64));
        }
        return elements_to_copy;
    } catch (...) { return 0; }
}

int HEonGPU_CKKS_MultipartyPublicKey_GetSeed(HE_CKKS_MultipartyPublicKey* mp_pk, C_RNGSeed_Data* out_seed_data) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey || !out_seed_data) return -1;
    
    // Initialize output struct
    out_seed_data->key_data = nullptr; out_seed_data->key_len = 0;
    out_seed_data->nonce_data = nullptr; out_seed_data->nonce_len = 0;
    out_seed_data->pstring_data = nullptr; out_seed_data->pstring_len = 0;

    try {
        const heongpu::RNGSeed& cpp_seed = mp_pk->cpp_mp_publickey->seed();

        if (!cpp_seed.key_.empty()) {
            out_seed_data->key_len = cpp_seed.key_.size();
            out_seed_data->key_data = static_cast<unsigned char*>(malloc(out_seed_data->key_len));
            if (!out_seed_data->key_data) { HEonGPU_Free_C_RNGSeed_Data_Members(out_seed_data); return -2; }
            std::memcpy(out_seed_data->key_data, cpp_seed.key_.data(), out_seed_data->key_len);
        }
        if (!cpp_seed.nonce_.empty()) {
            out_seed_data->nonce_len = cpp_seed.nonce_.size();
            out_seed_data->nonce_data = static_cast<unsigned char*>(malloc(out_seed_data->nonce_len));
            if (!out_seed_data->nonce_data) { HEonGPU_Free_C_RNGSeed_Data_Members(out_seed_data); return -2; }
            std::memcpy(out_seed_data->nonce_data, cpp_seed.nonce_.data(), out_seed_data->nonce_len);
        }
        if (!cpp_seed.personalization_string_.empty()) {
            out_seed_data->pstring_len = cpp_seed.personalization_string_.size();
            out_seed_data->pstring_data = static_cast<unsigned char*>(malloc(out_seed_data->pstring_len));
            if (!out_seed_data->pstring_data) { HEonGPU_Free_C_RNGSeed_Data_Members(out_seed_data); return -2; }
            std::memcpy(out_seed_data->pstring_data, cpp_seed.personalization_string_.data(), out_seed_data->pstring_len);
        }
        return 0; // Success
    } catch (...) { 
        HEonGPU_Free_C_RNGSeed_Data_Members(out_seed_data); // Clean up partially allocated memory on error
        return -3; 
    }
}

// Setter for MultipartyPublickey (same as PublicKey as seed is set at construction)
int HEonGPU_CKKS_MultipartyPublicKey_SetData(HE_CKKS_MultipartyPublicKey* mp_pk, const uint64_t* data_buffer, size_t num_elements, C_cudaStream_t stream) {
    if (!mp_pk || !mp_pk->cpp_mp_publickey || (!data_buffer && num_elements > 0)) return -1;
    try {
        heongpu::HostVector<heongpu::Data64> input_hv(num_elements);
        if (num_elements > 0 && data_buffer) {
            std::memcpy(input_hv.data(), data_buffer, num_elements * sizeof(heongpu::Data64));
        }
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        mp_pk->cpp_mp_publickey->set_data(input_hv, cpp_stream); // Calls base class set_data
        return 0;
    } catch (...) { return -2; }
}


} // extern "C"