#include "hip/hip_runtime.h"
#include "ciphertext_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh"
#include "memorypool.cuh"

#include "ckks/context.cuh"
#include "ckks/ciphertext.cuh"
#include "hostvector.cuh" // For heongpu::HostVector
#include "schemes.h"      // For heongpu::Data64 (uint64_t)
#include "storagemanager.cuh" // For heongpu::storage_type

#include <vector>
#include <sstream>
#include <iostream>
#include <algorithm>
#include <cstring>
#include <new>

typedef struct HE_CKKS_Ciphertext_s HE_CKKS_Ciphertext;

static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer provided." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

// Helper to map C++ storage_type to C_storage_type
static C_storage_type map_cpp_to_c_storage_type(heongpu::storage_type cpp_type) {
    switch (cpp_type) {
        case heongpu::storage_type::HOST:   return C_STORAGE_TYPE_HOST;
        case heongpu::storage_type::DEVICE: return C_STORAGE_TYPE_DEVICE;
        default:
            // Should not happen with a valid C++ enum
            return static_cast<C_storage_type>(-1); // Indicate error/unknown
    }
}
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context_from_opaque_ct(HE_CKKS_Context* context_c_api_ptr) {
    if (!context_c_api_ptr || !context_c_api_ptr->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context_c_api_ptr->cpp_context;
}
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_ct(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        } else {
            // Keep default or handle C_STORAGE_TYPE_INVALID if it's a possible input
            cpp_options.storage_ = heongpu::storage_type::DEVICE; // Defaulting to DEVICE
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}

extern "C" {

// --- Lifecycle & Serialization (from previous version, with minor safety improvements) ---

HE_CKKS_Ciphertext* HEonGPU_CKKS_Ciphertext_Create(HE_CKKS_Context* context_c_api_ptr,
                                                   const C_ExecutionOptions* options_c) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_from_opaque_ct(context_c_api_ptr);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Create failed: HE_CKKS_Context is null or invalid." << std::endl;
        return nullptr;
    }

    try {
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_ct(options_c);
        heongpu::Ciphertext<heongpu::Scheme::CKKS>* cpp_ct =
            new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_exec_options);
        
        if (!cpp_ct) {
            std::cerr << "HEonGPU_CKKS_Ciphertext_Create failed: C++ Ciphertext allocation failed." << std::endl;
            return nullptr;
        }

        HE_CKKS_Ciphertext* c_api_ciphertext = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_ciphertext) {
            std::cerr << "HEonGPU_CKKS_Ciphertext_Create failed: C API Ciphertext wrapper allocation failed." << std::endl;
            delete cpp_ct; 
            return nullptr;
        }
        c_api_ciphertext->cpp_ciphertext = cpp_ct;
        return c_api_ciphertext;

    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}


void HEonGPU_CKKS_Ciphertext_Delete(HE_CKKS_Ciphertext* ciphertext) {
    // This function should ONLY delete the C++ object pointed to by the wrapper.
    // The C-style wrapper struct (ciphertext) itself will be freed by the
    // calling language's runtime (Python's garbage collector in this case).
    if (ciphertext && ciphertext->cpp_ciphertext) {

        // std::cout <<"[C++ Debug] Before CipherText Deletion"<<std::endl;
        // MemoryPool::instance().print_memory_pool_status();

        hipStream_t stream = ciphertext->cpp_ciphertext->stream();
        ciphertext->cpp_ciphertext->memory_clear(stream);
        delete ciphertext->cpp_ciphertext;
        ciphertext->cpp_ciphertext = nullptr;
        HEonGPU_SynchronizeDevice();
        // std::cout <<"[C++ Debug] After CipherText Deletion"<<std::endl;
        // MemoryPool::instance().print_memory_pool_status();

    }
}


HE_CKKS_Ciphertext* HEonGPU_CKKS_Ciphertext_Clone(const HE_CKKS_Ciphertext* other_ciphertext) {
    if (!other_ciphertext || !other_ciphertext->cpp_ciphertext) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Clone failed: Invalid source ciphertext pointer." << std::endl;
        return nullptr;
    }
    try {
        heongpu::Ciphertext<heongpu::Scheme::CKKS>* cpp_cloned_ct =
            new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(*(other_ciphertext->cpp_ciphertext));
        if (!cpp_cloned_ct) {
            std::cerr << "HEonGPU_CKKS_Ciphertext_Clone failed: C++ Ciphertext (clone) allocation failed." << std::endl;
            return nullptr;
        }
        HE_CKKS_Ciphertext* c_api_cloned_ciphertext = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_cloned_ciphertext) {
            std::cerr << "HEonGPU_CKKS_Ciphertext_Clone failed: C API Ciphertext wrapper (clone) allocation failed." << std::endl;
            delete cpp_cloned_ct;
            return nullptr;
        }
        c_api_cloned_ciphertext->cpp_ciphertext = cpp_cloned_ct;
        return c_api_cloned_ciphertext;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Clone failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Clone failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

int HEonGPU_CKKS_Ciphertext_Assign_Copy(HE_CKKS_Ciphertext* dest_ciphertext,
                                        const HE_CKKS_Ciphertext* src_ciphertext) {
    if (!dest_ciphertext || !dest_ciphertext->cpp_ciphertext ||
        !src_ciphertext || !src_ciphertext->cpp_ciphertext) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Assign_Copy failed: Invalid destination or source ciphertext pointer." << std::endl;
        return -1; 
    }
    try {
        *(dest_ciphertext->cpp_ciphertext) = *(src_ciphertext->cpp_ciphertext);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Assign_Copy failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Assign_Copy failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_Ciphertext_Save(HE_CKKS_Ciphertext* ciphertext,
                                 unsigned char** out_bytes,
                                 size_t* out_len) {
    if (!ciphertext || !ciphertext->cpp_ciphertext || !out_bytes || !out_len) {
        if (out_bytes) *out_bytes = nullptr;
        if (out_len) *out_len = 0;
        return -1; 
    }
    *out_bytes = nullptr; 
    *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        ciphertext->cpp_ciphertext->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len == 0) {
             *out_bytes = nullptr; 
             return 0; 
        }
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) {
            *out_len = 0;
            std::cerr << "HEonGPU_CKKS_Ciphertext_Save failed: Memory allocation error." << std::endl;
            return -2;
        }
        std::memcpy(*out_bytes, str_data.data(), *out_len);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Save failed with C++ exception: " << e.what() << std::endl;
        if (*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Save failed due to an unknown C++ exception." << std::endl;
        if (*out_bytes) { free(*out_bytes); *out_bytes = nullptr; }
        *out_len = 0;
        return -3;
    }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_Ciphertext_Load(HE_CKKS_Context* context_c_api_ptr,
                                                 const unsigned char* bytes,
                                                 size_t len,
                                                 const C_ExecutionOptions* options_c) { // CHANGED
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_from_opaque_ct(context_c_api_ptr);
    if (!cpp_h_context) { return nullptr; }
    if (!bytes && len > 0) { return nullptr; }

    HE_CKKS_Ciphertext* c_api_ciphertext = nullptr;
    heongpu::Ciphertext<heongpu::Scheme::CKKS>* cpp_ct = nullptr;
    try {
        heongpu::ExecutionOptions cpp_exec_options = map_c_to_cpp_execution_options_ct(options_c);
        cpp_ct = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_exec_options); // CHANGED CALL
        if (!cpp_ct) { return nullptr; }

        if (len > 0 && bytes) { 
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_ct->load(iss); 
        }
        
        c_api_ciphertext = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_ciphertext) { delete cpp_ct; return nullptr; }
        c_api_ciphertext->cpp_ciphertext = cpp_ct;
        return c_api_ciphertext;
    } catch (...) { delete cpp_ct; delete c_api_ciphertext; return nullptr; }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_Ciphertext_Set_Scale(HE_CKKS_Ciphertext* ciphertext, double scale){
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in HEonGPU_CKKS_Ciphertext_Set_Scale." << std::endl;
        return 0; 
    }
    try {
        ciphertext->cpp_ciphertext->set_scale(scale);
        std::cout << "Scale has been set to " << scale << std::endl;
        return ciphertext;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Set_Scale failed with C++ exception: " << e.what() << std::endl;
        return 0; // Or error indicator
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_Set_Scale failed due to an unknown C++ exception." << std::endl;
        return 0; // Or error indicator
    }
}
// --- CKKS Ciphertext Getters ---

int HEonGPU_CKKS_Ciphertext_GetRingSize(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetRingSize." << std::endl;
        return 0; // 0 is not a valid ring size (indicates error)
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline int ring_size() const noexcept { return ring_size_; }
        return ciphertext->cpp_ciphertext->ring_size();
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetRingSize failed with C++ exception: " << e.what() << std::endl;
        return 0; // Or error indicator
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetRingSize failed due to an unknown C++ exception." << std::endl;
        return 0; // Or error indicator
    }
}

int HEonGPU_CKKS_Ciphertext_GetCoeffModulusCount(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetCoeffModulusCount." << std::endl;
        return 0; 
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline int coeff_modulus_count() const noexcept { return coeff_modulus_count_; }
        return ciphertext->cpp_ciphertext->coeff_modulus_count();
    } catch (...) { return 0; } // Simplified error handling for getters
}

int HEonGPU_CKKS_Ciphertext_GetCiphertextSize(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetCiphertextSize." << std::endl;
        return 0;
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline int cipher_size() const noexcept { return cipher_size_; }
        return ciphertext->cpp_ciphertext->size();
    } catch (...) { return 0; }
}

double HEonGPU_CKKS_Ciphertext_GetScale(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetScale." << std::endl;
        return -1.0; // Error indicator
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline double get_scale() const noexcept { return scale_; }
        return ciphertext->cpp_ciphertext->scale();
    } catch (...) { return -1.0; }
}

bool HEonGPU_CKKS_Ciphertext_IsInNttDomain(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in IsInNttDomain." << std::endl;
        return false; // Default / error
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline bool is_in_ntt_domain() const noexcept { return in_ntt_domain_; }
        return ciphertext->cpp_ciphertext->in_ntt_domain();
    } catch (...) { return false; }
}
int HEonGPU_CKKS_Ciphertext_GetDepth(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetDepth." << std::endl;
        return 0;
    }
    try {
        return ciphertext->cpp_ciphertext->depth();
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetDepth failed with C++ exception: " << e.what() << std::endl;
        return 0;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetDepth failed due to an unknown C++ exception." << std::endl;
        return 0;
    }
}

bool HEonGPU_CKKS_Ciphertext_Is_On_Device(HE_CKKS_Ciphertext* ciphertext) {
    if (!ciphertext || !ciphertext->cpp_ciphertext) {
        std::cerr << "Error: Invalid ciphertext pointer in GetStorageType." << std::endl;
        return static_cast<C_storage_type>(-1); // Error indicator
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // inline heongpu::storage_type get_storage_type() const noexcept { return storage_type_; }
        return ciphertext->cpp_ciphertext->is_on_device();
    } catch (...) { return static_cast<C_storage_type>(-1); }
}

size_t HEonGPU_CKKS_Ciphertext_GetData(HE_CKKS_Ciphertext* ciphertext,
                                       uint64_t* data_buffer, // C++ Data64 is uint64_t
                                       size_t buffer_elements,
                                       C_cudaStream_t stream) {
    if (!ciphertext || !ciphertext->cpp_ciphertext || !data_buffer) {
        std::cerr << "Error: Invalid arguments in GetData." << std::endl;
        return 0;
    }
    try {
        // Ciphertext<CKKS> has a public method like:
        // void get_data(HostVector<Data64>& cipher_coeffs_data_on_host, hipStream_t stream = hipStreamDefault);
        // This C++ method populates a HostVector, For C, we fill a user-provided buffer
        // The C++ get_data would likely handle copying from device to a temporary HostVector if needed

        // Create a temporary C++ HostVector to receive the data.
        heongpu::HostVector<Data64> temp_host_vector;
        
        // Call the C++ method.
        // The C++ method itself needs to be able to determine how many elements to copy based on its internal state
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        ciphertext->cpp_ciphertext->get_data(temp_host_vector, cpp_stream);

        // Copy from the temporary HostVector to the user's buffer.
        size_t elements_in_ct = temp_host_vector.size();
        size_t elements_to_copy = std::min(buffer_elements, elements_in_ct);

        if (elements_to_copy > 0) {
            std::memcpy(data_buffer, temp_host_vector.data(), elements_to_copy * sizeof(Data64));
        }
        
        // If buffer_elements < elements_in_ct, it's a partial copy
        // return how many were copied
        return elements_to_copy;

    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetData failed with C++ exception: " << e.what() << std::endl;
        return 0;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Ciphertext_GetData failed due to an unknown C++ exception." << std::endl;
        return 0;
    }
}


} // extern "C"