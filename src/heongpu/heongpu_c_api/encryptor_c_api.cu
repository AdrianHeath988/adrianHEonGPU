#include "encryptor_c_api.h"
#include "heongpu.cuh"
#include "heongpu_c_api_internal.h"
#include "ckks/context.cuh"
#include "ckks/publickey.cuh"
#include "ckks/secretkey.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/ciphertext.cuh"
#include "ckks/encryptor.cuh" // The C++ class we are wrapping

#include "storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow

// Define the opaque struct


typedef struct HE_CKKS_Encryptor_s HE_CKKS_Encryptor;

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context_enc(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr; // Assuming cpp_context from context_c_api.cu
    return context->cpp_context;
}
static heongpu::Publickey<heongpu::Scheme::CKKS>* get_cpp_publickey_enc(HE_CKKS_PublicKey* pk) {
    if (!pk || !pk->cpp_publickey) return nullptr; // Assuming cpp_publickey from publickey_c_api.cu
    return pk->cpp_publickey;
}
static heongpu::Secretkey<heongpu::Scheme::CKKS>* get_cpp_secretkey_enc(HE_CKKS_SecretKey* sk) {
    if (!sk || !sk->cpp_secretkey) return nullptr; // Assuming cpp_secretkey from secretkey_c_api.cu
    return sk->cpp_secretkey;
}
static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext_enc(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr; // Assuming cpp_plaintext from plaintext_c_api.cu
    return pt->cpp_plaintext;
}
static heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_cpp_ciphertext_enc(HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr; // Assuming cpp_ciphertext from ciphertext_c_api.cu
    return ct->cpp_ciphertext;
}

// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_enc(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}

extern "C" {

// --- CKKS HEEncryptor Lifecycle ---

HE_CKKS_Encryptor* HEonGPU_CKKS_Encryptor_Create_With_PublicKey(HE_CKKS_Context* context,
                                                                HE_CKKS_PublicKey* pk) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_enc(context);
    heongpu::Publickey<heongpu::Scheme::CKKS>* cpp_pk = get_cpp_publickey_enc(pk);
    if (!cpp_h_context || !cpp_pk) {
        std::cerr << "Encryptor_Create_With_PublicKey: Invalid context or public key." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEEncryptor<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_pk);
        if (!cpp_obj) { std::cerr << "Encryptor_Create_With_PublicKey: C++ allocation failed.\n"; return nullptr; }
        auto c_api_obj = new (std::nothrow) HE_CKKS_Encryptor_s;
        if (!c_api_obj) { delete cpp_obj; std::cerr << "Encryptor_Create_With_PublicKey: C API wrapper allocation failed.\n"; return nullptr; }
        c_api_obj->cpp_encryptor = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "Encryptor_Create_With_PublicKey Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Encryptor_Create_With_PublicKey Unknown Error" << std::endl; return nullptr; }
}

// HE_CKKS_Encryptor* HEonGPU_CKKS_Encryptor_Create_With_SecretKey(HE_CKKS_Context* context,
//                                                                 HE_CKKS_SecretKey* sk) {
//     heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context_enc(context);
//     heongpu::Secretkey<heongpu::Scheme::CKKS>* cpp_sk = get_cpp_secretkey_enc(sk);
//     if (!cpp_h_context || !cpp_sk) {
//         std::cerr << "Encryptor_Create_With_SecretKey: Invalid context or secret key." << std::endl;
//         return nullptr;
//     }
//     try {
//         auto cpp_obj = new (std::nothrow) heongpu::HEEncryptor<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_sk);
//         if (!cpp_obj) { std::cerr << "Encryptor_Create_With_SecretKey: C++ allocation failed.\n"; return nullptr; }
//         auto c_api_obj = new (std::nothrow) HE_CKKS_Encryptor_s;
//         if (!c_api_obj) { delete cpp_obj; std::cerr << "Encryptor_Create_With_SecretKey: C API wrapper allocation failed.\n"; return nullptr; }
//         c_api_obj->cpp_encryptor = cpp_obj;
//         return c_api_obj;
//     } catch (const std::exception& e) { std::cerr << "Encryptor_Create_With_SecretKey Error: " << e.what() << std::endl; return nullptr; }
//       catch (...) { std::cerr << "Encryptor_Create_With_SecretKey Unknown Error" << std::endl; return nullptr; }
// }

void HEonGPU_CKKS_Encryptor_Delete(HE_CKKS_Encryptor* encryptor) {
    if (encryptor) {
        delete encryptor->cpp_encryptor;
        delete encryptor;
    }
}

// --- CKKS Encryption Functions ---

int HEonGPU_CKKS_Encryptor_Encrypt_To(HE_CKKS_Encryptor* encryptor,
                                      HE_CKKS_Ciphertext* ct_out_c,
                                      HE_CKKS_Plaintext* pt_in_c,
                                      const C_ExecutionOptions* options_c) {
    if (!encryptor || !encryptor->cpp_encryptor || !ct_out_c || !get_cpp_ciphertext_enc(ct_out_c) || !pt_in_c || !get_cpp_plaintext_enc(pt_in_c)) {
        std::cerr << "Encrypt_To: Invalid argument(s).\n"; return -1;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_enc(options_c);
        encryptor->cpp_encryptor->encrypt(*(get_cpp_ciphertext_enc(ct_out_c)), *(get_cpp_plaintext_enc(pt_in_c)), cpp_options);
        return 0; // Success
    } catch (const std::exception& e) { std::cerr << "Encrypt_To Error: " << e.what() << std::endl; return -2; }
      catch (...) { std::cerr << "Encrypt_To Unknown Error" << std::endl; return -2; }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_Encryptor_Encrypt_New(HE_CKKS_Encryptor* encryptor,
                                                       HE_CKKS_Plaintext* pt_in_c,
                                                       const C_ExecutionOptions* options_c) {
    if (!encryptor || !encryptor->cpp_encryptor || !pt_in_c || !get_cpp_plaintext_enc(pt_in_c)) {
        std::cerr << "Encrypt_New: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_enc(options_c);
        heongpu::Ciphertext<heongpu::Scheme::CKKS> cpp_result_ct;
        heongpu::Plaintext<heongpu::Scheme::CKKS> cpp_plaintext = *(get_cpp_plaintext_enc(pt_in_c));
        encryptor->cpp_encryptor->encrypt(cpp_result_ct, cpp_plaintext, cpp_options);

        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result_ct));
        if (!cpp_heap_result) { std::cerr << "Encrypt_New: C++ result allocation failed.\n"; return nullptr; }

        HE_CKKS_Ciphertext* c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; std::cerr << "Encrypt_New: C API wrapper allocation failed.\n"; return nullptr; }
        
        c_api_result->cpp_ciphertext = cpp_heap_result; // Assuming cpp_ciphertext member in HE_CKKS_Ciphertext_s
        return c_api_result;
    } catch (const std::exception& e) { std::cerr << "Encrypt_New Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Encrypt_New Unknown Error" << std::endl; return nullptr; }
}

// --- CKKS Encryptor Seed/Offset Management ---

int HEonGPU_CKKS_Encryptor_GetSeed(HE_CKKS_Encryptor* encryptor) {
    if (!encryptor || !encryptor->cpp_encryptor) {
        std::cerr << "GetSeed: Invalid encryptor pointer.\n"; return -1; // Or some other error indicator
    }
    try {
        return encryptor->cpp_encryptor->get_seed();
    } catch (...) { return -1; } // Should not throw if getter is noexcept
}

void HEonGPU_CKKS_Encryptor_SetSeed(HE_CKKS_Encryptor* encryptor, int new_seed) {
    if (!encryptor || !encryptor->cpp_encryptor) {
        std::cerr << "SetSeed: Invalid encryptor pointer.\n"; return;
    }
    try {
        encryptor->cpp_encryptor->set_seed(new_seed);
    } catch (const std::exception& e) { std::cerr << "SetSeed Error: " << e.what() << std::endl;}
      catch (...) { std::cerr << "SetSeed Unknown Error" << std::endl;}
}

int HEonGPU_CKKS_Encryptor_GetOffset(HE_CKKS_Encryptor* encryptor) {
    if (!encryptor || !encryptor->cpp_encryptor) {
        std::cerr << "GetOffset: Invalid encryptor pointer.\n"; return -1; // Or some other error indicator
    }
    try {
        return encryptor->cpp_encryptor->get_offset();
    } catch (...) { return -1; }
}

void HEonGPU_CKKS_Encryptor_SetOffset(HE_CKKS_Encryptor* encryptor, int new_offset) {
    if (!encryptor || !encryptor->cpp_encryptor) {
        std::cerr << "SetOffset: Invalid encryptor pointer.\n"; return;
    }
    try {
        encryptor->cpp_encryptor->set_offset(new_offset);
    } catch (const std::exception& e) { std::cerr << "SetOffset Error: " << e.what() << std::endl;}
      catch (...) { std::cerr << "SetOffset Unknown Error" << std::endl;}
}

} // extern "C"