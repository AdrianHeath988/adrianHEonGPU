#include "hip/hip_runtime.h"
#include "context_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh" // Main HEonGPU include for HEContext, Scheme, types
#include "schemes.h"      // For heongpu::Scheme, heongpu::keyswitching_type
#include "secstdparams.h" // For heongpu::sec_level_type
#include "hostvector.cuh" // For heongpu::HostVector
#include "util.cuh"       // For heongpu::Data128, heongpu::Modulus64 (via modular_arith.cuh)
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream> // For potential error logging (stderr)
#include <sstream>  // For serialization to/from memory buffer
#include <algorithm> // For std::copy, std::min

// Define the opaque struct to hold the actual C++ HEContext object
typedef struct HE_CKKS_Context_s HE_CKKS_Context;

// Helper to map C enum to C++ enum for keyswitching_type
static heongpu::keyswitching_type map_c_keyswitch_type(C_keyswitching_type c_type) {
    switch (c_type) {
        case C_KEYSWITCHING_METHOD_I: return heongpu::keyswitching_type::KEYSWITCHING_METHOD_I;
        case C_KEYSWITCHING_METHOD_II: return heongpu::keyswitching_type::KEYSWITCHING_METHOD_II;
        default:
            // Handle error or default, though C enum should prevent invalid values if used correctly
            return heongpu::keyswitching_type::KEYSWITCHING_METHOD_I; // Fallback, consider error
    }
}

// Helper to map C enum to C++ enum for sec_level_type
static heongpu::sec_level_type map_c_sec_level(C_sec_level_type c_sec) {
    switch (c_sec) {
        case C_SEC_LEVEL_TYPE_NONE: return heongpu::sec_level_type::none;
        case C_SEC_LEVEL_TYPE_128: return heongpu::sec_level_type::sec128;
        case C_SEC_LEVEL_TYPE_192: return heongpu::sec_level_type::sec192;
        case C_SEC_LEVEL_TYPE_256: return heongpu::sec_level_type::sec256;
        default:
            return heongpu::sec_level_type::sec128; // Fallback
    }
}

void HEonGPU_Free_C_RNGSeed_Data_Members(C_RNGSeed_Data* seed_data) {
    if (seed_data) {
        if (seed_data->key_data) free(seed_data->key_data);
        if (seed_data->nonce_data) free(seed_data->nonce_data);
        if (seed_data->pstring_data) free(seed_data->pstring_data);
        seed_data->key_data = nullptr; seed_data->key_len = 0;
        seed_data->nonce_data = nullptr; seed_data->nonce_len = 0;
        seed_data->pstring_data = nullptr; seed_data->pstring_len = 0;
    }
}
extern "C" {

int HEonGPU_SynchronizeDevice() {
    hipError_t err = hipDeviceSynchronize();
    return static_cast<int>(err);
}

HE_CKKS_Context* HEonGPU_CKKS_Context_Create(C_keyswitching_type method,
                                             C_sec_level_type sec_level) {
    try {
        heongpu::keyswitching_type cpp_method = map_c_keyswitch_type(method);
        heongpu::sec_level_type cpp_sec_level = map_c_sec_level(sec_level);

        heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_ctx =
            new heongpu::HEContext<heongpu::Scheme::CKKS>(cpp_method, cpp_sec_level);

        HE_CKKS_Context* c_api_context = new HE_CKKS_Context_s;
        c_api_context->cpp_context = cpp_ctx;
        return c_api_context;
    } catch (const std::exception& e) {
        // Consider logging e.what()
        std::cerr << "HEonGPU_CKKS_Context_Create failed: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Create failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Context_Delete(HE_CKKS_Context* context) {
    if (context) {
        delete context->cpp_context;
        delete context;
    }
}

void HEonGPU_CKKS_Context_SetPolyModulusDegree(HE_CKKS_Context* context, size_t degree) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_poly_modulus_degree(degree);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetPolyModulusDegree failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetPolyModulusDegree failed due to an unknown exception." << std::endl;
        }
    }
}

int HEonGPU_CKKS_Context_SetCoeffModulusValues(HE_CKKS_Context* context,
                                               const uint64_t* log_q_bases_data,
                                               size_t log_q_bases_len,
                                               const uint64_t* log_p_bases_data,
                                               size_t log_p_bases_len) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = context->cpp_context;

    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusValues failed: Invalid context pointer." << std::endl;
        return -1; // Error
    }
    if ((log_q_bases_len > 0 && !log_q_bases_data) || (log_p_bases_len > 0 && !log_p_bases_data)) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusValues failed: Non-zero length with null data pointer." << std::endl;
        return -1; // Error for inconsistent arguments
    }

    try {
        std::vector<Data64> cpp_log_q_bases;
        if (log_q_bases_len > 0) {
            cpp_log_q_bases.assign(log_q_bases_data, log_q_bases_data + log_q_bases_len);
        }

        std::vector<Data64> cpp_log_p_bases;
        if (log_p_bases_len > 0) {
            cpp_log_p_bases.assign(log_p_bases_data, log_p_bases_data + log_p_bases_len);
        }
        
        cpp_h_context->set_coeff_modulus_values(cpp_log_q_bases, cpp_log_p_bases);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusValues failed with C++ exception: " << e.what() << std::endl;
        return -2; // Error
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusValues failed due to an unknown C++ exception." << std::endl;
        return -2; // Error
    }
}

int HEonGPU_CKKS_Context_SetCoeffModulusBitSizes(HE_CKKS_Context* context,
                                                 const int* log_q_bit_sizes_data,
                                                 size_t log_q_bit_sizes_len,
                                                 const int* log_p_bit_sizes_data,
                                                 size_t log_p_bit_sizes_len) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = (context->cpp_context);

    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusBitSizes failed: Invalid context pointer." << std::endl;
        return -1;
    }
    if ((log_q_bit_sizes_len > 0 && !log_q_bit_sizes_data) || (log_p_bit_sizes_len > 0 && !log_p_bit_sizes_data)) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusBitSizes failed: Non-zero length with null data pointer." << std::endl;
        return -1;
    }

    try {
        std::vector<int> cpp_log_q_bit_sizes;
        if (log_q_bit_sizes_len > 0) {
            cpp_log_q_bit_sizes.assign(log_q_bit_sizes_data, log_q_bit_sizes_data + log_q_bit_sizes_len);
        }

        std::vector<int> cpp_log_p_bit_sizes;
        if (log_p_bit_sizes_len > 0) {
            cpp_log_p_bit_sizes.assign(log_p_bit_sizes_data, log_p_bit_sizes_data + log_p_bit_sizes_len);
        }
        for(int i=0;i<log_q_bit_sizes_len;i++){
            std::cout << "The cpp_log_q_bit_sizes[i] is: " << cpp_log_q_bit_sizes[i] << std::endl;
        }
        for(int i=0;i<log_p_bit_sizes_len;i++){
            std::cout << "The cpp_log_p_bit_sizes[i] is: " << cpp_log_p_bit_sizes[i] << std::endl;
        }
        cpp_h_context->set_coeff_modulus_bit_sizes(cpp_log_q_bit_sizes, cpp_log_p_bit_sizes);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusBitSizes failed with C++ exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusBitSizes failed due to an unknown C++ exception." << std::endl;
        return -2;
    }
}


void HEonGPU_CKKS_Context_SetExactModulus(HE_CKKS_Context* context, bool exact_mod) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_poly_modulus_degree(exact_mod);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetExactModulus failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetExactModulus failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_PrintParameters(HE_CKKS_Context* context){
    if (context && context->cpp_context) {
        context->cpp_context->print_parameters();
    }
}


int HEonGPU_CKKS_Context_Generate(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->generate();
            return 0; // Success
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_Generate failed: " << e.what() << std::endl;
            return -1; // Failure
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_Generate failed due to an unknown exception." << std::endl;
            return -1; // Failure
        }
    }
    return -2; // Invalid context pointer
}

size_t HEonGPU_CKKS_Context_GetPolyModulusDegree(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_poly_modulus_degree();
        } catch (...) { return 0; }
    }
    return 0;
}

size_t HEonGPU_CKKS_Context_GetCoeffModulusSize(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_ciphertext_modulus_count();
        } catch (...) { return 0; }
    }
    return 0;
}

size_t HEonGPU_CKKS_Context_GetCoeffModulus(HE_CKKS_Context* context,
                                          C_Modulus64* moduli_buffer,
                                          size_t buffer_count) {
    if (!context || !context->cpp_context) {
        std::cerr << "C++ DEBUG: GetCoeffModulus called with invalid context." << std::endl;
        return 0;
    }

    try {
        std::vector<Modulus64> cpp_moduli = 
            context->cpp_context->get_key_modulus();

        std::cerr << "--- C++ DEBUG (Forced Flush) ---" << std::endl;
        std::cerr << "Function: HEonGPU_CKKS_Context_GetCoeffModulus" << std::endl;
        std::cerr << "Vector size from get_key_modulus(): " << cpp_moduli.size() << std::endl;
        for (size_t i = 0; i < cpp_moduli.size(); ++i) {
            std::cerr << "  - Modulus[" << i << "]: value = " << cpp_moduli[i].value << std::endl;
        }
        std::cerr << "----------------------------------" << std::endl;
        if (moduli_buffer == NULL) {
            return cpp_moduli.size();
        }
        size_t num_to_copy = std::min(buffer_count, cpp_moduli.size());
        for (size_t i = 0; i < num_to_copy; ++i) {
            moduli_buffer[i].value = cpp_moduli[i].value;
            moduli_buffer[i].bit   = cpp_moduli[i].bit;
            moduli_buffer[i].mu    = cpp_moduli[i].mu;
        }
        return num_to_copy;

    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_GetCoeffModulus failed: " << e.what() << std::endl;
        return 0; 
    }
}



// double HEonGPU_CKKS_Context_GetScale(HE_CKKS_Context* context) {
//     if (context && context->cpp_context) {
//         try {
//             return context->cpp_context->[HEONGPU DOES NOT SUPPORT]();
//         } catch (...) { return -1.0; } // Error indication
//     }
//     return -1.0;
// }



int HEonGPU_CKKS_Context_Serialize(HE_CKKS_Context* context, unsigned char** out_bytes, size_t* out_len) {
    if (!context || !context->cpp_context || !out_bytes || !out_len) {
        return -1; // Invalid arguments
    }
    try {
        std::ostringstream oss(std::ios::binary);
        context->cpp_context->save(oss);
        std::string str_data = oss.str();
        
        *out_len = str_data.length();
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) {
            *out_len = 0;
            return -2; // Memory allocation failed
        }
        std::copy(str_data.begin(), str_data.end(), *out_bytes);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_Serialize failed: " << e.what() << std::endl;
        if (*out_bytes) { // Should not happen if malloc failed, but defensive
            free(*out_bytes);
            *out_bytes = nullptr;
        }
        *out_len = 0;
        return -3;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Serialize failed due to an unknown exception." << std::endl;
         if (*out_bytes) {
            free(*out_bytes);
            *out_bytes = nullptr;
        }
        *out_len = 0;
        return -3;
    }
}

void HEonGPU_Free_C_RotationIndices_Data_Members(C_RotationIndices_Data* indices_data) {
    if (indices_data) {
        if (indices_data->galois_elements_data) {
            free(indices_data->galois_elements_data);
            indices_data->galois_elements_data = nullptr;
        }
        indices_data->galois_elements_len = 0;
        if (indices_data->rotation_steps_data) {
            free(indices_data->rotation_steps_data);
            indices_data->rotation_steps_data = nullptr;
        }
        indices_data->rotation_steps_len = 0;
    }
}

HE_CKKS_Context* HEonGPU_CKKS_Context_Deserialize(const unsigned char* bytes, size_t len) {
    if (!bytes || len == 0) {
        return nullptr;
    }
    try {
        // Create a temporary HEContext to call the non-static load method.
        // The constructor parameters for this temp object might not matter if load overwrites them,
        // but it's safer to use some defaults. The actual context parameters will come from the stream.
        
        heongpu::keyswitching_type default_ks_type = heongpu::keyswitching_type::KEYSWITCHING_METHOD_I;
        heongpu::sec_level_type default_sec_level = heongpu::sec_level_type::sec128;

        heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_ctx =
            new heongpu::HEContext<heongpu::Scheme::CKKS>(default_ks_type, default_sec_level);

        std::string str_data(reinterpret_cast<const char*>(bytes), len);
        std::istringstream iss(str_data, std::ios::binary);
        cpp_ctx->load(iss); // The load method will populate the cpp_ctx



        HE_CKKS_Context* c_api_context = new HE_CKKS_Context_s;
        c_api_context->cpp_context = cpp_ctx;
        return c_api_context;

    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_Deserialize failed: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Deserialize failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_FreeSerializedData(void* data_ptr) {
    if (data_ptr) {
        free(data_ptr);
    }
}

} // extern "C"