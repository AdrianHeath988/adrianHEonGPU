#include "hip/hip_runtime.h"
#include "context_c_api.h"
#include "heongpu.cuh" // Main HEonGPU include for HEContext, Scheme, types
#include "schemes.h"      // For heongpu::Scheme, heongpu::keyswitching_type
#include "secstdparams.h" // For heongpu::sec_level_type
#include "hostvector.cuh" // For heongpu::HostVector
#include "util.cuh"       // For heongpu::Data128, heongpu::Modulus64 (via modular_arith.cuh)

#include <vector>
#include <iostream> // For potential error logging (stderr)
#include <sstream>  // For serialization to/from memory buffer
#include <algorithm> // For std::copy, std::min

// Define the opaque struct to hold the actual C++ HEContext object
typedef struct HE_CKKS_Context_s HE_CKKS_Context;

// Helper to map C enum to C++ enum for keyswitching_type
static heongpu::keyswitching_type map_c_keyswitch_type(C_keyswitching_type c_type) {
    switch (c_type) {
        case C_KEYSWITCHING_METHOD_I: return heongpu::keyswitching_type::KEYSWITCHING_METHOD_I;
        case C_KEYSWITCHING_METHOD_II: return heongpu::keyswitching_type::KEYSWITCHING_METHOD_II;
        default:
            // Handle error or default, though C enum should prevent invalid values if used correctly
            return heongpu::keyswitching_type::KEYSWITCHING_METHOD_I; // Fallback, consider error
    }
}

// Helper to map C enum to C++ enum for sec_level_type
static heongpu::sec_level_type map_c_sec_level(C_sec_level_type c_sec) {
    switch (c_sec) {
        case C_SEC_LEVEL_TYPE_NONE: return heongpu::sec_level_type::none;
        case C_SEC_LEVEL_TYPE_128: return heongpu::sec_level_type::sec128;
        case C_SEC_LEVEL_TYPE_192: return heongpu::sec_level_type::sec192;
        case C_SEC_LEVEL_TYPE_256: return heongpu::sec_level_type::sec256;
        default:
            return heongpu::sec_level_type::sec128; // Fallback
    }
}

void HEonGPU_Free_C_RNGSeed_Data_Members(C_RNGSeed_Data* seed_data) {
    if (seed_data) {
        if (seed_data->key_data) free(seed_data->key_data);
        if (seed_data->nonce_data) free(seed_data->nonce_data);
        if (seed_data->pstring_data) free(seed_data->pstring_data);
        seed_data->key_data = nullptr; seed_data->key_len = 0;
        seed_data->nonce_data = nullptr; seed_data->nonce_len = 0;
        seed_data->pstring_data = nullptr; seed_data->pstring_len = 0;
    }
}
extern "C" {

HE_CKKS_Context* HEonGPU_CKKS_Context_Create(C_keyswitching_type method,
                                             C_sec_level_type sec_level,
                                             C_cudaStream_t stream) {
    try {
        heongpu::keyswitching_type cpp_method = map_c_keyswitch_type(method);
        heongpu::sec_level_type cpp_sec_level = map_c_sec_level(sec_level);
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);

        heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_ctx =
            new heongpu::HEContext<heongpu::Scheme::CKKS>(cpp_method, cpp_sec_level, cpp_stream);

        HE_CKKS_Context* c_api_context = new HE_CKKS_Context_s;
        c_api_context->cpp_context = cpp_ctx;
        return c_api_context;
    } catch (const std::exception& e) {
        // Consider logging e.what()
        std::cerr << "HEonGPU_CKKS_Context_Create failed: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Create failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_Context_Delete(HE_CKKS_Context* context) {
    if (context) {
        delete context->cpp_context;
        delete context;
    }
}

void HEonGPU_CKKS_Context_SetPolyModulusDegree(HE_CKKS_Context* context, size_t degree) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_poly_modulus_degree(degree);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetPolyModulusDegree failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetPolyModulusDegree failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_SetCoeffModulusPSValues(HE_CKKS_Context* context,
                                                  const int* coeff_modulus_ps_values,
                                                  size_t count) {
    if (context && context->cpp_context && coeff_modulus_ps_values) {
        try {
            std::vector<int> ps_values(coeff_modulus_ps_values, coeff_modulus_ps_values + count);
            context->cpp_context->set_coeff_modulus_ps_values(ps_values);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusPSValues failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusPSValues failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_SetCoeffModulusQSValues(HE_CKKS_Context* context,
                                                  const int* coeff_modulus_qs_values,
                                                  size_t count) {
    if (context && context->cpp_context && coeff_modulus_qs_values) {
        try {
            std::vector<int> qs_values(coeff_modulus_qs_values, coeff_modulus_qs_values + count);
            context->cpp_context->set_coeff_modulus_qs_values(qs_values);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusQSValues failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusQSValues failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_SetCoeffModulusDefaultValues(HE_CKKS_Context* context,
                                                       uint32_t num_primes,
                                                       uint32_t log_scale) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_coeff_modulus_default_values(num_primes, log_scale);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusDefaultValues failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetCoeffModulusDefaultValues failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_SetExactModulus(HE_CKKS_Context* context, bool exact_mod) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_exact_modulus(exact_mod);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetExactModulus failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetExactModulus failed due to an unknown exception." << std::endl;
        }
    }
}

void HEonGPU_CKKS_Context_SetScale(HE_CKKS_Context* context, double scale) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->set_scale(scale);
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_SetScale failed: " << e.what() << std::endl;
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_SetScale failed due to an unknown exception." << std::endl;
        }
    }
}

int HEonGPU_CKKS_Context_Generate(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->generate();
            return 0; // Success
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_Generate failed: " << e.what() << std::endl;
            return -1; // Failure
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_Generate failed due to an unknown exception." << std::endl;
            return -1; // Failure
        }
    }
    return -2; // Invalid context pointer
}

size_t HEonGPU_CKKS_Context_GetPolyModulusDegree(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_poly_modulus_degree();
        } catch (...) { return 0; }
    }
    return 0;
}

size_t HEonGPU_CKKS_Context_GetCoeffModulusSize(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_coeff_modulus_size();
        } catch (...) { return 0; }
    }
    return 0;
}

size_t HEonGPU_CKKS_Context_GetCoeffModulus(HE_CKKS_Context* context,
                                            C_Modulus64* moduli_buffer,
                                            size_t buffer_count) {
    if (context && context->cpp_context && moduli_buffer) {
        try {
            const heongpu::HostVector<heongpu::Modulus64>& cpp_moduli = 
                context->cpp_context->get_coeff_modulus_host();
            size_t num_to_copy = std::min(buffer_count, cpp_moduli.size());
            for (size_t i = 0; i < num_to_copy; ++i) {
                moduli_buffer[i].value = cpp_moduli[i].value;
                moduli_buffer[i].bit   = cpp_moduli[i].bit;
                moduli_buffer[i].mu    = cpp_moduli[i].mu;
            }
            return num_to_copy;
        } catch (const std::exception& e) {
            std::cerr << "HEonGPU_CKKS_Context_GetCoeffModulus failed: " << e.what() << std::endl;
            return 0; 
        } catch (...) {
            std::cerr << "HEonGPU_CKKS_Context_GetCoeffModulus failed due to an unknown exception." << std::endl;
            return 0;
        }
    }
    return 0;
}



double HEonGPU_CKKS_Context_GetScale(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_scale();
        } catch (...) { return -1.0; } // Error indication
    }
    return -1.0;
}

C_cudaStream_t HEonGPU_CKKS_Context_GetCUDAStream(HE_CKKS_Context* context) {
    if (context && context->cpp_context) {
        try {
            return static_cast<C_cudaStream_t>(context->cpp_context->get_context_data_stream());
        } catch (...) { return nullptr;}
    }
    return nullptr;
}

int HEonGPU_CKKS_Context_Serialize(HE_CKKS_Context* context, unsigned char** out_bytes, size_t* out_len) {
    if (!context || !context->cpp_context || !out_bytes || !out_len) {
        return -1; // Invalid arguments
    }
    try {
        std::ostringstream oss(std::ios::binary);
        context->cpp_context->save(oss);
        std::string str_data = oss.str();
        
        *out_len = str_data.length();
        *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
        if (!(*out_bytes)) {
            *out_len = 0;
            return -2; // Memory allocation failed
        }
        std::copy(str_data.begin(), str_data.end(), *out_bytes);
        return 0; // Success
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_Serialize failed: " << e.what() << std::endl;
        if (*out_bytes) { // Should not happen if malloc failed, but defensive
            free(*out_bytes);
            *out_bytes = nullptr;
        }
        *out_len = 0;
        return -3;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Serialize failed due to an unknown exception." << std::endl;
         if (*out_bytes) {
            free(*out_bytes);
            *out_bytes = nullptr;
        }
        *out_len = 0;
        return -3;
    }
}

void HEonGPU_Free_C_RotationIndices_Data_Members(C_RotationIndices_Data* indices_data) {
    if (indices_data) {
        if (indices_data->galois_elements_data) {
            free(indices_data->galois_elements_data);
            indices_data->galois_elements_data = nullptr;
        }
        indices_data->galois_elements_len = 0;
        if (indices_data->rotation_steps_data) {
            free(indices_data->rotation_steps_data);
            indices_data->rotation_steps_data = nullptr;
        }
        indices_data->rotation_steps_len = 0;
    }
}

HE_CKKS_Context* HEonGPU_CKKS_Context_Deserialize(const unsigned char* bytes, size_t len, C_cudaStream_t stream) {
    if (!bytes || len == 0) {
        return nullptr;
    }
    try {
        // Create a temporary HEContext to call the non-static load method.
        // The constructor parameters for this temp object might not matter if load overwrites them,
        // but it's safer to use some defaults. The actual context parameters will come from the stream.
        
        heongpu::keyswitching_type default_ks_type = heongpu::keyswitching_type::KEYSWITCHING_METHOD_I;
        heongpu::sec_level_type default_sec_level = heongpu::sec_level_type::sec128;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);

        heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_ctx =
            new heongpu::HEContext<heongpu::Scheme::CKKS>(default_ks_type, default_sec_level, cpp_stream);

        std::string str_data(reinterpret_cast<const char*>(bytes), len);
        std::istringstream iss(str_data, std::ios::binary);
        cpp_ctx->load(iss); // The load method will populate the cpp_ctx



        HE_CKKS_Context* c_api_context = new HE_CKKS_Context_s;
        c_api_context->cpp_context = cpp_ctx;
        return c_api_context;

    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_Context_Deserialize failed: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_Context_Deserialize failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_FreeSerializedData(void* data_ptr) {
    if (data_ptr) {
        free(data_ptr);
    }
}

} // extern "C"