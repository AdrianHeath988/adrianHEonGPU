#include "operator_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/encoder.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/ciphertext.cuh"
#include "ckks/evaluationkey.cuh"
#include "ckks/operator.cuh" // The C++ classes we are wrapping

#include "storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow


// Define opaque structs
typedef struct HE_CKKS_ArithmeticOperator_s HE_CKKS_ArithmeticOperator;
typedef struct HE_CKKS_LogicOperator_s HE_CKKS_LogicOperator;
// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr;
    return context->cpp_context;
}
static heongpu::HEEncoder<heongpu::Scheme::CKKS>* get_cpp_encoder(HE_CKKS_Encoder* encoder) {
    if (!encoder || !encoder->cpp_encoder) return nullptr;
    return encoder->cpp_encoder;
}
static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static const heongpu::Plaintext<heongpu::Scheme::CKKS>* get_const_cpp_plaintext(const HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_cpp_ciphertext(HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static const heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_const_cpp_ciphertext(const HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static heongpu::Relinkey<heongpu::Scheme::CKKS>* get_cpp_relinkey(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return nullptr;
    return rk->cpp_relinkey;
}
static heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_cpp_galoiskey(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}


// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_op(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        // If C_STORAGE_TYPE_INVALID or other, it uses default from cpp_options.
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEArithmeticOperator Lifecycle ---
HE_CKKS_ArithmeticOperator* HEonGPU_CKKS_ArithmeticOperator_Create(HE_CKKS_Context* context, HE_CKKS_Encoder* encoder) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_h_encoder = get_cpp_encoder(encoder);
    if (!cpp_h_context || !cpp_h_encoder) {
        std::cerr << "ArithmeticOperator_Create: Invalid context or encoder." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_h_encoder);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_ArithmeticOperator_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_arith_op = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "ArithmeticOperator_Create Error: " << e.what() << std::endl; return nullptr;} 
      catch (...) { std::cerr << "ArithmeticOperator_Create Unknown Error" << std::endl; return nullptr;}
}

void HEonGPU_CKKS_ArithmeticOperator_Delete(HE_CKKS_ArithmeticOperator* op) {
    if (op) { delete op->cpp_arith_op; delete op; }
}

// --- CKKS HEArithmeticOperator Operations ---

// Addition Example (In-place)
void HEonGPU_CKKS_ArithmeticOperator_Add_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (const std::exception& e) { std::cerr << "Add_Inplace Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "Add_Inplace Unknown Error" << std::endl;}
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct3_out->cpp_ciphertext), cpp_options);
        return ct3_out;
    } catch (const std::exception& e) { std::cerr << "Add Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Add Unknown Error" << std::endl; return nullptr; }
}

// Add_Plain_Inplace
void HEonGPU_CKKS_ArithmeticOperator_Add_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (const std::exception& e) { std::cerr << "Add_Plain_Inplace Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "Add_Plain_Inplace Unknown Error" << std::endl;}
}

// Add_Plain (returns new Ciphertext)
HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        
        op->cpp_arith_op->add_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct3_out->cpp_ciphertext), cpp_options);
        return ct3_out;
    } catch (const std::exception& e) { std::cerr << "Add_Plain Error: " << e.what() << std::endl; return nullptr;}
      catch (...) { std::cerr << "Add_Plain Unknown Error" << std::endl; return nullptr;}
}


// --- Implementations for Subtraction (similar to Addition) ---
void HEonGPU_CKKS_ArithmeticOperator_Sub_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct3_out->cpp_ciphertext), cpp_options);
        
        return ct3_out;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Sub_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct3_out->cpp_ciphertext), cpp_options);
        
        return ct3_out;
    } catch (...) { return nullptr; }
}

// --- Negation ---
void HEonGPU_CKKS_ArithmeticOperator_Negate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->negate_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Negate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->negate(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        return ct_out;
    } catch (...) { return nullptr; }
}

// --- Implementations for Multiplication (Pattern: check args, map options, call C++ method, wrap result if new object) ---
void HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct_out->cpp_ciphertext), cpp_options);
        
        return ct_out;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Multiply_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext || !ct_out || !ct_out->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        return ct_out;
    } catch (...) { return nullptr; }
}



// --- Relinearize ---
void HEonGPU_CKKS_ArithmeticOperator_Relinearize_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->relinearize_inplace(*(ct_in_out->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
    } catch (...) { /* error handling */ }
}

// --- ModDrop / Rescale ---
void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        
        return ct_out;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Plaintext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Plaintext* pt_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !pt_in_out || !pt_in_out->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(pt_in_out->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}


void HEonGPU_CKKS_ArithmeticOperator_Rescale_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rescale_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}



// --- Rotation / Conjugation ---
void HEonGPU_CKKS_ArithmeticOperator_Rotate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_rows_inplace(*(ct_in_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), steps, cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Rotate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_rows(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey),steps, cpp_options);
        
        return ct_out;
    } catch (...) { return nullptr; }
}


HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Conjugate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->conjugate(*(ct_in->cpp_ciphertext),*(ct_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), cpp_options);
        
        return ct_out;
    } catch (...) { return nullptr; }
}


// --- Bootstrapping ---
// Note: C++ bootstrap methods return new Ciphertext objects.

// Bootstrapping will be wrapped after non-bootstrapping works.

// #define WRAP_BOOTSTRAP_FUNC(FuncName, CppFuncName) \
// HE_CKKS_Ciphertext* FuncName(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_c, HE_CKKS_RelinKey* relin_key_c, HE_CKKS_GaloisKey* galois_key_conj_c, HE_CKKS_GaloisKey* galois_key_rot_c, const C_ExecutionOptions* options_c) { \
//     if (!op || !op->cpp_arith_op || !ct_in_c || !ct_in_c->cpp_ciphertext || \
//         !relin_key_c || !relin_key_c->cpp_relinkey || \
//         !galois_key_conj_c || !galois_key_conj_c->cpp_galoiskey || \
//         !galois_key_rot_c || !galois_key_rot_c->cpp_galoiskey) { \
//         std::cerr << #FuncName " Error: Invalid argument(s).\n"; return nullptr; \
//     } \
//     try { \
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
//         auto cpp_result = op->cpp_arith_op->CppFuncName(*(ct_in_c->cpp_ciphertext), *(relin_key_c->cpp_relinkey), *(galois_key_conj_c->cpp_galoiskey), *(galois_key_rot_c->cpp_galoiskey), cpp_options); \
//         auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result)); \
//         if (!cpp_heap_result) return nullptr; \
//         auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s; \
//         if (!c_api_result) { delete cpp_heap_result; return nullptr; } \
//         c_api_result->cpp_ciphertext = cpp_heap_result; \
//         return c_api_result; \
//     } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return nullptr; } \
//       catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return nullptr; } \
// }

// WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap, bootstrap)
// WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Slim, bootstrap_slim)
// WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Bit, bootstrap_bit)
// WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Gate, bootstrap_gate)


// --- CKKS HELogicOperator Lifecycle ---
// I believe this to be unnecassary for Orion operations.

// HE_CKKS_LogicOperator* HEonGPU_CKKS_LogicOperator_Create(HE_CKKS_Context* context, HE_CKKS_Encoder* encoder) {
//     heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
//     heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_h_encoder = get_cpp_encoder(encoder);
//      if (!cpp_h_context || !cpp_h_encoder) {
//         std::cerr << "LogicOperator_Create: Invalid context or encoder." << std::endl;
//         return nullptr;
//     }
//     try {
//         auto cpp_obj = new (std::nothrow) heongpu::HELogicOperator<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_h_encoder);
//         if (!cpp_obj) return nullptr;
//         auto c_api_obj = new (std::nothrow) HE_CKKS_LogicOperator_s;
//         if (!c_api_obj) { delete cpp_obj; return nullptr; }
//         c_api_obj->cpp_logic_op = cpp_obj;
//         return c_api_obj;
//     } catch (const std::exception& e) { std::cerr << "LogicOperator_Create Error: " << e.what() << std::endl; return nullptr;}
//       catch (...) { std::cerr << "LogicOperator_Create Unknown Error" << std::endl; return nullptr;}
// }

// void HEonGPU_CKKS_LogicOperator_Delete(HE_CKKS_LogicOperator* op) {
//     if (op) { delete op->cpp_logic_op; delete op; }
// }

// // --- CKKS HELogicOperator Operations ---
// void HEonGPU_CKKS_LogicOperator_NOT_Approximation_Inplace(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
//     if (!op || !op->cpp_logic_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
//     try {
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
//         op->cpp_logic_op->NOT_approximation_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
//     } catch (...) { /* error handling */ }
// }

// HE_CKKS_Ciphertext* HEonGPU_CKKS_LogicOperator_NOT_Approximation(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct_in_c, const C_ExecutionOptions* options_c) {
//     if (!op || !op->cpp_logic_op || !ct_in_c || !ct_in_c->cpp_ciphertext) return nullptr;
//     try {
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
//         auto cpp_result = op->cpp_logic_op->NOT_approximation(*(ct_in_c->cpp_ciphertext), cpp_options);
//         auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
//         if (!cpp_heap_result) return nullptr;
//         auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
//         if (!c_api_result) { delete cpp_heap_result; return nullptr; }
//         c_api_result->cpp_ciphertext = cpp_heap_result;
//         return c_api_result;
//     } catch (...) { return nullptr; }
// }

// // (XOR and XNOR functions follow a similar pattern, taking GaloisKey and RelinKey)
// #define WRAP_LOGIC_BINARY_OP_INPLACE(FuncName, CppFuncName) \
// void FuncName(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct1_in_out_c, HE_CKKS_Ciphertext* ct2_in_c, HE_CKKS_GaloisKey* galois_key_c, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) { \
//     if (!op || !op->cpp_logic_op || !ct1_in_out_c || !ct1_in_out_c->cpp_ciphertext || \
//         !ct2_in_c || !ct2_in_c->cpp_ciphertext || \
//         !galois_key_c || !galois_key_c->cpp_galoiskey || \
//         !relin_key_c || !relin_key_c->cpp_relinkey) { \
//         std::cerr << #FuncName " Error: Invalid argument(s).\n"; return; \
//     } \
//     try { \
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
//         op->cpp_logic_op->CppFuncName(*(ct1_in_out_c->cpp_ciphertext), *(ct2_in_c->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), *(relin_key_c->cpp_relinkey), cpp_options); \
//     } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; } \
//       catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; } \
// }

// #define WRAP_LOGIC_BINARY_OP_NEW(FuncName, CppFuncName) \
// HE_CKKS_Ciphertext* FuncName(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct1_in_c, HE_CKKS_Ciphertext* ct2_in_c, HE_CKKS_GaloisKey* galois_key_c, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) { \
//     if (!op || !op->cpp_logic_op || !ct1_in_c || !ct1_in_c->cpp_ciphertext || \
//         !ct2_in_c || !ct2_in_c->cpp_ciphertext || \
//         !galois_key_c || !galois_key_c->cpp_galoiskey || \
//         !relin_key_c || !relin_key_c->cpp_relinkey) { \
//         std::cerr << #FuncName " Error: Invalid argument(s).\n"; return nullptr; \
//     } \
//     try { \
//         heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
//         auto cpp_result = op->cpp_logic_op->CppFuncName(*(ct1_in_c->cpp_ciphertext), *(ct2_in_c->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), *(relin_key_c->cpp_relinkey), cpp_options); \
//         auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result)); \
//         if (!cpp_heap_result) return nullptr; \
//         auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s; \
//         if (!c_api_result) { delete cpp_heap_result; return nullptr; } \
//         c_api_result->cpp_ciphertext = cpp_heap_result; \
//         return c_api_result; \
//     } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return nullptr; } \
//       catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return nullptr; } \
// }

// WRAP_LOGIC_BINARY_OP_INPLACE(HEonGPU_CKKS_LogicOperator_XOR_Approximation_Inplace, XOR_approximation_inplace)
// WRAP_LOGIC_BINARY_OP_NEW(HEonGPU_CKKS_LogicOperator_XOR_Approximation, XOR_approximation)
// WRAP_LOGIC_BINARY_OP_INPLACE(HEonGPU_CKKS_LogicOperator_XNOR_Approximation_Inplace, XNOR_approximation_inplace)
// WRAP_LOGIC_BINARY_OP_NEW(HEonGPU_CKKS_LogicOperator_XNOR_Approximation, XNOR_approximation)


} // extern "C"