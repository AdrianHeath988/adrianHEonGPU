#include "operator_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/encoder.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/ciphertext.cuh"
#include "ckks/evaluationkey.cuh"
#include "ckks/operator.cuh" // The C++ classes we are wrapping

#include "storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow


// Define opaque structs
typedef struct HE_CKKS_ArithmeticOperator_s HE_CKKS_ArithmeticOperator;
typedef struct HE_CKKS_LogicOperator_s HE_CKKS_LogicOperator;
// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr;
    return context->cpp_context;
}
static heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>* get_cpp_arith_op(HE_CKKS_ArithmeticOperator* op) {
    if (!op || !op->cpp_arith_op) return nullptr;
    return op->cpp_arith_op;
}

static heongpu::HEEncoder<heongpu::Scheme::CKKS>* get_cpp_encoder(HE_CKKS_Encoder* encoder) {
    if (!encoder || !encoder->cpp_encoder) return nullptr;
    return encoder->cpp_encoder;
}
static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static const heongpu::Plaintext<heongpu::Scheme::CKKS>* get_const_cpp_plaintext(const HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_cpp_ciphertext(HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static const heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_const_cpp_ciphertext(const HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static heongpu::Relinkey<heongpu::Scheme::CKKS>* get_cpp_relinkey(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return nullptr;
    return rk->cpp_relinkey;
}
static heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_cpp_galoiskey(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}
static heongpu::ExecutionOptions map_c_to_cpp_execution_options(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}

// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_op(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        // If C_STORAGE_TYPE_INVALID or other, it uses default from cpp_options.
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEArithmeticOperator Lifecycle ---
HE_CKKS_ArithmeticOperator* HEonGPU_CKKS_ArithmeticOperator_Create(HE_CKKS_Context* context, HE_CKKS_Encoder* encoder) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_h_encoder = get_cpp_encoder(encoder);
    if (!cpp_h_context || !cpp_h_encoder) {
        std::cerr << "ArithmeticOperator_Create: Invalid context or encoder." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_h_encoder);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_ArithmeticOperator_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_arith_op = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "ArithmeticOperator_Create Error: " << e.what() << std::endl; return nullptr;} 
      catch (...) { std::cerr << "ArithmeticOperator_Create Unknown Error" << std::endl; return nullptr;}
}

void HEonGPU_CKKS_ArithmeticOperator_Delete(HE_CKKS_ArithmeticOperator* op) {
    if (op) { delete op->cpp_arith_op; delete op; }
}

// --- CKKS HEArithmeticOperator Operations ---

// Addition Example (In-place)
void HEonGPU_CKKS_ArithmeticOperator_Add_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (const std::exception& e) { std::cerr << "Add_Inplace Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "Add_Inplace Unknown Error" << std::endl;}
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct3_out->cpp_ciphertext), cpp_options);
        return ct3_out;
    } catch (const std::exception& e) { std::cerr << "Add Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Add Unknown Error" << std::endl; return nullptr; }
}

// Add_Plain_Inplace
void HEonGPU_CKKS_ArithmeticOperator_Add_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (const std::exception& e) { 
        std::cerr << "Add_Plain_Inplace Error: " << e.what() << std::endl; 
        if (ct_in_out && ct_in_out->cpp_ciphertext) {
            std::cerr << "    Offending Ciphertext Depth: " << ct_in_out->cpp_ciphertext->depth() << std::endl;
        }
        if (pt_in && pt_in->cpp_plaintext) {
            std::cerr << "    Offending Plaintext Depth: " << pt_in->cpp_plaintext->depth() << std::endl;
        }
    }
      catch (...) { std::cerr << "Add_Plain_Inplace Unknown Error" << std::endl;}
}

// Add_Plain (returns new Ciphertext)
HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        
        op->cpp_arith_op->add_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct3_out->cpp_ciphertext), cpp_options);
        return ct3_out;
    } catch (const std::exception& e) { std::cerr << "Add_Plain Error: " << e.what() << std::endl; return nullptr;}
      catch (...) { std::cerr << "Add_Plain Unknown Error" << std::endl; return nullptr;}
}


// --- Implementations for Subtraction (similar to Addition) ---
void HEonGPU_CKKS_ArithmeticOperator_Sub_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct3_out->cpp_ciphertext), cpp_options);
        
        return ct3_out;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Sub_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct3_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct3_out->cpp_ciphertext), cpp_options);
        
        return ct3_out;
    } catch (...) { return nullptr; }
}

// --- Negation ---
void HEonGPU_CKKS_ArithmeticOperator_Negate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->negate_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Negate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->negate(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        return ct_out;
    } catch (...) { return nullptr; }
}

// --- Implementations for Multiplication (Pattern: check args, map options, call C++ method, wrap result if new object) ---
void HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    std::cout << "[C++ DEBUG] Entered HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain_Inplace." << std::endl;

    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        // Added a more descriptive error message here
        std::cerr << "Multiply_Plain_Inplace Error: Received a null pointer for one of the arguments." << std::endl;
        return;
    }

    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    
    // --- Modified error handling to print exceptions ---
    } catch (const std::exception& e) {
        // This will catch standard C++ exceptions (like std::invalid_argument) and print their messages.
        std::cerr << "Multiply_Plain_Inplace caught a standard exception: " << e.what() << std::endl;
        if (ct_in_out && ct_in_out->cpp_ciphertext) {
            std::cerr << "    Offending Ciphertext Depth: " << ct_in_out->cpp_ciphertext->depth() << std::endl;
        }
        if (pt_in && pt_in->cpp_plaintext) {
            std::cerr << "    Offending Plaintext Depth: " << pt_in->cpp_plaintext->depth() << std::endl;
        }
    } catch (...) {
        // This is a catch-all for any other non-standard exceptions.
        std::cerr << "Multiply_Plain_Inplace caught an unknown exception." << std::endl;
    }
}


HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Multiply_Plain Error: Received a null pointer for one of the arguments." << std::endl;
        return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), *(ct_out->cpp_ciphertext), cpp_options);
        
        return ct_out;
    } catch (const std::exception& e) {
        // This will catch standard C++ exceptions and print their messages.
        std::cerr << "Multiply_Plain caught a standard exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        // This is a catch-all for any other non-standard exceptions.
        std::cerr << "Multiply_Plain caught an unknown exception." << std::endl;
        return nullptr;
    }
}


void HEonGPU_CKKS_ArithmeticOperator_Multiply_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Multiply_Inplace Error: Received a null pointer for one of the arguments." << std::endl;
        return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (const std::exception& e) {
        std::cerr << "Multiply_Inplace caught a standard exception: " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "Multiply_Inplace caught an unknown exception." << std::endl;
    }
}


HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext || !ct_out || !ct_out->cpp_ciphertext) {
        std::cerr << "Multiply Error: Received a null pointer for one of the arguments." << std::endl;
        return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        return ct_out;
    } catch (const std::exception& e) {
        std::cerr << "Multiply caught a standard exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "Multiply caught an unknown exception." << std::endl;
        return nullptr;
    }
}



// --- Relinearize ---
void HEonGPU_CKKS_ArithmeticOperator_Relinearize_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->relinearize_inplace(*(ct_in_out->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
    } catch (...) { /* error handling */ }
}

// --- ModDrop / Rescale ---
void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    // Initial null pointer check remains the same
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) {
        std::cerr << "ModDrop_Inplace Error: Invalid operator or ciphertext pointer." << std::endl;
        return;
    }

    // --- Add this try...catch block for detailed error reporting ---
    try {
        std::cout << "    in HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext_Inplace: " << std::endl;
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
        std::cout << "    rescale_required: "
                  << ct_in_out->cpp_ciphertext->rescale_required() << std::endl;
        std::cout << "    relinearization_required: "
                  << ct_in_out->cpp_ciphertext->relinearization_required() << std::endl;
    } catch (const std::exception& e) {
        // This will print the specific C++ exception message to your console.
        std::cerr << "[C++ EXCEPTION] A standard exception was caught in ModDrop_Inplace: "
                  << e.what() << std::endl;
    } catch (...) {
        // This is a fallback for non-standard exceptions.
        std::cerr << "[C++ EXCEPTION] An unknown exception was caught in ModDrop_Inplace." << std::endl;
    }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) {
        std::cerr << "ModDrop_Ciphertext Error: Received a null pointer for one of the arguments." << std::endl;
        return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), cpp_options);
        
        return ct_out;
    } catch (const std::exception& e) {
        // This will catch standard C++ exceptions and print their messages.
        std::cerr << "ModDrop_Ciphertext caught a standard exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        // This is a catch-all for any other non-standard exceptions.
        std::cerr << "ModDrop_Ciphertext caught an unknown exception." << std::endl;
        return nullptr;
    }
}


void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Plaintext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Plaintext* pt_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !pt_in_out || !pt_in_out->cpp_plaintext) {
        std::cerr << "ModDrop_Plaintext_Inplace Error: Received a null pointer for one of the arguments." << std::endl;
        return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(pt_in_out->cpp_plaintext), cpp_options);
    } catch (const std::exception& e) {
        std::cerr << "ModDrop_Plaintext_Inplace caught a standard exception: " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "ModDrop_Plaintext_Inplace caught an unknown exception." << std::endl;
    }
}


void HEonGPU_CKKS_ArithmeticOperator_Rescale_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    // Initial null pointer check remains the same
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) {
        std::cerr << "Rescale_Inplace Error: Invalid operator or ciphertext pointer." << std::endl;
        return;
    }


    try {
        
        std::cout << "    in HEonGPU_CKKS_ArithmeticOperator_Rescale_Inplace: " << std::endl;
        std::cout << "  ct_in pointer: " << ct_in_out << std::endl;
        if (ct_in_out) std::cout << "  ct_in->cpp_ciphertext: " << ct_in_out->cpp_ciphertext << std::endl;
        std::cout << "    rescale_required: "
                  << ct_in_out->cpp_ciphertext->rescale_required() << std::endl;
        std::cout << "    relinearization_required: "
                  << ct_in_out->cpp_ciphertext->relinearization_required() << std::endl;
        if(!ct_in_out->cpp_ciphertext->rescale_required()){
            return;
        }
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rescale_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (const std::exception& e) {
        // This will print the specific C++ exception message to your console.
        std::cerr << "[C++ EXCEPTION] A standard exception was caught in Rescale_Inplace: "
                  << e.what() << std::endl;
    } catch (...) {
        // This is a fallback for non-standard exceptions.
        std::cerr << "[C++ EXCEPTION] An unknown exception was caught in Rescale_Inplace." << std::endl;
    }
}




// --- Rotation / Conjugation ---
void HEonGPU_CKKS_ArithmeticOperator_Rotate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    std::cerr  << "[C++ DEBUG] Rotate Inplace Check:" << std::endl;
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) {
        std::cerr << "Rotate_Inplace Error: Received a null pointer for one of the arguments." << std::endl;
        return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_rows_inplace(*(ct_in_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), steps, cpp_options);
    
    } catch (const std::exception& e) {
        // This will catch standard C++ exceptions and print their messages.
        std::cerr << "Rotate_Inplace caught a standard exception: " << e.what() << std::endl;
    } catch (...) {
        // This is a catch-all for any other non-standard exceptions.
        std::cerr << "Rotate_Inplace caught an unknown exception." << std::endl;
    }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Rotate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    std::cerr << "[C++ DEBUG] Rotate Check:" << std::endl;
    std::cout << "  op pointer: " << op << std::endl;
    if (op) std::cout << "  op->cpp_arith_op: " << op->cpp_arith_op << std::endl;
    std::cout << "  ct_in pointer: " << ct_in << std::endl;
    if (ct_in) std::cout << "  ct_in->cpp_ciphertext: " << ct_in->cpp_ciphertext << std::endl;
    if (ct_in && ct_in->cpp_ciphertext) {
        std::cout << "  [C++ DEBUG] Flags before Rotate:" << std::endl;
        std::cout << "    rescale_required: "
                  << ct_in->cpp_ciphertext->rescale_required() << std::endl;
        std::cout << "    relinearization_required: "
                  << ct_in->cpp_ciphertext->relinearization_required() << std::endl;
    }


    std::cout << "  galois_key_c pointer: " << galois_key_c << std::endl;
    if (galois_key_c) std::cout << "  galois_key_c->cpp_galoiskey: " << galois_key_c->cpp_galoiskey << std::endl;

    
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_rows(*(ct_in->cpp_ciphertext), *(ct_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), steps, cpp_options);
        return ct_out;
    } catch (const std::exception& e) {
        // This will print the actual C++ error message to your console
        std::cerr << "[C++ EXCEPTION] A standard exception was caught in Rotate: "
                  << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        // This is a fallback for non-standard exceptions
        std::cerr << "[C++ EXCEPTION] An unknown exception was caught in Rotate." << std::endl;
        return nullptr;
    }

}


HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Conjugate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_Ciphertext* ct_out, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->conjugate(*(ct_in->cpp_ciphertext),*(ct_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), cpp_options);
        
        return ct_out;
    } catch (...) { return nullptr; }
}


// --- Bootstrapping ---
// Note: C++ bootstrap methods return new Ciphertext objects.
int HEonGPU_CKKS_ArithmeticOperator_GenerateBootstrappingParams(HE_CKKS_ArithmeticOperator* op,
                                                                double scale,
                                                                const C_BootstrappingConfig* config) {
    heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>* cpp_op = get_cpp_arith_op(op);
    if (!cpp_op || !config) {
        std::cerr << "GenerateBootstrappingParams Error: Invalid operator or config pointer.\n";
        return -1;
    }
    try {
        heongpu::BootstrappingConfig cpp_config(config->CtoS_piece, config->StoC_piece, config->taylor_number, config->less_key_mode);
        cpp_op->generate_bootstrapping_params(scale, cpp_config);
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "GenerateBootstrappingParams failed with exception: " << e.what() << std::endl;
        return -2;
    } catch (...) {
        std::cerr << "GenerateBootstrappingParams failed with unknown exception." << std::endl;
        return -2;
    }
}

int HEonGPU_CKKS_ArithmeticOperator_GetBootstrappingKeyIndices(HE_CKKS_ArithmeticOperator* op,
                                                               int** out_indices,
                                                               size_t* out_count) {
    heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>* cpp_op = get_cpp_arith_op(op);
    if (!cpp_op || !out_indices || !out_count) {
        if (out_indices) *out_indices = nullptr;
        if (out_count) *out_count = 0;
        return -1;
    }
    *out_indices = nullptr;
    *out_count = 0;
    try {
        std::vector<int> cpp_indices = cpp_op->bootstrapping_key_indexs();
        *out_count = cpp_indices.size();
        if (*out_count > 0) {
            *out_indices = static_cast<int*>(malloc(*out_count * sizeof(int)));
            if (!*out_indices) {
                *out_count = 0;
                std::cerr << "GetBootstrappingKeyIndices: malloc failed.\n";
                return -2;
            }
            std::memcpy(*out_indices, cpp_indices.data(), *out_count * sizeof(int));
        }
        return 0;
    } catch (const std::exception& e) {
        std::cerr << "GetBootstrappingKeyIndices failed with exception: " << e.what() << std::endl;
        if (*out_indices) { free(*out_indices); *out_indices = nullptr; }
        *out_count = 0;
        return -3;
    } catch (...) {
        std::cerr << "GetBootstrappingKeyIndices failed with unknown exception." << std::endl;
        if (*out_indices) { free(*out_indices); *out_indices = nullptr; }
        *out_count = 0;
        return -3;
    }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_RegularBootstrapping(HE_CKKS_ArithmeticOperator* op,
                                                                         HE_CKKS_Ciphertext* ct_in_c,
                                                                         HE_CKKS_GaloisKey* galois_key_c,
                                                                         HE_CKKS_RelinKey* relin_key_c,
                                                                         const C_ExecutionOptions* options_c) {
    heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>* cpp_op = get_cpp_arith_op(op);
    if (!cpp_op || !ct_in_c || !galois_key_c || !relin_key_c) {
        std::cerr << "RegularBootstrapping Error: Invalid argument(s).\n";
        return nullptr;
    }

    heongpu::Ciphertext<heongpu::Scheme::CKKS>* cpp_ct_in = get_cpp_ciphertext(ct_in_c);
    heongpu::Galoiskey<heongpu::Scheme::CKKS>* cpp_gk = get_cpp_galoiskey(galois_key_c);
    heongpu::Relinkey<heongpu::Scheme::CKKS>* cpp_rk = get_cpp_relinkey(relin_key_c);

    if (!cpp_ct_in || !cpp_gk || !cpp_rk) {
        std::cerr << "RegularBootstrapping Error: Failed to unwrap C API handles.\n";
        return nullptr;
    }
    
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options(options_c);
        heongpu::Ciphertext<heongpu::Scheme::CKKS> cpp_result_ct =
            cpp_op->regular_bootstrapping(*cpp_ct_in, *cpp_gk, *cpp_rk, cpp_options);

        // Wrap the returned C++ object in a new C API handle
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result_ct));
        if (!cpp_heap_result) return nullptr;

        HE_CKKS_Ciphertext* c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;

    } catch (const std::exception& e) {
        std::cerr << "RegularBootstrapping failed with exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "RegularBootstrapping failed with unknown exception." << std::endl;
        return nullptr;
    }
}



} // extern "C"