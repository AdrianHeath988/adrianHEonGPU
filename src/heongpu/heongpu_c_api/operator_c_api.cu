#include "operator_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/encoder.cuh"
#include "ckks/plaintext.cuh"
#include "ckks/ciphertext.cuh"
#include "ckks/evaluationkey.cuh"
#include "ckks/operator.cuh" // The C++ classes we are wrapping

#include "util/storagemanager.cuh" // For heongpu::ExecutionOptions

#include <vector>
#include <iostream> // For error logging
#include <new>      // For std::nothrow


// Define opaque structs
struct HE_CKKS_ArithmeticOperator_s {
    heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>* cpp_arith_op;
};

struct HE_CKKS_LogicOperator_s {
    heongpu::HELogicOperator<heongpu::Scheme::CKKS>* cpp_logic_op;
};

// Helper to safely access underlying C++ pointers from opaque C pointers
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) return nullptr;
    return context->cpp_context;
}
static heongpu::HEEncoder<heongpu::Scheme::CKKS>* get_cpp_encoder(HE_CKKS_Encoder* encoder) {
    if (!encoder || !encoder->cpp_encoder) return nullptr;
    return encoder->cpp_encoder;
}
static heongpu::Plaintext<heongpu::Scheme::CKKS>* get_cpp_plaintext(HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static const heongpu::Plaintext<heongpu::Scheme::CKKS>* get_const_cpp_plaintext(const HE_CKKS_Plaintext* pt) {
    if (!pt || !pt->cpp_plaintext) return nullptr;
    return pt->cpp_plaintext;
}
static heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_cpp_ciphertext(HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static const heongpu::Ciphertext<heongpu::Scheme::CKKS>* get_const_cpp_ciphertext(const HE_CKKS_Ciphertext* ct) {
    if (!ct || !ct->cpp_ciphertext) return nullptr;
    return ct->cpp_ciphertext;
}
static heongpu::Relinkey<heongpu::Scheme::CKKS>* get_cpp_relinkey(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return nullptr;
    return rk->cpp_relinkey;
}
static heongpu::Galoiskey<heongpu::Scheme::CKKS>* get_cpp_galoiskey(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    return gk->cpp_galoiskey;
}


// Helper to map C types to C++ ExecutionOptions
static heongpu::ExecutionOptions map_c_to_cpp_execution_options_op(const C_ExecutionOptions* c_options) {
    heongpu::ExecutionOptions cpp_options; // Defaults from C++ struct definition
    if (c_options) {
        cpp_options.stream_ = static_cast<hipStream_t>(c_options->stream);
        if (c_options->storage == C_STORAGE_TYPE_HOST) {
            cpp_options.storage_ = heongpu::storage_type::HOST;
        } else if (c_options->storage == C_STORAGE_TYPE_DEVICE) {
            cpp_options.storage_ = heongpu::storage_type::DEVICE;
        }
        // If C_STORAGE_TYPE_INVALID or other, it uses default from cpp_options.
        cpp_options.keep_initial_condition_ = c_options->keep_initial_condition;
    }
    return cpp_options;
}


extern "C" {

// --- CKKS HEArithmeticOperator Lifecycle ---
HE_CKKS_ArithmeticOperator* HEonGPU_CKKS_ArithmeticOperator_Create(HE_CKKS_Context* context, HE_CKKS_Encoder* encoder) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_h_encoder = get_cpp_encoder(encoder);
    if (!cpp_h_context || !cpp_h_encoder) {
        std::cerr << "ArithmeticOperator_Create: Invalid context or encoder." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_h_encoder);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_ArithmeticOperator_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_arith_op = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "ArithmeticOperator_Create Error: " << e.what() << std::endl; return nullptr;} 
      catch (...) { std::cerr << "ArithmeticOperator_Create Unknown Error" << std::endl; return nullptr;}
}

void HEonGPU_CKKS_ArithmeticOperator_Delete(HE_CKKS_ArithmeticOperator* op) {
    if (op) { delete op->cpp_arith_op; delete op; }
}

// --- CKKS HEArithmeticOperator Operations ---

// Addition Example (In-place)
void HEonGPU_CKKS_ArithmeticOperator_Add_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (const std::exception& e) { std::cerr << "Add_Inplace Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "Add_Inplace Unknown Error" << std::endl;}
}

// Addition Example (Out-of-place, C API returns new object)
HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) {
        std::cerr << "Add: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        // The C++ `add` method returns a new Ciphertext object
        heongpu::Ciphertext<heongpu::Scheme::CKKS> cpp_result_ct = 
            op->cpp_arith_op->add(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
        
        // Wrap the C++ result in a C API object
        auto cpp_result_on_heap = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result_ct));
        if (!cpp_result_on_heap) { std::cerr << "Add: Failed to allocate C++ result.\n"; return nullptr; }

        HE_CKKS_Ciphertext* c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_result_on_heap; std::cerr << "Add: Failed to allocate C API wrapper.\n"; return nullptr; }
        
        c_api_result->cpp_ciphertext = cpp_result_on_heap;
        return c_api_result;
    } catch (const std::exception& e) { std::cerr << "Add Error: " << e.what() << std::endl; return nullptr; }
      catch (...) { std::cerr << "Add Unknown Error" << std::endl; return nullptr; }
}

// Add_Plain_Inplace
void HEonGPU_CKKS_ArithmeticOperator_Add_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain_Inplace: Invalid argument(s).\n"; return;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->add_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (const std::exception& e) { std::cerr << "Add_Plain_Inplace Error: " << e.what() << std::endl; }
      catch (...) { std::cerr << "Add_Plain_Inplace Unknown Error" << std::endl;}
}

// Add_Plain (returns new Ciphertext)
HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Add_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) {
        std::cerr << "Add_Plain: Invalid argument(s).\n"; return nullptr;
    }
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        heongpu::Ciphertext<heongpu::Scheme::CKKS> cpp_result_ct = 
            op->cpp_arith_op->add_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
        
        auto cpp_result_on_heap = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result_ct));
        if (!cpp_result_on_heap) return nullptr;
        HE_CKKS_Ciphertext* c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_result_on_heap; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_result_on_heap;
        return c_api_result;
    } catch (const std::exception& e) { std::cerr << "Add_Plain Error: " << e.what() << std::endl; return nullptr;}
      catch (...) { std::cerr << "Add_Plain Unknown Error" << std::endl; return nullptr;}
}


// --- Implementations for Subtraction (similar to Addition) ---
void HEonGPU_CKKS_ArithmeticOperator_Sub_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->sub_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Sub_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->sub_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Sub(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->sub(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// --- Negation ---
void HEonGPU_CKKS_ArithmeticOperator_Negate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->negate_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Negate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->negate(*(ct_in->cpp_ciphertext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// --- Implementations for Multiplication (Pattern: check args, map options, call C++ method, wrap result if new object) ---
void HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_plain_inplace(*(ct_in_out->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply_Plain(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const HE_CKKS_Plaintext* pt_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !pt_in || !pt_in->cpp_plaintext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->multiply_plain(*(ct_in->cpp_ciphertext), *(pt_in->cpp_plaintext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Multiply_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct1_in_out, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in_out || !ct1_in_out->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->multiply_inplace(*(ct1_in_out->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Multiply(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct1_in, const HE_CKKS_Ciphertext* ct2_in, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct1_in || !ct1_in->cpp_ciphertext || !ct2_in || !ct2_in->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->multiply(*(ct1_in->cpp_ciphertext), *(ct2_in->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// --- Square ---
void HEonGPU_CKKS_ArithmeticOperator_Square_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->square_inplace(*(ct_in_out->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Square(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->square(*(ct_in->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// --- Relinearize ---
void HEonGPU_CKKS_ArithmeticOperator_Relinearize_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !relin_key_c || !relin_key_c->cpp_relinkey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->relinearize_inplace(*(ct_in_out->cpp_ciphertext), *(relin_key_c->cpp_relinkey), cpp_options);
    } catch (...) { /* error handling */ }
}

// --- ModDrop / Rescale ---
void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_ModDrop_Ciphertext(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->mod_drop(*(ct_in->cpp_ciphertext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_ModDrop_Plaintext_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Plaintext* pt_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !pt_in_out || !pt_in_out->cpp_plaintext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->mod_drop_inplace(*(pt_in_out->cpp_plaintext), cpp_options);
    } catch (...) { /* error handling */ }
}


void HEonGPU_CKKS_ArithmeticOperator_Rescale_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rescale_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Rescale(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->rescale(*(ct_in->cpp_ciphertext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// --- Rotation / Conjugation ---
void HEonGPU_CKKS_ArithmeticOperator_Rotate_Left_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_left_inplace(*(ct_in_out->cpp_ciphertext), steps, *(galois_key_c->cpp_galoiskey), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Rotate_Left(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->rotate_left(*(ct_in->cpp_ciphertext), steps, *(galois_key_c->cpp_galoiskey), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// (Rotate_Right and Conjugate functions follow the same pattern as Rotate_Left)
void HEonGPU_CKKS_ArithmeticOperator_Rotate_Right_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->rotate_right_inplace(*(ct_in_out->cpp_ciphertext), steps, *(galois_key_c->cpp_galoiskey), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Rotate_Right(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, int steps, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->rotate_right(*(ct_in->cpp_ciphertext), steps, *(galois_key_c->cpp_galoiskey), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_ArithmeticOperator_Conjugate_Inplace(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_out, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in_out || !ct_in_out->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_arith_op->conjugate_inplace(*(ct_in_out->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_ArithmeticOperator_Conjugate(HE_CKKS_ArithmeticOperator* op, const HE_CKKS_Ciphertext* ct_in, HE_CKKS_GaloisKey* galois_key_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_arith_op || !ct_in || !ct_in->cpp_ciphertext || !galois_key_c || !galois_key_c->cpp_galoiskey) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_arith_op->conjugate(*(ct_in->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}


// --- Bootstrapping ---
// Note: C++ bootstrap methods return new Ciphertext objects.
#define WRAP_BOOTSTRAP_FUNC(FuncName, CppFuncName) \
HE_CKKS_Ciphertext* FuncName(HE_CKKS_ArithmeticOperator* op, HE_CKKS_Ciphertext* ct_in_c, HE_CKKS_RelinKey* relin_key_c, HE_CKKS_GaloisKey* galois_key_conj_c, HE_CKKS_GaloisKey* galois_key_rot_c, const C_ExecutionOptions* options_c) { \
    if (!op || !op->cpp_arith_op || !ct_in_c || !ct_in_c->cpp_ciphertext || \
        !relin_key_c || !relin_key_c->cpp_relinkey || \
        !galois_key_conj_c || !galois_key_conj_c->cpp_galoiskey || \
        !galois_key_rot_c || !galois_key_rot_c->cpp_galoiskey) { \
        std::cerr << #FuncName " Error: Invalid argument(s).\n"; return nullptr; \
    } \
    try { \
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
        auto cpp_result = op->cpp_arith_op->CppFuncName(*(ct_in_c->cpp_ciphertext), *(relin_key_c->cpp_relinkey), *(galois_key_conj_c->cpp_galoiskey), *(galois_key_rot_c->cpp_galoiskey), cpp_options); \
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result)); \
        if (!cpp_heap_result) return nullptr; \
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s; \
        if (!c_api_result) { delete cpp_heap_result; return nullptr; } \
        c_api_result->cpp_ciphertext = cpp_heap_result; \
        return c_api_result; \
    } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return nullptr; } \
      catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return nullptr; } \
}

WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap, bootstrap)
WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Slim, bootstrap_slim)
WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Bit, bootstrap_bit)
WRAP_BOOTSTRAP_FUNC(HEonGPU_CKKS_ArithmeticOperator_Bootstrap_Gate, bootstrap_gate)


// --- CKKS HELogicOperator Lifecycle ---
HE_CKKS_LogicOperator* HEonGPU_CKKS_LogicOperator_Create(HE_CKKS_Context* context, HE_CKKS_Encoder* encoder) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    heongpu::HEEncoder<heongpu::Scheme::CKKS>* cpp_h_encoder = get_cpp_encoder(encoder);
     if (!cpp_h_context || !cpp_h_encoder) {
        std::cerr << "LogicOperator_Create: Invalid context or encoder." << std::endl;
        return nullptr;
    }
    try {
        auto cpp_obj = new (std::nothrow) heongpu::HELogicOperator<heongpu::Scheme::CKKS>(*cpp_h_context, *cpp_h_encoder);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_LogicOperator_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_logic_op = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "LogicOperator_Create Error: " << e.what() << std::endl; return nullptr;}
      catch (...) { std::cerr << "LogicOperator_Create Unknown Error" << std::endl; return nullptr;}
}

void HEonGPU_CKKS_LogicOperator_Delete(HE_CKKS_LogicOperator* op) {
    if (op) { delete op->cpp_logic_op; delete op; }
}

// --- CKKS HELogicOperator Operations ---
void HEonGPU_CKKS_LogicOperator_NOT_Approximation_Inplace(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct_in_out, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_logic_op || !ct_in_out || !ct_in_out->cpp_ciphertext) return;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        op->cpp_logic_op->NOT_approximation_inplace(*(ct_in_out->cpp_ciphertext), cpp_options);
    } catch (...) { /* error handling */ }
}

HE_CKKS_Ciphertext* HEonGPU_CKKS_LogicOperator_NOT_Approximation(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct_in_c, const C_ExecutionOptions* options_c) {
    if (!op || !op->cpp_logic_op || !ct_in_c || !ct_in_c->cpp_ciphertext) return nullptr;
    try {
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c);
        auto cpp_result = op->cpp_logic_op->NOT_approximation(*(ct_in_c->cpp_ciphertext), cpp_options);
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result));
        if (!cpp_heap_result) return nullptr;
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s;
        if (!c_api_result) { delete cpp_heap_result; return nullptr; }
        c_api_result->cpp_ciphertext = cpp_heap_result;
        return c_api_result;
    } catch (...) { return nullptr; }
}

// (XOR and XNOR functions follow a similar pattern, taking GaloisKey and RelinKey)
#define WRAP_LOGIC_BINARY_OP_INPLACE(FuncName, CppFuncName) \
void FuncName(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct1_in_out_c, HE_CKKS_Ciphertext* ct2_in_c, HE_CKKS_GaloisKey* galois_key_c, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) { \
    if (!op || !op->cpp_logic_op || !ct1_in_out_c || !ct1_in_out_c->cpp_ciphertext || \
        !ct2_in_c || !ct2_in_c->cpp_ciphertext || \
        !galois_key_c || !galois_key_c->cpp_galoiskey || \
        !relin_key_c || !relin_key_c->cpp_relinkey) { \
        std::cerr << #FuncName " Error: Invalid argument(s).\n"; return; \
    } \
    try { \
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
        op->cpp_logic_op->CppFuncName(*(ct1_in_out_c->cpp_ciphertext), *(ct2_in_c->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), *(relin_key_c->cpp_relinkey), cpp_options); \
    } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; } \
      catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; } \
}

#define WRAP_LOGIC_BINARY_OP_NEW(FuncName, CppFuncName) \
HE_CKKS_Ciphertext* FuncName(HE_CKKS_LogicOperator* op, HE_CKKS_Ciphertext* ct1_in_c, HE_CKKS_Ciphertext* ct2_in_c, HE_CKKS_GaloisKey* galois_key_c, HE_CKKS_RelinKey* relin_key_c, const C_ExecutionOptions* options_c) { \
    if (!op || !op->cpp_logic_op || !ct1_in_c || !ct1_in_c->cpp_ciphertext || \
        !ct2_in_c || !ct2_in_c->cpp_ciphertext || \
        !galois_key_c || !galois_key_c->cpp_galoiskey || \
        !relin_key_c || !relin_key_c->cpp_relinkey) { \
        std::cerr << #FuncName " Error: Invalid argument(s).\n"; return nullptr; \
    } \
    try { \
        heongpu::ExecutionOptions cpp_options = map_c_to_cpp_execution_options_op(options_c); \
        auto cpp_result = op->cpp_logic_op->CppFuncName(*(ct1_in_c->cpp_ciphertext), *(ct2_in_c->cpp_ciphertext), *(galois_key_c->cpp_galoiskey), *(relin_key_c->cpp_relinkey), cpp_options); \
        auto cpp_heap_result = new (std::nothrow) heongpu::Ciphertext<heongpu::Scheme::CKKS>(std::move(cpp_result)); \
        if (!cpp_heap_result) return nullptr; \
        auto c_api_result = new (std::nothrow) HE_CKKS_Ciphertext_s; \
        if (!c_api_result) { delete cpp_heap_result; return nullptr; } \
        c_api_result->cpp_ciphertext = cpp_heap_result; \
        return c_api_result; \
    } catch (const std::exception& e) { std::cerr << #FuncName " Error: " << e.what() << std::endl; return nullptr; } \
      catch (...) { std::cerr << #FuncName " Unknown Error" << std::endl; return nullptr; } \
}

WRAP_LOGIC_BINARY_OP_INPLACE(HEonGPU_CKKS_LogicOperator_XOR_Approximation_Inplace, XOR_approximation_inplace)
WRAP_LOGIC_BINARY_OP_NEW(HEonGPU_CKKS_LogicOperator_XOR_Approximation, XOR_approximation)
WRAP_LOGIC_BINARY_OP_INPLACE(HEonGPU_CKKS_LogicOperator_XNOR_Approximation_Inplace, XNOR_approximation_inplace)
WRAP_LOGIC_BINARY_OP_NEW(HEonGPU_CKKS_LogicOperator_XNOR_Approximation, XNOR_approximation)


} // extern "C"