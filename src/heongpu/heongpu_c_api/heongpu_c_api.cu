#include "hip/hip_runtime.h"
#include "heongpu_c_api.h"
#include "heongpu.cuh" // Main HEonGPU include, provides access to BFV/CKKS context etc.

// Define the opaque struct BFVContext_s to actually hold the HEonGPU context
// This definition is hidden from the C header users.
struct BFVContext_s {
    heongpu::HEContext<heongpu::Scheme::BFV>* cpp_context;
};

extern "C" {

BFVContext* HEonGPU_BFV_Context_Create(int keyswitch_method_int, 
                                       size_t poly_modulus_degree,
                                       int plain_modulus) {
    try {
        // Map integer to heongpu::keyswitching_type
        // For simplicity, let's assume 0 = METHOD_I, 1 = METHOD_HYBRID (BV), 2 = METHOD_GHS
        // This should be properly documented and managed.
        heongpu::keyswitching_type ks_type;
        switch (keyswitch_method_int) {
            case 0: ks_type = heongpu::keyswitching_type::KEYSWITCHING_METHOD_I; break;
            case 1: ks_type = heongpu::keyswitching_type::KEYSWITCHING_METHOD_HYBRID; break;
            case 2: ks_type = heongpu::keyswitching_type::KEYSWITCHING_METHOD_GHS; break;
            default: return nullptr; // Invalid type
        }

        heongpu::HEContext<heongpu::Scheme::BFV>* cpp_ctx = 
            new heongpu::HEContext<heongpu::Scheme::BFV>(ks_type);

        cpp_ctx->set_poly_modulus_degree(poly_modulus_degree);
        // For BFV, set_coeff_modulus_default_values takes the number of primes.
        // The original README example used 1. Let's stick to that for simplicity for now.
        // A more robust C API might take num_primes as a parameter.
        cpp_ctx->set_coeff_modulus_default_values(1); 
        cpp_ctx->set_plain_modulus(plain_modulus);
        // Note: The 'generate()' from the README example is split into specific setters and
        // a final generate call. Here, we assume parameters are set, and GenerateParams will finalize.

        BFVContext* c_api_context = new BFVContext;
        c_api_context->cpp_context = cpp_ctx;
        return c_api_context;

    } catch (...) {
        // In a real scenario, log the error
        return nullptr;
    }
}

void HEonGPU_BFV_Context_GenerateParams(BFVContext* context) {
    if (context && context->cpp_context) {
        try {
            context->cpp_context->generate();
        } catch (...) {
            // Handle or log error
        }
    }
}

size_t HEonGPU_BFV_Context_GetPolyModulusDegree(BFVContext* context) {
    if (context && context->cpp_context) {
        try {
            return context->cpp_context->get_poly_modulus_degree();
        } catch (...) {
            // Handle or log error
            return 0; 
        }
    }
    return 0; // Invalid context
}

void HEonGPU_BFV_Context_Delete(BFVContext* context) {
    if (context) {
        delete context->cpp_context; // Delete the C++ object
        delete context;             // Delete the C API struct
    }
}

} // extern "C"