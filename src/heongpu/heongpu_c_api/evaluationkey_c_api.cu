#include "hip/hip_runtime.h"
#include "evaluationkey_c_api.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/evaluationkey.cuh" // The C++ classes we are wrapping
#include "keygeneration.cuh"      // For heongpu::RotationIndices
#include "hostvector.cuh"
#include "schemes.h"
#include "storagemanager.cuh"

#include <vector>
#include <sstream>
#include <iostream>
#include <algorithm> // For std::min
#include <cstring>   // For std::memcpy
#include <new>       // For std::nothrow

// Define opaque structs

typedef struct HE_CKKS_RelinKey_s HE_CKKS_RelinKey;
typedef struct HE_CKKS_MultipartyRelinKey_s HE_CKKS_MultipartyRelinKey;
typedef struct HE_CKKS_GaloisKey_s HE_CKKS_GaloisKey;


// Helper to safely access underlying C++ HEContext pointer
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}

// Helper C++ enums to C enums
static C_scheme_type map_cpp_to_c_scheme_type_evk(heongpu::scheme_type cpp_type) {
    switch (cpp_type) {
        case heongpu::scheme_type::none: return C_SCHEME_TYPE_NONE;
        case heongpu::scheme_type::bfv:  return C_SCHEME_TYPE_BFV;
        case heongpu::scheme_type::ckks: return C_SCHEME_TYPE_CKKS;
        case heongpu::scheme_type::bgv:  return C_SCHEME_TYPE_BGV;
        default: return static_cast<C_scheme_type>(-1); 
    }
}

static C_keyswitching_type map_cpp_to_c_keyswitch_type_evk(heongpu::keyswitching_type cpp_type) {
    switch (cpp_type) {
        case heongpu::keyswitching_type::NONE:                 return C_KEYSWITCHING_TYPE_NONE;
        case heongpu::keyswitching_type::KEYSWITCHING_METHOD_I:  return C_KEYSWITCHING_TYPE_METHOD_I;
        case heongpu::keyswitching_type::KEYSWITCHING_METHOD_II: return C_KEYSWITCHING_TYPE_METHOD_II;
        case heongpu::keyswitching_type::KEYSWITCHING_METHOD_III:return C_KEYSWITCHING_TYPE_METHOD_III;
        default: return C_KEYSWITCHING_TYPE_INVALID;
    }
}

static C_storage_type map_cpp_to_c_storage_type_evk(heongpu::storage_type cpp_type) {
    switch (cpp_type) {
        case heongpu::storage_type::HOST:   return C_STORAGE_TYPE_HOST;
        case heongpu::storage_type::DEVICE: return C_STORAGE_TYPE_DEVICE;
        default: return C_STORAGE_TYPE_INVALID;
    }
}


extern "C" {

// --- CKKS RelinKey Functions ---
HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Create(HE_CKKS_Context* context, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    try {
        auto cpp_obj = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*cpp_h_context, store_in_gpu);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_RelinKey_Delete(HE_CKKS_RelinKey* rk) {
    if (rk) { delete rk->cpp_relinkey; delete rk; }
}

HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Clone(const HE_CKKS_RelinKey* other_rk) {
    if (!other_rk || !other_rk->cpp_relinkey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*(other_rk->cpp_relinkey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_relinkey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_RelinKey_Assign_Copy(HE_CKKS_RelinKey* dest_rk, const HE_CKKS_RelinKey* src_rk) {
    if (!dest_rk || !dest_rk->cpp_relinkey || !src_rk || !src_rk->cpp_relinkey) return -1;
    try {
        *(dest_rk->cpp_relinkey) = *(src_rk->cpp_relinkey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_RelinKey_Save(HE_CKKS_RelinKey* rk, unsigned char** out_bytes, size_t* out_len) {
    if (!rk || !rk->cpp_relinkey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        rk->cpp_relinkey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; std::cerr << "RelinKey_Save: malloc failed\n"; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (const std::exception& e) { std::cerr << "RelinKey_Save exception: " << e.what() << std::endl; if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
      catch (...) { std::cerr << "RelinKey_Save unknown exception" << std::endl; if(*out_bytes){free(*out_bytes);*out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
    heongpu::Relinkey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_RelinKey* c_api_obj = nullptr;
    try {
        cpp_obj = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*cpp_h_context, store_in_gpu_on_load);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss);
        }
        c_api_obj = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "RelinKey_Load exception: " << e.what() << std::endl; delete cpp_obj; delete c_api_obj; return nullptr; }
      catch (...) { std::cerr << "RelinKey_Load unknown exception" << std::endl; delete cpp_obj; delete c_api_obj; return nullptr; }
}

C_scheme_type HEonGPU_CKKS_RelinKey_GetScheme(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_evk(rk->cpp_relinkey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}
C_keyswitching_type HEonGPU_CKKS_RelinKey_GetKeyswitchType(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return C_KEYSWITCHING_TYPE_INVALID;
    try { return map_cpp_to_c_keyswitch_type_evk(rk->cpp_relinkey->get_keyswitch_type()); } catch (...) { return C_KEYSWITCHING_TYPE_INVALID; }
}
int HEonGPU_CKKS_RelinKey_GetRingSize(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return 0;
    try { return rk->cpp_relinkey->ring_size_nk(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_RelinKey_GetQPrimeSize(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return 0;
    try { return rk->cpp_relinkey->Q_prime_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_RelinKey_GetQSize(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return 0;
    try { return rk->cpp_relinkey->Q_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_RelinKey_GetDFactor(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return 0;
    try { return rk->cpp_relinkey->d_factor(); } catch (...) { return 0; }
}
bool HEonGPU_CKKS_RelinKey_IsGenerated(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return false;
    try { return rk->cpp_relinkey->is_generated(); } catch (...) { return false; }
}
C_storage_type HEonGPU_CKKS_RelinKey_GetStorageType(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_evk(rk->cpp_relinkey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
size_t HEonGPU_CKKS_RelinKey_GetData(HE_CKKS_RelinKey* rk, uint64_t* data_buffer, size_t buffer_elements, C_cudaStream_t stream) {
    if (!rk || !rk->cpp_relinkey || (!data_buffer && buffer_elements > 0)) return 0;
    try {
        heongpu::HostVector<heongpu::Data64> temp_hv;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        rk->cpp_relinkey->get_data(temp_hv, cpp_stream);
        size_t count = std::min(buffer_elements, temp_hv.size());
        if (count > 0 && data_buffer) std::memcpy(data_buffer, temp_hv.data(), count * sizeof(uint64_t));
        return count;
    } catch (...) { return 0; }
}
int HEonGPU_CKKS_RelinKey_SetData(HE_CKKS_RelinKey* rk, const uint64_t* data_buffer, size_t num_elements, C_cudaStream_t stream) {
    if (!rk || !rk->cpp_relinkey || (!data_buffer && num_elements > 0)) return -1;
    try {
        heongpu::HostVector<heongpu::Data64> input_hv(num_elements);
        if (num_elements > 0 && data_buffer) std::memcpy(input_hv.data(), data_buffer, num_elements * sizeof(uint64_t));
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        rk->cpp_relinkey->set_data(input_hv, cpp_stream);
        return 0;
    } catch (...) { return -2; }
}

// --- CKKS MultipartyRelinKey Functions ---
HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Create(HE_CKKS_Context* context, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    try {
        auto cpp_obj = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*cpp_h_context, store_in_gpu);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_mp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_MultipartyRelinKey_Delete(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (mp_rk) { delete mp_rk->cpp_mp_relinkey; delete mp_rk; }
}

HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Clone(const HE_CKKS_MultipartyRelinKey* other_mp_rk) {
    if (!other_mp_rk || !other_mp_rk->cpp_mp_relinkey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*(other_mp_rk->cpp_mp_relinkey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_mp_relinkey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_MultipartyRelinKey_Assign_Copy(HE_CKKS_MultipartyRelinKey* dest_mp_rk, const HE_CKKS_MultipartyRelinKey* src_mp_rk) {
    if (!dest_mp_rk || !dest_mp_rk->cpp_mp_relinkey || !src_mp_rk || !src_mp_rk->cpp_mp_relinkey) return -1;
    try {
        *(dest_mp_rk->cpp_mp_relinkey) = *(src_mp_rk->cpp_mp_relinkey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_MultipartyRelinKey_Save(HE_CKKS_MultipartyRelinKey* mp_rk, unsigned char** out_bytes, size_t* out_len) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        mp_rk->cpp_mp_relinkey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (...) { if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
    heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_MultipartyRelinKey* c_api_obj = nullptr;
    try {
        cpp_obj = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*cpp_h_context, store_in_gpu_on_load);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss);
        }
        c_api_obj = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_mp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (...) { delete cpp_obj; delete c_api_obj; return nullptr; }
}

C_scheme_type HEonGPU_CKKS_MultipartyRelinKey_GetScheme(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_evk(mp_rk->cpp_mp_relinkey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}
C_keyswitching_type HEonGPU_CKKS_MultipartyRelinKey_GetKeyswitchType(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return C_KEYSWITCHING_TYPE_INVALID;
    try { return map_cpp_to_c_keyswitch_type_evk(mp_rk->cpp_mp_relinkey->get_keyswitch_type()); } catch (...) { return C_KEYSWITCHING_TYPE_INVALID; }
}
int HEonGPU_CKKS_MultipartyRelinKey_GetRingSize(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return 0;
    try { return mp_rk->cpp_mp_relinkey->ring_size_nk(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_MultipartyRelinKey_GetQPrimeSize(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return 0;
    try { return mp_rk->cpp_mp_relinkey->Q_prime_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_MultipartyRelinKey_GetQSize(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return 0;
    try { return mp_rk->cpp_mp_relinkey->Q_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_MultipartyRelinKey_GetDFactor(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return 0;
    try { return mp_rk->cpp_mp_relinkey->d_factor(); } catch (...) { return 0; }
}
bool HEonGPU_CKKS_MultipartyRelinKey_IsGenerated(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return false;
    try { return mp_rk->cpp_mp_relinkey->is_generated(); } catch (...) { return false; }
}
C_storage_type HEonGPU_CKKS_MultipartyRelinKey_GetStorageType(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_evk(mp_rk->cpp_mp_relinkey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
size_t HEonGPU_CKKS_MultipartyRelinKey_GetData(HE_CKKS_MultipartyRelinKey* mp_rk, uint64_t* data_buffer, size_t buffer_elements, C_cudaStream_t stream) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey || (!data_buffer && buffer_elements > 0)) return 0;
    try {
        heongpu::HostVector<heongpu::Data64> temp_hv;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        mp_rk->cpp_mp_relinkey->get_data(temp_hv, cpp_stream);
        size_t count = std::min(buffer_elements, temp_hv.size());
        if (count > 0 && data_buffer) std::memcpy(data_buffer, temp_hv.data(), count * sizeof(uint64_t));
        return count;
    } catch (...) { return 0; }
}
int HEonGPU_CKKS_MultipartyRelinKey_SetData(HE_CKKS_MultipartyRelinKey* mp_rk, const uint64_t* data_buffer, size_t num_elements, C_cudaStream_t stream) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey || (!data_buffer && num_elements > 0)) return -1;
    try {
        heongpu::HostVector<heongpu::Data64> input_hv(num_elements);
        if (num_elements > 0 && data_buffer) std::memcpy(input_hv.data(), data_buffer, num_elements * sizeof(uint64_t));
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        mp_rk->cpp_mp_relinkey->set_data(input_hv, cpp_stream);
        return 0;
    } catch (...) { return -2; }
}


// --- CKKS GaloisKey Functions ---
HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Create(HE_CKKS_Context* context, const C_RotationIndices_Const_Data* rot_indices_c, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context || !rot_indices_c) return nullptr;
    try {
        heongpu::RotationIndices cpp_rot_indices;
        if (rot_indices_c->galois_elements_data && rot_indices_c->galois_elements_len > 0) {
            cpp_rot_indices.galois_elements.assign(rot_indices_c->galois_elements_data, rot_indices_c->galois_elements_data + rot_indices_c->galois_elements_len);
        }
        if (rot_indices_c->rotation_steps_data && rot_indices_c->rotation_steps_len > 0) {
            cpp_rot_indices.rotation_steps.assign(rot_indices_c->rotation_steps_data, rot_indices_c->rotation_steps_data + rot_indices_c->rotation_steps_len);
        }

        auto cpp_obj = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_rot_indices, store_in_gpu);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_galoiskey = cpp_obj;
        return c_api_obj;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_GaloisKey_Delete(HE_CKKS_GaloisKey* gk) {
    if (gk) { delete gk->cpp_galoiskey; delete gk; }
}

HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Clone(const HE_CKKS_GaloisKey* other_gk) {
    if (!other_gk || !other_gk->cpp_galoiskey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*(other_gk->cpp_galoiskey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_galoiskey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_GaloisKey_Assign_Copy(HE_CKKS_GaloisKey* dest_gk, const HE_CKKS_GaloisKey* src_gk) {
    if (!dest_gk || !dest_gk->cpp_galoiskey || !src_gk || !src_gk->cpp_galoiskey) return -1;
    try {
        *(dest_gk->cpp_galoiskey) = *(src_gk->cpp_galoiskey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_GaloisKey_Save(HE_CKKS_GaloisKey* gk, unsigned char** out_bytes, size_t* out_len) {
    if (!gk || !gk->cpp_galoiskey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        gk->cpp_galoiskey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (...) { if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, const C_RotationIndices_Const_Data* rot_indices_for_reconstruction, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
    // RotationIndices are crucial for GaloisKey constructor.
    // If they are part of the serialized stream, this load process needs to match how save works.
    // The C++ load method for Galoiskey in evaluationkey.cu loads rot_indices_ itself.
    // So, for construction before load, we can pass default/empty rot_indices.
    heongpu::RotationIndices temp_rot_indices;
     if (rot_indices_for_reconstruction) { // Use if provided, primarily for clarity if load doesn't fully init this.
        if (rot_indices_for_reconstruction->galois_elements_data && rot_indices_for_reconstruction->galois_elements_len > 0) {
            temp_rot_indices.galois_elements.assign(rot_indices_for_reconstruction->galois_elements_data, rot_indices_for_reconstruction->galois_elements_data + rot_indices_for_reconstruction->galois_elements_len);
        }
        if (rot_indices_for_reconstruction->rotation_steps_data && rot_indices_for_reconstruction->rotation_steps_len > 0) {
            temp_rot_indices.rotation_steps.assign(rot_indices_for_reconstruction->rotation_steps_data, rot_indices_for_reconstruction->rotation_steps_data + rot_indices_for_reconstruction->rotation_steps_len);
        }
    }


    heongpu::Galoiskey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_GaloisKey* c_api_obj = nullptr;
    try {
        cpp_obj = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*cpp_h_context, temp_rot_indices, store_in_gpu_on_load);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss); // This should load the actual rot_indices_ from the stream
        }
        c_api_obj = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_galoiskey = cpp_obj;
        return c_api_obj;
    } catch (...) { delete cpp_obj; delete c_api_obj; return nullptr; }
}

C_scheme_type HEonGPU_CKKS_GaloisKey_GetScheme(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return static_cast<C_scheme_type>(-1);
    try { return map_cpp_to_c_scheme_type_evk(gk->cpp_galoiskey->get_scheme()); } catch (...) { return static_cast<C_scheme_type>(-1); }
}
C_keyswitching_type HEonGPU_CKKS_GaloisKey_GetKeyswitchType(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return C_KEYSWITCHING_TYPE_INVALID;
    try { return map_cpp_to_c_keyswitch_type_evk(gk->cpp_galoiskey->get_keyswitch_type()); } catch (...) { return C_KEYSWITCHING_TYPE_INVALID; }
}
int HEonGPU_CKKS_GaloisKey_GetRingSize(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return 0;
    try { return gk->cpp_galoiskey->ring_size_nk(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_GaloisKey_GetQPrimeSize(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return 0;
    try { return gk->cpp_galoiskey->Q_prime_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_GaloisKey_GetQSize(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return 0;
    try { return gk->cpp_galoiskey->Q_size(); } catch (...) { return 0; }
}
int HEonGPU_CKKS_GaloisKey_GetDFactor(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return 0;
    try { return gk->cpp_galoiskey->d_factor(); } catch (...) { return 0; }
}
bool HEonGPU_CKKS_GaloisKey_IsGenerated(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return false;
    try { return gk->cpp_galoiskey->is_generated(); } catch (...) { return false; }
}
C_storage_type HEonGPU_CKKS_GaloisKey_GetStorageType(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return C_STORAGE_TYPE_INVALID;
    try { return map_cpp_to_c_storage_type_evk(gk->cpp_galoiskey->get_storage_type()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
size_t HEonGPU_CKKS_GaloisKey_GetData(HE_CKKS_GaloisKey* gk, uint64_t* data_buffer, size_t buffer_elements, C_cudaStream_t stream) {
    if (!gk || !gk->cpp_galoiskey || (!data_buffer && buffer_elements > 0)) return 0;
    try {
        heongpu::HostVector<heongpu::Data64> temp_hv;
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        gk->cpp_galoiskey->get_data(temp_hv, cpp_stream);
        size_t count = std::min(buffer_elements, temp_hv.size());
        if (count > 0 && data_buffer) std::memcpy(data_buffer, temp_hv.data(), count * sizeof(uint64_t));
        return count;
    } catch (...) { return 0; }
}
int HEonGPU_CKKS_GaloisKey_SetData(HE_CKKS_GaloisKey* gk, const uint64_t* data_buffer, size_t num_elements, C_cudaStream_t stream) {
    if (!gk || !gk->cpp_galoiskey || (!data_buffer && num_elements > 0)) return -1;
    try {
        heongpu::HostVector<heongpu::Data64> input_hv(num_elements);
        if (num_elements > 0 && data_buffer) std::memcpy(input_hv.data(), data_buffer, num_elements * sizeof(uint64_t));
        hipStream_t cpp_stream = static_cast<hipStream_t>(stream);
        gk->cpp_galoiskey->set_data(input_hv, cpp_stream);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_GaloisKey_GetRotationIndices(HE_CKKS_GaloisKey* gk, C_RotationIndices_Data* out_indices_data) {
    if (!gk || !gk->cpp_galoiskey || !out_indices_data) return -1;
    
    out_indices_data->galois_elements_data = nullptr; out_indices_data->galois_elements_len = 0;
    out_indices_data->rotation_steps_data = nullptr; out_indices_data->rotation_steps_len = 0;
    try {
        const heongpu::RotationIndices& cpp_indices = gk->cpp_galoiskey->rot_indices();
        if (!cpp_indices.galois_elements.empty()) {
            out_indices_data->galois_elements_len = cpp_indices.galois_elements.size();
            out_indices_data->galois_elements_data = static_cast<int*>(malloc(out_indices_data->galois_elements_len * sizeof(int)));
            if (!out_indices_data->galois_elements_data) { HEonGPU_Free_C_RotationIndices_Data_Members(out_indices_data); return -2; }
            std::memcpy(out_indices_data->galois_elements_data, cpp_indices.galois_elements.data(), out_indices_data->galois_elements_len * sizeof(int));
        }
        if (!cpp_indices.rotation_steps.empty()) {
            out_indices_data->rotation_steps_len = cpp_indices.rotation_steps.size();
            out_indices_data->rotation_steps_data = static_cast<int*>(malloc(out_indices_data->rotation_steps_len * sizeof(int)));
            if (!out_indices_data->rotation_steps_data) { HEonGPU_Free_C_RotationIndices_Data_Members(out_indices_data); return -2; }
            std::memcpy(out_indices_data->rotation_steps_data, cpp_indices.rotation_steps.data(), out_indices_data->rotation_steps_len * sizeof(int));
        }
        return 0; // Success
    } catch (...) { HEonGPU_Free_C_RotationIndices_Data_Members(out_indices_data); return -3; }
}


} // extern "C"