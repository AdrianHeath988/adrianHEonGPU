#include "hip/hip_runtime.h"
#include "evaluationkey_c_api.h"
#include "heongpu_c_api_internal.h"
#include "heongpu.cuh"

#include "ckks/context.cuh"
#include "ckks/evaluationkey.cuh" // The C++ classes we are wrapping
#include "ckks/keygenerator.cuh"
#include "keygeneration.cuh"      // For heongpu::RotationIndices
#include "hostvector.cuh"
#include "schemes.h"
#include "storagemanager.cuh"
#include "random.cuh"

#include <vector>
#include <stdint.h>
#include <sstream>
#include <iostream>
#include <algorithm> // For std::min
#include <cstring>   // For std::memcpy
#include <new>       // For std::nothrow

// Define opaque structs

typedef struct HE_CKKS_RelinKey_s HE_CKKS_RelinKey;
typedef struct HE_CKKS_MultipartyRelinKey_s HE_CKKS_MultipartyRelinKey;
typedef struct HE_CKKS_GaloisKey_s HE_CKKS_GaloisKey;


// Helper to safely access underlying C++ HEContext pointer
static heongpu::HEContext<heongpu::Scheme::CKKS>* get_cpp_context(HE_CKKS_Context* context) {
    if (!context || !context->cpp_context) {
        std::cerr << "Error: Invalid HE_CKKS_Context pointer." << std::endl;
        return nullptr;
    }
    return context->cpp_context;
}



extern "C" {

// --- CKKS RelinKey Functions ---
HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Create(HE_CKKS_Context* context, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    try {
        auto cpp_obj = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (...) { return nullptr; }
}

void HEonGPU_CKKS_RelinKey_Delete(HE_CKKS_RelinKey* rk) {
    if (rk) { delete rk->cpp_relinkey; delete rk; }
}

HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Clone(const HE_CKKS_RelinKey* other_rk) {
    if (!other_rk || !other_rk->cpp_relinkey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*(other_rk->cpp_relinkey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_relinkey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_RelinKey_Assign_Copy(HE_CKKS_RelinKey* dest_rk, const HE_CKKS_RelinKey* src_rk) {
    if (!dest_rk || !dest_rk->cpp_relinkey || !src_rk || !src_rk->cpp_relinkey) return -1;
    try {
        *(dest_rk->cpp_relinkey) = *(src_rk->cpp_relinkey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_RelinKey_Save(HE_CKKS_RelinKey* rk, unsigned char** out_bytes, size_t* out_len) {
    if (!rk || !rk->cpp_relinkey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        rk->cpp_relinkey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; std::cerr << "RelinKey_Save: malloc failed\n"; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (const std::exception& e) { std::cerr << "RelinKey_Save exception: " << e.what() << std::endl; if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
      catch (...) { std::cerr << "RelinKey_Save unknown exception" << std::endl; if(*out_bytes){free(*out_bytes);*out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_RelinKey* HEonGPU_CKKS_RelinKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
    heongpu::Relinkey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_RelinKey* c_api_obj = nullptr;
    try {
        cpp_obj = new (std::nothrow) heongpu::Relinkey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss);
        }
        c_api_obj = new (std::nothrow) HE_CKKS_RelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) { std::cerr << "RelinKey_Load exception: " << e.what() << std::endl; delete cpp_obj; delete c_api_obj; return nullptr; }
      catch (...) { std::cerr << "RelinKey_Load unknown exception" << std::endl; delete cpp_obj; delete c_api_obj; return nullptr; }
}


bool HEonGPU_CKKS_RelinKey_IsOnDevice(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) return C_STORAGE_TYPE_INVALID;
    try { return (rk->cpp_relinkey->is_on_device()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
uint64_t* HEonGPU_CKKS_RelinKey_GetDataPointer(HE_CKKS_RelinKey* rk) {
    if (!rk || !rk->cpp_relinkey) {
        std::cerr << "GetDataPointer: Invalid RelinKey pointer." << std::endl;
        return nullptr;
    }
    try {
        // This directly calls the C++ `data()` method. Note: Data64 is uint64_t
        return reinterpret_cast<uint64_t*>(rk->cpp_relinkey->data());
    } catch (const std::exception& e) {
        std::cerr << "GetDataPointer failed with exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "GetDataPointer failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

uint64_t* HEonGPU_CKKS_RelinKey_GetDataPointerForLevel(HE_CKKS_RelinKey* rk, size_t level_index) {
    if (!rk || !rk->cpp_relinkey) {
        std::cerr << "GetDataPointerForLevel: Invalid RelinKey pointer." << std::endl;
        return nullptr;
    }
    try {
        // This directly calls the C++ `data(size_t)` method.
        return reinterpret_cast<uint64_t*>(rk->cpp_relinkey->data(level_index));
    } catch (const std::exception& e) {
        std::cerr << "GetDataPointerForLevel failed with exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "GetDataPointerForLevel failed due to an unknown exception." << std::endl;
        return nullptr;
    }
}

// --- CKKS MultipartyRelinKey Functions ---
HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Create(HE_CKKS_Context* context, const C_RNGSeed_Const_Data* seed_c_data, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed: Invalid context pointer." << std::endl;
        return nullptr;
    }
    if (!seed_c_data) {
        std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed: Seed pointer cannot be null." << std::endl;
        return nullptr;
    }

    try {
        // Convert C RNGSeed struct to C++ RNGSeed object
        heongpu::RNGSeed cpp_seed;
        if (seed_c_data->key_data && seed_c_data->key_len > 0) {
            cpp_seed.key_.assign(seed_c_data->key_data, seed_c_data->key_data + seed_c_data->key_len);
        }
        if (seed_c_data->nonce_data && seed_c_data->nonce_len > 0) {
            cpp_seed.nonce_.assign(seed_c_data->nonce_data, seed_c_data->nonce_data + seed_c_data->nonce_len);
        }
        if (seed_c_data->pstring_data && seed_c_data->pstring_len > 0) {
            cpp_seed.personalization_string_.assign(seed_c_data->pstring_data, seed_c_data->pstring_data + seed_c_data->pstring_len);
        }
        
        // Call the C++ constructor with the seed
        auto cpp_obj = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*cpp_h_context, cpp_seed);
        if (!cpp_obj) {
             std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed: C++ allocation failed." << std::endl;
            return nullptr;
        }

        auto c_api_obj = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_obj) {
            delete cpp_obj;
            std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed: C API wrapper allocation failed." << std::endl;
            return nullptr;
        }
        c_api_obj->cpp_mp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed with C++ exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_MultipartyRelinKey_Create failed due to an unknown C++ exception." << std::endl;
        return nullptr;
    }
}

void HEonGPU_CKKS_MultipartyRelinKey_Delete(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (mp_rk) { delete mp_rk->cpp_mp_relinkey; delete mp_rk; }
}

HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Clone(const HE_CKKS_MultipartyRelinKey* other_mp_rk) {
    if (!other_mp_rk || !other_mp_rk->cpp_mp_relinkey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*(other_mp_rk->cpp_mp_relinkey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_mp_relinkey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_MultipartyRelinKey_Assign_Copy(HE_CKKS_MultipartyRelinKey* dest_mp_rk, const HE_CKKS_MultipartyRelinKey* src_mp_rk) {
    if (!dest_mp_rk || !dest_mp_rk->cpp_mp_relinkey || !src_mp_rk || !src_mp_rk->cpp_mp_relinkey) return -1;
    try {
        *(dest_mp_rk->cpp_mp_relinkey) = *(src_mp_rk->cpp_mp_relinkey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_MultipartyRelinKey_Save(HE_CKKS_MultipartyRelinKey* mp_rk, unsigned char** out_bytes, size_t* out_len) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        mp_rk->cpp_mp_relinkey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (...) { if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_MultipartyRelinKey* HEonGPU_CKKS_MultipartyRelinKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
    heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_MultipartyRelinKey* c_api_obj = nullptr;
    try {
        heongpu::RNGSeed temp_seed;

        cpp_obj = new (std::nothrow) heongpu::MultipartyRelinkey<heongpu::Scheme::CKKS>(*cpp_h_context, temp_seed);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss);
        }
        c_api_obj = new (std::nothrow) HE_CKKS_MultipartyRelinKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_mp_relinkey = cpp_obj;
        return c_api_obj;
    } catch (...) { delete cpp_obj; delete c_api_obj; return nullptr; }
}


bool HEonGPU_CKKS_MultipartyRelinKey_IsOnDevice(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return C_STORAGE_TYPE_INVALID;
    try { return (mp_rk->cpp_mp_relinkey->is_on_device()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
uint64_t* HEonGPU_CKKS_MultipartyRelinKey_GetDataPointer(HE_CKKS_MultipartyRelinKey* mp_rk) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return nullptr;
    try { return reinterpret_cast<uint64_t*>(mp_rk->cpp_mp_relinkey->data()); }
    catch (...) { return nullptr; }
}

uint64_t* HEonGPU_CKKS_MultipartyRelinKey_GetDataPointerForLevel(HE_CKKS_MultipartyRelinKey* mp_rk, size_t level_index) {
    if (!mp_rk || !mp_rk->cpp_mp_relinkey) return nullptr;
    try { return reinterpret_cast<uint64_t*>(mp_rk->cpp_mp_relinkey->data(level_index)); }
    catch (...) { return nullptr; }
}


// --- CKKS GaloisKey Functions ---
HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Create(HE_CKKS_Context* context, bool store_in_gpu) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    try {

        auto cpp_obj = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) return nullptr;
        auto c_api_obj = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_galoiskey = cpp_obj;
        return c_api_obj;
    } catch (...) { return nullptr; }
}
HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Create_With_Shifts(HE_CKKS_Context* context, int* shift_vec, size_t num_shifts) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) {
        return nullptr;
    }
    // Note: It's valid for shift_vec to be null if num_shifts is 0 (for default keys).

    try {
        std::vector<int> shifts;
        if (shift_vec && num_shifts > 0) {
            shifts.assign(shift_vec, shift_vec + num_shifts);
        }
        auto c_api_obj = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_obj) {
            std::cerr << "GaloisKey_Create_With_Shifts: Failed to allocate C-API wrapper." << std::endl;
            return nullptr;
        }
        c_api_obj->cpp_galoiskey = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*cpp_h_context, shifts);
        if (!c_api_obj->cpp_galoiskey) {
            delete c_api_obj;
            std::cerr << "GaloisKey_Create_With_Shifts: Failed to allocate C++ Galoiskey object." << std::endl;
            return nullptr;
        }
        return c_api_obj;
        
    } catch (const std::exception& e) {
        std::cerr << "HEonGPU_CKKS_GaloisKey_Create_With_Shifts failed with exception: " << e.what() << std::endl;
        return nullptr;
    } catch (...) {
        std::cerr << "HEonGPU_CKKS_GaloisKey_Create_With_Shifts failed with an unknown exception." << std::endl;
        return nullptr;
    }
}
void HEonGPU_CKKS_GaloisKey_Delete(HE_CKKS_GaloisKey* gk) {
    if (gk) { delete gk->cpp_galoiskey; delete gk; }
}

HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Clone(const HE_CKKS_GaloisKey* other_gk) {
    if (!other_gk || !other_gk->cpp_galoiskey) return nullptr;
    try {
        auto cpp_clone = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*(other_gk->cpp_galoiskey));
        if (!cpp_clone) return nullptr;
        auto c_api_clone = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_clone) { delete cpp_clone; return nullptr; }
        c_api_clone->cpp_galoiskey = cpp_clone;
        return c_api_clone;
    } catch (...) { return nullptr; }
}

int HEonGPU_CKKS_GaloisKey_Assign_Copy(HE_CKKS_GaloisKey* dest_gk, const HE_CKKS_GaloisKey* src_gk) {
    if (!dest_gk || !dest_gk->cpp_galoiskey || !src_gk || !src_gk->cpp_galoiskey) return -1;
    try {
        *(dest_gk->cpp_galoiskey) = *(src_gk->cpp_galoiskey);
        return 0;
    } catch (...) { return -2; }
}

int HEonGPU_CKKS_GaloisKey_Save(HE_CKKS_GaloisKey* gk, unsigned char** out_bytes, size_t* out_len) {
    if (!gk || !gk->cpp_galoiskey || !out_bytes || !out_len) { if(out_bytes)*out_bytes=nullptr; if(out_len)*out_len=0; return -1; }
    *out_bytes = nullptr; *out_len = 0;
    try {
        std::ostringstream oss(std::ios::binary);
        gk->cpp_galoiskey->save(oss);
        std::string str_data = oss.str();
        *out_len = str_data.length();
        if (*out_len > 0) {
            *out_bytes = static_cast<unsigned char*>(malloc(*out_len));
            if (!*out_bytes) { *out_len = 0; return -2; }
            std::memcpy(*out_bytes, str_data.data(), *out_len);
        }
        return 0;
    } catch (...) { if(*out_bytes){free(*out_bytes); *out_bytes=nullptr;} *out_len=0; return -3; }
}

HE_CKKS_GaloisKey* HEonGPU_CKKS_GaloisKey_Load(HE_CKKS_Context* context, const unsigned char* bytes, size_t len, bool store_in_gpu_on_load) {
    heongpu::HEContext<heongpu::Scheme::CKKS>* cpp_h_context = get_cpp_context(context);
    if (!cpp_h_context) return nullptr;
    if (!bytes && len > 0) return nullptr;
     


    heongpu::Galoiskey<heongpu::Scheme::CKKS>* cpp_obj = nullptr;
    HE_CKKS_GaloisKey* c_api_obj = nullptr;
    try {
        cpp_obj = new (std::nothrow) heongpu::Galoiskey<heongpu::Scheme::CKKS>(*cpp_h_context);
        if (!cpp_obj) return nullptr;
        if (len > 0 && bytes) {
            std::string str_data(reinterpret_cast<const char*>(bytes), len);
            std::istringstream iss(str_data, std::ios::binary);
            cpp_obj->load(iss); // This should load the actual rot_indices_ from the stream
        }
        c_api_obj = new (std::nothrow) HE_CKKS_GaloisKey_s;
        if (!c_api_obj) { delete cpp_obj; return nullptr; }
        c_api_obj->cpp_galoiskey = cpp_obj;
        return c_api_obj;
    } catch (...) { delete cpp_obj; delete c_api_obj; return nullptr; }
}

bool HEonGPU_CKKS_GaloisKey_IsOnDevice(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return C_STORAGE_TYPE_INVALID;
    try { return (gk->cpp_galoiskey->is_on_device()); } catch (...) { return C_STORAGE_TYPE_INVALID; }
}
uint64_t* HEonGPU_CKKS_GaloisKey_GetDataPointerForLevel(HE_CKKS_GaloisKey* gk, size_t level_index) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    try { return reinterpret_cast<uint64_t*>(gk->cpp_galoiskey->data(level_index)); }
    catch (...) { return nullptr; }
}

uint64_t* HEonGPU_CKKS_GaloisKey_GetDataPointerForColumnRotation(HE_CKKS_GaloisKey* gk) {
    if (!gk || !gk->cpp_galoiskey) return nullptr;
    try { return reinterpret_cast<uint64_t*>(gk->cpp_galoiskey->c_data()); }
    catch (...) { return nullptr; }
}

} // extern "C"